//
//  main.cpp
//  DS
//
//  Created by Shubham Gupta on 31/03/17.
//  Copyright © 2017 Shubham Gupta. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <string.h>
#include <algorithm>
#include <fstream>
#include <sstream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iomanip>
#include <ctime>

using namespace std;

#define PI 3.1415926535897932
#define DPI 6.283185307179586
#define SPI 1.772453850905516
#define BOLTZ 1.380658e-23
#define AVOG 6.022169e26


void ALLOCATE_GAS();
void HARD_SPHERE();
void ARGON();
void IDEAL_NITROGEN();
void REAL_OXYGEN();
void IDEAL_AIR();
void REAL_AIR();
void HELIUM_ARGON_XENON();
void OXYGEN_HYDROGEN();
void INITIALISE_SAMPLES();
void DERIVED_GAS_DATA();
void SET_INITIAL_STATE_1D();
void MOLECULES_ENTER_1D();
void FIND_CELL_1D(double &,int &,int &);
void FIND_CELL_MB_1D(double &,int &,int &,double &);
void RVELC(double &,double &,double &);
void SROT(int &,double &,double &);
void SVIB(int &,double &,int &, int&);
void SELE(int &,double &,double &);
void CQAX(double&,double &,double&);
void LBS(double,double,double&);
void REFLECT_1D(int&,int,double&);
void RBC(double &, double &, double & , double &, double &,double &);
void AIFX(double & ,double &, double & , double &, double &, double&, double &, double&);
void REMOVE_MOL(int &);
void INDEX_MOLS();
void SAMPLE_FLOW();
void ADAPT_CELLS_1D();
void EXTEND_MNM(double);
void DISSOCIATION();
void ENERGY(int ,double &);
void COLLISIONS();
void SETXT();
void READ_RESTART();
void WRITE_RESTART();
void READ_DATA();
void OUTPUT_RESULTS();
void MOLECULES_MOVE_1D();



class Managed 
{
public:
  void *operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
  }

  void operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
  }
};

class CALC : public Managed
{
public:
    //declares the variables associated with the calculation
    int  NVER,MVER,IMEG,NREL,MOLSC,ISF,ISAD,ISECS,IGS,IREM,NNC,IMTS,ERROR,NLINE,ICLASS,NCLASS,NMCC,NMI,NMP,ICN;
    double FTIME,TLIM,FNUM,DTM,TREF,TSAMP,TOUT,SAMPRAT,OUTRAT,RANF,TOTCOLI,TOTMOVI,TENERGY,DTSAMP,DTOUT,TPOUT,FRACSAM,TOTMOV,TOTCOL,ENTMASS,ENTREM,CPDTM,TPDTM,TNORM,FNUMF;
    double *VNMAX,*TDISS,*TRECOMB,*ALOSS,*EME,*AJM,*COLL_TOTCOL;
    double **TCOL;

    void d_allocate(int x, double*&arr){
        hipMallocManaged(&arr, x*sizeof(double));
    }
    void d_allocate(int x, int y, double**&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for(int i =0; i< x; ++i)
            hipMallocManaged(&arr[i], y*sizeof(double));
    }
    //NVER.MVER.NREL the version number
    //IMEG the initial number of megabytes to be used by the program
    //MOLSC the target number of molecules per sampling cell
    //FTIME the flow time
    //TLIM the time at which the calculation stops
    //FNUM the number of real molecules represented by each simulated molecule
    //CPDTM the maximum number of collisions per time step (standard 0.2)
    //TPDTM the maximum number of sampling cell transit times of the flow per time step
    //TOTMOV total molecule moves
    //TOTCOL total collisions
    //TDISS(L) dissociations of species L since sample reset
    //TRECOMB(L) recombinations of species L since sample reset
    //ENTMASS the current entry mass of which a fraction FREM is to be removed
    //ENTREM the remainder (always negative) after molecule removal
    //VNMAX(L) the maximum normal velocity component of species L
    //TCOL species dependent collision counter
    //ISF 0,1 for steady, unsteady flow sampling
    //ISAD 0,1 to not automatically adapt cells each output interval in unsteady sampling, 1 to automatically adapt
    //ISECS 0,1 for no secondary stream,a secondary stream that applies for positive values of x
    //IREM data item to set type of molecule removal
    //NNC 0 for normal collisions, 1 for nearest neighbor collisions
    //IMTS 0 for uniform move time steps, 1 for time steps that vary over the cells, 2 for fixed time steps
    //IGS 0 for initial gas, 1 for stream(s) or reference gas
    //ICLASS class of flow
    //NCLASS the dimension of PX for the class of flow
    //NMCC desired number of molecules in a collision cell
    //NMI the initial number of molecules
    //TNORM normalizing time (may vary e.g. mean collision time , or a transit time)
    //ALOSS(L) number of molecules of speciel L lost in the move rourine
    //EME(L) number of species L that enter the front boundary
    //AJM(L) the adjustment number to allow for negative downstream entry numbers
    //NMP the number of molecules at the start of the move routine
    //ICN 0 if molecules with ITYPE(2)=4 are not kept constant, 1 to keep molecule number constant
    //FNUMF adjustment factor that is applied to automatically generated value
};

class MOLECS : public Managed
{
    //declares the variables associated with the molecules
public:
    int *IPCELL,*IPSP,*ICREF,*IPCP;
    int **IPVIB;
    
    void i_allocate(int x, int *&arr){
        hipMallocManaged(&arr, x*sizeof(int));
    }
    void i_allocate(int x, int y, int **&arr){
        hipMallocManaged(&arr, x*sizeof(int));
        for(int i =0; i< x; ++i)
            hipMallocManaged(&arr[i], y*sizeof(int));
    }
    
    double **PX,**PV;
    double *PTIM,*PROT,*PELE;
    
    void d_allocate(int x, double *&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        
    }
    void d_allocate(int x, int y, double **&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for(int i =0; i< x; ++i){
            try{
                hipMallocManaged(&arr[i], y*sizeof(double));
            }
            catch (std::bad_alloc& ba){
                std::cerr << "bad_alloc caught: " << ba.what() << '\n';
            }
        }
    }
    int NM,MNM;
    
    //PX(1,2 or 3,N) x,y,z position coordinates of molecule N
    //PTIM(N) molecule time
    //IPSP(N) the molecular species
    //IPCELL(N) the collision cell number
    //ICREF the cross-reference array (molecule numbers in order of collision cells)
    //IPCP(N) the code number of the last collision partner of molecule
    //PV(1-3,N) u,v,w velocity components
    //PROT(N) rotational energy
    //IPVIB(K,N) level of vibrational mode K of molecule N
    //PELE(N) electronic energy
    //NM number of molecules
    //MNM the maximum number of molecules
    
};

class GAS : public Managed
{
    
    //declares the variables associated with the molecular species and the stream definition
public:
    double RMAS,CXSS,RGFS,VMPM,FDEN,FPR,FMA,FPM,CTM;
    double FND[3],FTMP[3],FVTMP[3],VFX[3],VFY[3],TSURF[3],FSPEC[3],VSURF[3];
    double *ERS,*CR,*TNEX,*PSF,*SLER,*FP;
    double **FSP,**SP,**SPR,**SPV,**VMP;
    double ***SPM,***SPVM,***ENTR,***QELC,***SPRT;
    double ****SPEX,****SPRC,****SPRP;
    double *****SPREX;
    void d_allocate(int x, double *&arr){
        hipMallocManaged(&arr, x*sizeof(double));
    }
    void d_allocate(int x, int y, double **&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for(int i =0; i< x; ++i)
            hipMallocManaged(&arr[i], y*sizeof(double));
    }
    void d_allocate(int x, int y, int z, double***&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for (int i = 0; i < x; ++i)
        {
            hipMallocManaged(&arr[i], y*sizeof(double));
            for (int j = 0; j < y; ++j)
                hipMallocManaged(&arr[i][j], z*sizeof(double));
        }
        
    }
    void d_allocate(int x, int y, int z, int w, double ****&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for (int i = 0; i < x; ++i)
        {
            hipMallocManaged(&arr[i], y*sizeof(double));
            for (int j = 0; j < y; ++j)
            {
                hipMallocManaged(&arr[i][j], z*sizeof(double));
                for(int k=0; k<z; ++k)
                    hipMallocManaged(&arr[i][j][k], w*sizeof(double));
            }
        }
        
    }
    void d_allocate(int x, int y, int z, int w, int v, double*****&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for (int i = 0; i < x; ++i)
        {
            hipMallocManaged(&arr[i], y*sizeof(double));
            for (int j = 0; j < y; ++j)
            {
                hipMallocManaged(&arr[i][j], z*sizeof(double));
                for(int k=0; k<z; ++k)
                {
                    hipMallocManaged(&arr[i][j][k], w*sizeof(double));
                    for(int l=0; l<w; ++l)
                        hipMallocManaged(&arr[i][j][k][l], v*sizeof(double));
                }
            }
        }
    }
    
    int MSP,MMVM,MMRM,MNSR,IGAS,MMEX,MEX,MELE,MVIBL;
    int *ISP,*ISPV,*NELL;
    int **ISPR,**LIS,**LRS,**ISRCD,**ISPRC,**ISPRK,**TREACG,**TREACL,**NSPEX,**NSLEV;
    int ***ISPVM,***NEX;
    int ****ISPEX;
    void i_allocate(int x, int *&arr){
        hipMallocManaged(&arr, x);
    }
    void i_allocate(int x, int y, int **&arr){
        hipMallocManaged(&arr, x*sizeof(int));
        for(int i =0; i< x; ++i)
            hipMallocManaged(&arr[i], y*sizeof(int));
    }
    void i_allocate(int x, int y, int z, int ***&arr){
        hipMallocManaged(&arr, x*sizeof(int));
        for (int i = 0; i < x; ++i)
        {
            hipMallocManaged(&arr[i], y*sizeof(int));
            for (int j = 0; j < y; ++j)
                hipMallocManaged(&arr[i][j], z*sizeof(int));
        }
        
    }
    void i_allocate(int x, int y, int z, int w, int ****&arr){
        hipMallocManaged(&arr, x*sizeof(int));
        for (int i = 0; i < x; ++i)
        {
            hipMallocManaged(&arr[i], y*sizeof(int));
            for (int j = 0; j < y; ++j)
            {
                hipMallocManaged(&arr[i][j], z*sizeof(int));
                for(int k=0; k<z; ++k)
                    hipMallocManaged(&arr[i][j][k], w*sizeof(int));
            }
        }
    }
    
    //MSP the number of molecular species
    //MMVM the maximum number of vibrational modes of any species
    //MEX number of exchange or chain reactions
    //MELE the maximum number of electronic states of any molecule
    //MVIBL the maximum number of vibrational levels for detailed balance lists
    //MMEX the maximum number of exchange reactions involving the same precollision pair of molecules
    //MMRM 0 if gass is completely monatomic, 1 if some species have rotation
    //MNSR the number oF surface reactions
    //SP(1,L) the reference diameter of species L
    //SP(2,L) the reference temperature of species L
    //SP(3,L) the viscosity-temperature power law of species L
    //SP(4,L) the reciprocal of the VSS scattering parameter
    //SP(5,L) molecular mass of species L
    //SP(6,L) the heat of formation at 273 K.
    //ISPR(1,L) number of rotational degrees of freedom of species L
    //ISPR(2,L) 0,1 for constant, polynomial rotational relaxation collision number
    //SPR(1,L) constant rotational relaxation collision number of species L
    //          or the constant in a second order polynomial in temperature
    //SPR(2,L) the coefficient of temperature in the polynomial
    //SPR(3,L) the coefficient of temperature squared in the polynomial
    //SPM(1,L,M) the reduced mass for species L,M
    //SPM(2,L,M) the reference collision cross-section for species L,M
    //SPM(3,L,M) the mean value of the viscosity-temperature power law
    //SPM(4,L,M) the reference diameter for L,M collisions
    //SPM(5,L,M) the reference temperature for species L,M
    //SPM(6,L,M) reciprocal of the gamma function of (5/2-w) for species L,M
    //SPM(7,L,M) rotational relaxation collision number for species L,M, or const in polynomial
    //SPM(8,L,M) reciprocal of VSS scattering parameter
    //ISPV(L) the number of vibrational modes
    //SPVM(1,K,L) the characteristic vibrational temperature
    //SPVM(2,K,L) constant Zv, or reference Zv for mode K
    //SPVM(3,K,L) -1. for constant Zv, or reference temperature
    //SPVM(4,K,L) the characteristic dissociation temperature
    //SPVM(5,K,L) the arbitrary rate reduction factor
    //ISPVM(1,K,L) the species code of the first dissociation product
    //ISPVM(2,K,L) the species code of the second dissociation product
    //NELL(L) the number of electronic levels of species L
    //QELC(N,M,L) for up to M levels of form g*exp(-a/T) in the electronic partition function for species L
    //            N=1 for the degeneracy g
    //            N=2 for the coefficient a
    //            N=3 for the ratio of the excitation cross-section to the elastic cross-section
    //ISPRC(L,M) the species of the recombined molecule from species L and M
    //ISPRK(L,M) the applicable vibrational mode of this species
    //SPRC(1,L,M,K) the constant a in the ternary collision volume
    //SPRC(2,L,M,K) the temperature exponent b in the ternary collision volume
    //SPRT(1,L,M) lower temperature value for SPRP
    //SPRT(2,L,M) higher temperature value for SPRP
    //SPRP(1,L,M,K) the cumulative dissociation distribution to level K for products L and M at the lower temperature
    //SPRP(2,L,M,K) ditto at higher temperature, for application to post-recombination molecule//
    //NSPEX(L,M) the number of exchange reactios with L,M as the pre-collision species
    //in the following variables, J is the reaction number (1 to NSPEX(L,M))
    //ISPEX(J,1,L,M) the species that splits in an exchange reaction
    //ISPEX(J,2,L,M) the other pre-reaction species (all ISPEX are set to 0 if no exchange reaction)
    //ISPEX(J,3,L,M) the post-reaction molecule that splits in the opposite reaction
    //ISPEX(J,4,L,M) the other post-reaction species
    //ISPEX(J,5,L,M) the vibrational mode of the molecule that splits
    //ISPEX(J,6,L,M) degeneracy of this reaction
    //ISPEX(J,7,L,M) the vibrational mode of the molecule that splits
    //SPEX(1,J,L,M) the constant a in the reaction probability for the reverse reaction
    //SPEX(2,J,L,M) the temperature exponent b in the reaction probability (reverse reaction only)
    //SPEX(3,J,L,M)  for the heat of reaction
    //SPEX(4,J,L,M)   the lower temperature for SPREX
    //SPEX(5,J,L,M)   the higher temperature for SPREX
    //SPEX(6,J,L,M)   the energy barrier
    //SPREX(1,J,L,M,K) at lower temperature, the Jth reverse exchange reaction of L,M cumulative level K viv. dist of post reac mol
    //SPREX(2,J,L,M,K) ditto at higher temperature
    //TNEX(N) total number of exchange reaction N
    //NEX(N,L,M) the code number of the Nth exchange or chain reaction in L,M collisions
    //RMAS reduced mass for single species case
    //CXSS reference cross-section for single species case
    //RGFS reciprocal of gamma function for single species case
    //for the following, J=1 for the reference gas and/or the minimum x boundary, J=2 for the secondary sream at maximum x boundary
    //FND(J) stream or reference gas number density
    //FTMP(J) stream temperature
    //FVTMP(J) the vibrational and any electronic temperature in the freestream
    //VFX(J)  the x velocity components of the stream
    //VFY(J) the y velocity component in the stream
    //FSP(N,J)) fraction of species N in the stream
    //FMA stream Mach number
    //VMP(N,J) most probable molecular velocity of species N at FTMP(J)
    //VMPM the maximum value of VMP in stream 1
    //ENTR(M,L,K) entry/removal information for species L at K=1 for 1, K=2 for XB(2)
    //    M=1 number per unut time
    //    M=2 remainder
    //    M=3 speed ratio
    //   M=4 first constant
    //    M=5 second constant
    //    M=6 the maxinum normal velocity component in the removal zone (> XREM)
    //LIS(1,N) the species code of the first incident molecule
    //LIS(2,N) the species code of the second incident molecule (0 if none)
    //LRS(1,N) the species code of the first reflected molecule
    //LRS(2,N) the species code of the second reflected molecule (0 if none)
    //LRS(3,N) the species code of the third reflected molecule (0 if none)
    //LRS(4,N) the species code of the fourth reflected molecule (0 if none)
    //LRS(5,N) the species code of the fifth reflected molecule (0 if none)
    //LRS(6,N) the species code of the sixth reflected molecule (0 if none)
    //ERS(N) the energy of the reaction (+ve for recombination, -ve for dissociation)
    //NSRSP(L) number of surface reactions that involve species L as incident molecule
    //ISRCD(N,L) code number of Nth surface reaction with species L as incident molecule
    //CTM mean collision time in stream
    //FPM mean free path in stream
    //FDEN stream 1 density
    //FPR stream 1 pressure
    //FMA stream 1 Mach number
    //RMAS reduced mass for single species case
    //CXSS reference cross-section for single species case
    //RGFS reciprocal of gamma function for single species case
    //CR(L) collision rate of species L
    //FP(L) mean free path of species L
    //TREACG(N,L) the total number of species L gained from reaction type N=1 for dissociation, 2 for recombination, 3 for forward exchange, 4 for reverse exchange
    //TREACL(N,L) the total number of species L lost from reaction type N=1 for dissociation, 2 for recombination, 3 for forward exchange, 4 for reverse exchange
    //NSLEV(2,L)  1 exo, 2 endo: vibrational levels to be made up for species L in detailed balance enforcement after reaction
    //SLER(L) rotational energy to be made up for species L in detailed balance enforcement after exothermic reaction
};
class OUTPUT : public Managed
{
public:
    //declares the variables associated with the sampling and output
    int NSAMP,NMISAMP,NOUT,NDISSOC,NRECOMB,NTSAMP;
    //int NDISSL[201];
    int *NDISSL;
    OUTPUT(){
        hipMallocManaged(&NDISSL,201*sizeof(int));
    };
    double TISAMP,XVELS,YVELS,AVDTM;
    double *COLLS,*WCOLLS,*CLSEP,*SREAC,*STEMP,*TRANSTEMP,*ROTTEMP,*VIBTEMP,*ELTEMP;
    double **VAR,**VARS,**CSSS,**SUMVIB;
    double ***CS,***VARSP,***VIBFRAC;
    double ****CSS;
    void d_allocate(int x, double *&arr){
        hipMallocManaged(&arr, x*sizeof(double));
    }
    void d_allocate(int x, int y, double **&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for(int i =0; i< x; ++i)
            hipMallocManaged(&arr[i], y*sizeof(double));
    }
    void d_allocate(int x, int y, int z, double ***&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for (int i = 0; i < x; ++i)
        {
            hipMallocManaged(&arr[i], y*sizeof(double));
            for (int j = 0; j < y; ++j)
                hipMallocManaged(&arr[i][j], z*sizeof(double));
        }
    }
    void d_allocate(int x, int y, int z, int w, double ****&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for (int i = 0; i < x; ++i)
        {
            hipMallocManaged(&arr[i], y*sizeof(double));
            for (int j = 0; j < y; ++j)
            {
                hipMallocManaged(&arr[i][j], z*sizeof(double));
                for(int k=0; k<z; ++k)
                    hipMallocManaged(&arr[i][j][k], w*sizeof(double));
            }
        }
    }
    //NSAMP the number of samples
    //TISAMP the time at which the sampling was last reset
    //MNISAMP the number of molecules at the last reset
    //AVDTM the average value of DTM in the cells
    //NOUT the number of output intervals
    //COLLS(N) total number of collisions in sampling cell N
    //WCOLLS(N) total weighted collisins in N
    //CLSEP(N) sum of collision pair separation in cell N
    //CS(0,N,L) sampled number of species L in cell N
    //CS(1,N,L) sampled weighted number of species L in cell N
    //--all the following CS are weighted sums
    //CS(2,N,L), CS(3,N,L), CS(4,N,L) sampled sum of u, v, w
    //CS(5,N,L), CS(6,N,L), CS(7,N,L) sampled sum of u*u, v*v, w*w
    //CS(8,N,L) sampled sum of rotational energy of species L in cell N
    //CS(9,N,L) sampled sum of electronic energy of species L in cell N
    //CS(9+K,N,L) sampled sum of vibrational level of species L in cell N
    //              K is the mode
    //
    //in CSS, M=1 for incident molecules and M=2 for reflected molecules
    //J=1 for surface at x=XB(1), 2 for surface at x=XB(2)
    //
    //CSS(0,J,L,M) number sum of molecules of species L
    //CSS(1,J,L,M) weighted number sum of molecules of species L
    //--all the following CSS are weighted
    //CSS(2,J,L,M) normal momentum sum to surface
    //CSS(3,J,L,M) y momentum sum to surface
    //CSS(4,J,L,M) z momentum sum to surface
    //CSS(5,J,L,M) tranlational energy sum to surface
    //CSS(6,J,L,M) rotational energy sum to surface
    //CSS(7,J,L,M) vibrational energy sum to the surface
    //CSS(8,J,L,M) electronic energy sum to the surface
    //
    //CSSS(1,J) weighted sum (over incident AND reflected molecules) of 1/normal vel. component
    //--all the following CSSS are weighted
    //CSSS(2,J) similar sum of molecular mass / normal vel. component
    //CSSS(3,J) similar sum of molecular mass * parallel vel. component / normal vel. component
    //CSSS(4,J) similar sum of molecular mass * speed squared / normal vel. component
    //CSSS(5,J) similar sum of rotational energy / normal vel. component
    //CSSS(6,J) similar sum of rotational degrees of freedom /normal velocity component
    //
    //SREAC(N) the number of type N surface reactions
    //
    //VAR(M,N) the flowfield properties in cell N
    //M=1 the x coordinate
    //M=2 sample size
    //M=3 number density
    //M=4 density
    //M=5 u velocity component
    //M=6 v velocity component
    //M=7 w velocity component
    //M=8 translational temperature
    //M=9 rotational temperature
    //M=10 vibrational temperature
    //M=11 temperature
    //M=12 Mach number
    //M=13 molecules per cell
    //M=14 mean collision time / rate
    //M=15 mean free path
    //M=16 ratio (mean collisional separation) / (mean free path)
    //M=17 flow speed
    //M=18 scalar pressure nkT
    //M=19 x component of translational temperature TTX
    //M=20 y component of translational temperature TTY
    //M=21 z component of translational temperature TTZ
    //M=22 electronic temperature
    //
    //VARSP(M,N,L) the flowfield properties for species L in cell N
    //M=0 the sample size
    //M=1 the fraction
    //M=2 the temperature component in the x direction
    //M=3 the temperature component in the y direction
    //M=4 the temperature component in the z direction
    //M=5 the translational temperature
    //M=6 the rotational temperature
    //M=7 the vibrational temperature
    //M=8 the temperature
    //M=9 the x component of the diffusion velocity
    //M=10 the y component of the diffusion velocity
    //M=11 the z component of the diffusion velocity
    //M=12 the electronic temperature
    //
    //VARS(N,M) surface property N on interval L of surface M
    //
    //N=0 the unweighted sample (remainder of variables are weighted for cyl. and sph. flows)
    //N=1 the incident sample
    //N=2 the reflected sample
    //N=3 the incident number flux
    //N=4 the reflected number flux
    //N=5 the incident pressure
    //N=6 the reflected pressure
    //N=7 the incident parallel shear tress
    //N=8 the reflected parallel shear stress
    //N=9 the incident normal-to-plane shear stress
    //N=10 the reflected normal shear stress
    //N=11 the incident translational heat flux
    //N=12 the reflected translational heat fluc
    //N=13 the incident rotational heat flux
    //N=14 the reflected rotational heat flux
    //N=15 the incident vibrational heat flux
    //N=16 the reflected vibrational heat flux
    //N=17 the incident heat flux from surface reactions
    //N=18 the reflected heat flux from surface reactions
    //N=19 slip velocity
    //N=20 temperature slip
    //N=21 rotational temperature slip
    //N=22 the net pressure
    //N=23 the net parallel in-plane shear
    //N=24 the net parallel normal-to-plane shear
    //N=25 the net translational energy flux
    //N=26 the net rotational heat flux
    //N=27 the net vibrational heat flux
    //N=28 the heat flux from reactions
    //N=29 total incident heat transfer
    //N=30 total reflected heat transfer
    //N=31 net heat transfer
    //N=32 surface temperature   --not implemented
    //N=33 incident electronic energy
    //N=34 reflected electronic energy
    //N=35 net electronic energy
    //N=35+K the percentage of species K
    //
    //COLLS(N) the number of collisions in sampling cell N
    //WCOLLS(N) weighted number
    //CLSEP(N) the total collision partner separation distance in sampling cell N
    //
    //VIBFRAC(L,K,M) the sum of species L mode K in level M
    //SUMVIB(L,K) the total sample in VIBFRAC
    //
    //THE following variables apply in the sampling of distribution functions
    //(some are especially for the dissociation of oxygen
    //
    //NDISSOC the number of dissociations
    //NRECOMB the number of recombinations
    //NDISSL(L) the number of dissociations from level
    //NTSAMP the number of temperature samples
    //STEMP(L) the temperature of species L
    //TRANSTEMP(L) the translational temperature of species N
    //ROTTEMP(L) rotational temperature of species N
    //VIBTEMP(L) vibrational temperature of species N
    //ELTEMP(L) electronic temperature of species N
    //
};

class GEOM_1D : public Managed
{
public:
    //declares the variables associated with the flowfield geometry and cell structure
    //for homogeneous gas and one-dimensional flow studies
    int NCELLS,NCCELLS,NCIS,NDIV,MDIV,ILEVEL,IFX,JFX,IVB,IWF;
    //int ITYPE[3];
    int *ITYPE;
    int *ICELL;
    int ** ICCELL,**JDIV;
    void i_allocate(int x, int *&arr){
        hipMallocManaged(&arr, x*sizeof(int));
    }
    void i_allocate(int x, int y, int **&arr){
        hipMallocManaged(&arr, x*sizeof(int));
        for(int i =0; i< x; ++i)
            hipMallocManaged(&arr[i], y*sizeof(int));
    }
    double DDIV,XS,VELOB,WFM,AWF,FREM,XREM;
    //double XB[3];
    double *XB;
    double **CELL,**CCELL;
    void d_allocate(int x, int y, double**&arr){
        hipMallocManaged(&arr, x*sizeof(double));
        for(int i =0; i< x; ++i)
            hipMallocManaged(&arr[i], y*sizeof(double));
    }
    GEOM_1D(){
        hipMallocManaged(&ITYPE, 3*sizeof(int));
        hipMallocManaged(&XB, 3*sizeof(double));   
    }
    //
    //XB(1), XB(2) the minimum, maximum x coordinate
    //DDIV the width of a division
    //ITYPE(K) the tpe of boundary at the minimum x (K=1) and maximum x (K=2) boundaries
    //          0 for a stream boundary
    //          1 for a plane of symmetry
    //          2 for a solid surface
    //          3 for a vacuum
    //NCELLS the number of sampling cells
    //NCCELLS the number of collision cells
    //NCIS the number of collision cells in a sampling cell
    //  MDIV the maximum number of sampling cell divisions at any level of subdivision
    //IVB 0,1 for stationary, moving outer boundary
    //IWF 0 for no radial weighting factors, 1 for radial weighting factors
    //WFM, set in data as the maximum weighting factor, then divided by the maximum radius
    //AWF overall ratio of real to weighted molecules
    //VELOB the speed of the outer boundary
    //ILEV level of subdivision in adaption (0 before adaption)
    //JDIV(N,M) (-cell number) or (start address -1 in JDIV(N+1,M), where M is MDIV
    //IFX 0 for plane flow, 1 for cylindrical flow, 3 for spherical flow
    //JFX  IFX+1
    //CELL(M,N) information on sampling cell N
    //    M=1 x coordinate
    //    M=2 minimum x coordinate
    //    M=3 maximum x cooedinate
    //    M=4 volume
    //ICELL(N) number of collision cells preceding those in sampling cell N
    //CCELL(M,N) information on collision cell N
    //    M=1 volume
    //    M=2 remainder in collision counting
    //    M=3 half the allowed time step
    //    M=4 maximum value of product of cross-section and relative velocity
    //    M=5 collision cell time
    //ICCELL(M,N) integer information on collision cell N
    //    M=1 the (start address -1) in ICREF of molecules in collision cell N
    //    M=2 the number of molecules in collision cell N
    //    M=3 the sampling cell in which the collision cell lies
    //FREM fraction of molecule removal
    //XREM the coordinate at which removal commences
    //
};

double colltime=0.0;
clock_t start;
fstream file_9;
fstream file_18;
CALC *calc = new CALC;
GAS *gas = new GAS;
MOLECS *molecs = new MOLECS;
GEOM_1D *geom = new GEOM_1D;
OUTPUT *output =new OUTPUT;


// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                              (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
// old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));
//     } while (assumed != old);
//     return __longlong_as_double(old);
// }

__device__ float generate( hiprandState* globalState, int ind )
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__device__ void SROT(hiprandState* globalState, int &L,double &TEMP,double &ROTE, GAS* gas, CALC *calc)
{
    int I;
    double A,B,ERM;
    //
    if(gas->ISPR[1][L] == 2){
        // CALL RANDOM_NUMBER(RANF)
        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
        ROTE=-logf(calc->RANF)*BOLTZ*TEMP;   //equation (4.8)
    }
    else{
        A=0.5e00*gas->ISPR[1][L]-1.e00;
        I=0;
        while(I == 0){
            // CALL RANDOM_NUMBER(RANF)
            calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
            ERM=calc->RANF*10.e00;
            //there is an energy cut-off at 10 kT
            B=(powf((ERM/A),A))*expf(A-ERM);      //equation (4.9)
            // CALL RANDOM_NUMBER(RANF)
            calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
            if(B > calc->RANF) I=1;
        }
        ROTE=ERM*BOLTZ*TEMP;
    }
    return;
}

__device__ void SVIB(hiprandState* globalState, int &L,double &TEMP,int &IVIB, int &K, GAS *gas, CALC *calc)
{
    //sets a typical vibrational state at temp. TEMP of mode K of species L
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int N;
    //    double TEMP;
    //    int IVIB;
    //
    // CALL RANDOM_NUMBER(RANF)
    calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
    N=-logf(calc->RANF)*TEMP/gas->SPVM[1][K][L];                 //eqn(4.10)
    //the state is truncated to an integer
    IVIB=N;
}

__device__ void LBS(hiprandState* globalState, double XMA,double XMB,double &ERM)
{
    //selects a Larsen-Borgnakke energy ratio using eqn (11.9)
    //
    double PROB,RANF;
    int I,N;
    //
    //I is an indicator
    //PROB is a probability
    //ERM ratio of rotational to collision energy
    //XMA degrees of freedom under selection-1
    //XMB remaining degrees of freedom-1
    //
    I=0;
    while(I == 0){
        // CALL RANDOM_NUMBER(RANF)
        RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
        ERM=RANF;
        if((XMA < 1.e-6) || (XMB < 1.e-6)){
            //    IF (XMA < 1.E-6.AND.XMB < 1.E-6) RETURN
            //above can never occur if one mode is translational
            if(XMA < 1.e-6) PROB=powf((1.e00-ERM),XMB);
            if(XMB < 1.e-6) PROB=powf((1.e00-ERM),XMA);
        }
        else
            PROB=powf(((XMA+XMB)*ERM/XMA),XMA)*powf(((XMA+XMB)*(1.e00-ERM)/XMB),XMB);
        
        // CALL RANDOM_NUMBER(RANF)
        RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
        if(PROB > RANF) I=1;
    }
    //
    return;
}


int main()
{
    // //CALC calc;
    // //MOLECS molecs;
    // //GAS gas;
    // //OUTPUT output;
    // //GEOM_1D geom;
    //
    // IMPLICIT NONE\

    //


    int IRUN,ICONF,N,M,IADAPT,IRETREM,ISET;
    double A;
    //
    fstream file_7;
    
    calc->NVER=1;          //for major changes, e.g. to basic architecture
    calc->MVER=1 ;         //significant changes, but must change whenever the data in a DSnD.DAT file changes
    calc->NREL=1  ;        //the release number
    //
    //***********************
    //set constants
    // PI=3.1415926535897932D00
    // DPI=6.283185307179586D00
    // SPI=1.772453850905516D00
    // BOLTZ=1.380658D-23
    // AVOG=6.022169D26
    //***********************
    //
    //*************************************************
    //****  ADJUSTABLE COMPUTATIONAL PARAMETERS  ****
    //*************************************************
    //
    calc->NMCC=15;    //DEFAULT=15--desired number of simulated molecules in a collision cell
    //
    calc->CPDTM=0.2;   //DEFAULT=0.2--fraction of the local mean collision time that is the desired maximum time step
    //
    calc->TPDTM=0.5 ;  //DEFAULT=0.5--the fraction or multiple of a sampling cell transit time that is the desired maximum time step
    //
    calc->NNC=1;       //DEFAULT=0--0 to select collision partner randomly from collision cell, 1 for nearest-neighbor collisions
    //
    calc->SAMPRAT=5;   //DEFAULT=5--the number of time steps in a sampling interval
    //
    calc->OUTRAT=10;   //50   //DEFAULT=50--the number of flow samples in a file output interval
    //
    calc->FRACSAM=0.5; //0.5 //DEFAULT=0.5--fraction of the output interval interval over which a time-averaged sample is taken in an unsteady flow
    //
    calc->ISAD=0;      //DEFAULT=0--0,1 to not adapt, to adapt cells automatically at start of output interval in an unsteady flow (not yet implemented)
    //
    calc->IMTS=2;      //DEFAULT=0--0 to set the move time step to the instantaneous overall time step that changes with time
    //                         1 to use a cell dependent collision time
    //                         2 to keep the time step fixed at the initial value
    //
    calc->FNUMF=1;   //DEFAULT=1--adjustment factor to the automatically generated value for the number of real molecules
    //                                  that are represented by each simulated molecule.
    //          (The adjustment may be large because the automatic setting assumes that the whole flowfield is at the stream conditions.)
    //
    //automatic adjustments may be applied for some application classes (e.g homogeneous gas studies)
    //
    calc->TLIM=1.e-5;  //DEFAULT=1.D20 sets an indefinite run - set if a define STOP time is required
    //
    //************************************************
    //
    //open a diagnostic file and check whether an instance of the program is already running
    //
    //    fstream file_9;
    cout<<"DSMC PROGRAM"<<endl;
    file_9.open("DIAG.TXT", ios::trunc | ios::out);
    if(file_9.is_open()){
        file_9<<"File DIAG.TXT has been opened"<<endl;
        cout<<"File DIAG.TXT has been opened"<<endl;
    }
    else{
        cout<<"Stop the DS1.EXE that is already running and try again"<<endl;
        //return 0;
    }
    
    //    OPEN (9,FILE='DIAG.TXT',FORM='FORMATTED',STATUS='REPLACE')
    //    WRITE (9,*,IOSTAT=ERROR)
    //    IF (ERROR /= 0) THEN
    //    WRITE (*,*) 'Stop the DS1.EXE that is already running and try again'
    //    STOP
    //    ELSE
    //    WRITE (9,*) 'File DIAG.TXT has been opened'
    //    END IF
    
    //
    //open a molecule number history file
    //OPEN (13,FILE='MolNum.DAT',FORM='FORMATTED',STATUS='REPLACE')
    //
    //initialise run variables
    IRUN=0;
    geom->IVB=0;  //will be reset to 1 by restart program if there is a moving wall
    //
    while((IRUN < 1) || (IRUN > 2)){
        cout<< "DSMC Version" <<calc->NVER<<'.'<<calc->MVER<<'.'<<calc->NREL<<endl;
        cout<< "enter 1 to continue a current run"<<endl;
        cout<< "enter 2 to start a new run :-"<<endl;
        //
        cin>>  IRUN;
    }
    if(IRUN == 1) file_9<< "Continuing an existing run"<<endl;//WRITE (9,*) 'Continuing an existing run'
    if(IRUN == 2) {
        cout<< "Enter 1 to confirm, 0 to continue current run :-"<<endl;
        cin>> ICONF;
        if(ICONF == 1)
            file_9<<"Starting a new run"<<endl;//WRITE (9,*) 'Starting a new run'
        else{
            IRUN=1;
            file_9<<"Continuing an existing run"<<endl;
            //WRITE (9,*) 'Continuing an existing run'
        }
    }
    //
    if(IRUN == 2){          //new run
        cout<< "Enter 0 for a homogeneous gas, or"<<endl;
        cout<< "Enter 1 for a one-dimensional flow, or"<<endl;
        cout<< "Enter 2 for a two-dimensional plane flow, or"<<endl;
        cout<< "Enter 3 for a three dimensional flow, or"<<endl;
        cout<< "enter 4 for an axially-symmetric flow :-"<<endl;
        cin>> calc->ICLASS;
        calc->NCLASS=2;      //default 2D
        if(calc->ICLASS < 2) calc->NCLASS=1;   //0D or 1D
        if(calc->ICLASS == 3) calc->NCLASS=3;  //3D
        cout<<"Enter 0 for an eventually steady flow, or"<<endl;
        cout<<"enter 1 for a continuing unsteady flow :-"<<endl;
        cin>> calc->ISF;
        
        
        file_7.open("RUN_CLASS.TXT", ios::trunc |ios::out);
        if(file_7.is_open()){
            cout<<"RUN_CLASS.TXT is opened"<<endl;
        }
        else{
            cout<<"RUN_CLASS.TXT not opened"<<endl;
            cin.get();
        }
        file_7<<calc->ICLASS<<calc->ISF;
        file_7.close();
        //        OPEN (7,FILE='RUN_CLASS.TXT',FORM='FORMATTED',STATUS='REPLACE')
        //        WRITE (7,*) ICLASS,ISF
        //        CLOSE (7)
        file_9<<"Starting a new run with ICLASS, ISF "<<calc->ICLASS<<" "<<calc->ISF<<endl;
        //        WRITE (9,*) 'Starting a new run with ICLASS, ISF',ICLASS,ISF
        cout<<"Starting a new run with ICLASS, ISF "<<calc->ICLASS<<" "<<calc->ISF<<endl;
    }
    //
    if(IRUN == 1){       //continued run
        file_7.open("RUN_CLASS.TXT" , ios::in );
        if(file_7.is_open()){
            cout<<"RUN_CLASS.TXT is opened"<<endl;
        }
        else{
            cout<<"RUN_CLASS.TXT not opened"<<endl;
            cin.get();
        }
        file_7 >>calc->ICLASS>>calc->ISF;
        file_7.close();
        //        OPEN (7,FILE='RUN_CLASS.TXT',FORM='FORMATTED',STATUS='OLD')
        //        READ (7,*) ICLASS,ISF
        //        CLOSE(7)
        READ_RESTART();
        //
        calc->TSAMP=calc->FTIME+calc->DTSAMP;
        calc->TOUT=calc->FTIME+calc->DTOUT;
        if((gas->MEX > 0) && (calc->ISF == 1)){
            cout<<"Enter 0 to continue the reaction sample or"<<endl;
            cout<<"enter 1 to continue with a new reaction sample :-"<<endl;
            cin>> N;
            if(N == 1){
                //memset(gas->TNEX,0.e00,sizeof(*gas->TNEX));
                //memset(calc->TDISS,0.e00,sizeof(*calc->TDISS));
                //memset(calc->TRECOMB,0.e00,sizeof(*calc->TRECOMB));
                for(int i=0;i<gas->MEX+1;i++)
                    gas->TNEX[i]= 0.e00;
                for(int i=0;i<gas->MSP+1;i++)
                    calc->TDISS[i]=0.e00;
                for(int i=0;i<gas->MSP+1;i++)
                    calc->TRECOMB[i]=0.e00;
            }
        }
        //
        if((calc->ISAD == 0) && (calc->ISF == 0)){
            cout<<"Enter 0 to continue the current sample or"<<endl;
            cout<<"enter 1 to continue with a new sample :-"<<endl;
            cin>> N;
            if(N == 1){
                if((geom->ITYPE[2] == 4) && (calc->ICN == 0)){
                    cout<<"Enter 0 to continue to not enforce constant molecule number"<<endl;
                    cout<<"enter 1 to start to enforce constant molecule number :-"<<endl;
                    cin>> M;
                    if(M == 1) calc->ICN=1;
                }
                cout<<"Enter 1 to adapt the cells, or 0 to continue with current cells:-"<<endl;
                cin>>IADAPT;
                if(IADAPT == 1){
                    cout<<"Adapting cells"<<endl;
                    ADAPT_CELLS_1D() ;
                    INDEX_MOLS();
                    WRITE_RESTART();
                }
                else
                    cout<<"Continuing with existing cells"<<endl;
                //
                if(calc->IREM == 2){
                    cout<<"Enter 1 to reset the removal details, or 0 to continue with current details:-"<<endl;
                    cin>>IRETREM;
                    if(IRETREM == 1){
                        geom->FREM=-1.e00;
                        while((geom->FREM < -0.0001) || (geom->FREM > 5.0)){
                            cout<<"Enter the fraction of entering molecules that are removed:-"<<endl;
                            cin>>geom->FREM;
                            cout<<"The ratio of removed to entering mlecules is \t"<<geom->FREM<<endl;
                            //                            WRITE (*,999) FREM
                        }
                        file_9<<"The ratio of removed to entering mlecules is \t"<<geom->FREM<<endl;
                        //                        WRITE (9,999) FREM
                        //                        999       FORMAT (' The ratio of removed to entering molecules is ',G15.5)
                        if(geom->FREM > 1.e-10){
                            geom->XREM=geom->XB[1]-1.0;
                            while((geom->XREM < geom->XB[1]-0.0001) || (geom->XREM > geom->XB[2]+0.0001)){
                                cout<<"Enter x coordinate of the upstream removal limit:-"<<endl;
                                cin>>geom->XREM;
                                cout<<"The molecules are removed from \t"<<geom->XREM<<" to "<<geom->XB[2]<<endl; //988
                                //                                WRITE (*,998) XREM,XB(2)
                            }
                            file_9<<"The molecules are removed from \t"<<geom->XREM<<" to "<<geom->XB[2]<<endl;
                            //                            WRITE (9,998) XREM,XB(2)
                            //                            998         FORMAT (' The molecules are removed from ',G15.5,' to',G15.5)
                        }
                    }
                }
                //
                INITIALISE_SAMPLES();
            }
        }
    }
    //
    if(IRUN == 2){
        //
        READ_DATA();
        //
        if(calc->ICLASS < 2) SET_INITIAL_STATE_1D();
        //
        if(calc->ICLASS == 0) ENERGY(0,A);
        //
        WRITE_RESTART();
        //
    }
    //
    while(calc->FTIME < calc->TLIM){
        //
        //
        calc->FTIME=calc->FTIME+calc->DTM;
        //
        file_9<<"  TIME  "<<setw(20)<<setprecision(10)<<calc->FTIME<<"  NM  "<<molecs->NM<<"  COLLS  "<<std::left<<setw(20)<<setprecision(10)<<calc->TOTCOL<<"Collision_time : "<<colltime<<endl;
        //        WRITE (9,*) 'TIME',FTIME,' NM',NM,' COLLS',TOTCOL
        cout<< "  TIME   "<<setw(20)<<setprecision(10)<<calc->FTIME<<"  NM  "<<molecs->NM<<"  COLLS  "<<std::left<<setw(20)<<setprecision(10)<<calc->TOTCOL<<"Collision_time : "<<colltime<<endl;
        //
        //  WRITE (13,*) FTIME/TNORM,FLOAT(NM)/FLOAT(NMI)      //uncomment if a MOLFILE.DAT is to be generated
        //
        //  WRITE (*,*) 'MOVE'
        //cout<<"MOVE"<<endl;
        MOLECULES_MOVE_1D();
        //
        if((geom->ITYPE[1] == 0) || (geom->ITYPE[2] == 0) || (geom->ITYPE[2] == 4)) MOLECULES_ENTER_1D();
        //
        //  WRITE (*,*) 'INDEX'
        //ut<<"INDEX"<<endl;
        // cout<<calc->TOUT<<endl;
        // cin.get();
        INDEX_MOLS();
        //
         // WRITE (*,*) 'COLLISIONS'
        COLLISIONS();
        //
        // if(gas->MMVM > 0) {
        //     cout<<"DISSOCIATION"<<endl;
        //     DISSOCIATION();
        // }
        //
        if(calc->FTIME > calc->TSAMP){
            //    WRITE (*,*) 'SAMPLE'
            if(calc->ISF == 0) SAMPLE_FLOW();
            if((calc->ISF == 1) && (calc->FTIME < calc->TPOUT+(1.e00-calc->FRACSAM)*calc->DTOUT)){
                calc->TSAMP=calc->TSAMP+calc->DTSAMP;
                INITIALISE_SAMPLES();
            }
            if((calc->ISF == 1) && (calc->FTIME >= calc->TPOUT+(1.e00-calc->FRACSAM)*calc->DTOUT)) SAMPLE_FLOW();
        }
        //
        if(calc->FTIME > calc->TOUT){
            cout<<"writing OUTPUT"<<endl;
            //    WRITE (*,*) 'OUTPUT'
            WRITE_RESTART();
            //
            OUTPUT_RESULTS();
            calc->TPOUT=calc->FTIME;
        }
        //
    }
    return 0;
    //
}

template <typename T>
string to_string(T value)
{
    std::ostringstream os ;
    os << value ;
    return os.str() ;
}

void ALLOCATE_GAS()
{
    // //GAS gas;
    // //CALC calc;
    gas->d_allocate(gas->MSP+1,3,gas->FSP);
    gas->d_allocate(7,gas->MSP+1,gas->SP);
    gas->d_allocate(4,gas->MSP+1,gas->SPR);
    gas->d_allocate(9,gas->MSP+1,gas->MSP,gas->SPM);
    gas->i_allocate(3,gas->MSP+1,gas->ISPR);
    gas->i_allocate(gas->MSP+1,gas->ISPV);
    gas->d_allocate(7,gas->MSP+1,3,gas->ENTR);
    gas->d_allocate(gas->MSP+1,3,gas->VMP);
    calc->d_allocate(gas->MSP+1,calc->VNMAX);
    gas->d_allocate(gas->MSP+1,gas->CR);
    calc->d_allocate(gas->MSP+1,gas->MSP+1,calc->TCOL);
    gas->i_allocate(gas->MSP+1,gas->MSP+1,gas->ISPRC);
    gas->i_allocate(gas->MSP+1,gas->MSP+1,gas->ISPRK);
    gas->d_allocate(5,gas->MSP+1,gas->MSP+1,gas->MSP+1,gas->SPRC);
    gas->i_allocate(gas->MSP+1,gas->NELL);
    gas->d_allocate(4,gas->MELE+1,gas->MSP+1,gas->QELC);
    gas->d_allocate(3,gas->MSP+1,gas->MSP+1,gas->MVIBL+1,gas->SPRP);
    gas->d_allocate(3,gas->MSP+1,gas->MSP+1,gas->SPRT);
    calc->d_allocate(gas->MSP+1,calc->AJM);
    gas->d_allocate(gas->MSP+1,gas->FP);
    calc->d_allocate(gas->MSP+1,calc->ALOSS);
    calc->d_allocate(gas->MSP+1,calc->EME);
    
    /*ALLOCATE (FSP(MSP,2),SP(6,MSP),SPR(3,MSP),SPM(8,MSP,MSP),ISPR(2,MSP),ISPV(MSP),ENTR(6,MSP,2),      &
     VMP(MSP,2),VNMAX(MSP),CR(MSP),TCOL(MSP,MSP),ISPRC(MSP,MSP),ISPRK(MSP,MSP),SPRC(4,MSP,MSP,MSP),                        &
     NELL(MSP),QELC(3,MELE,MSP),SPRP(2,MSP,MSP,0:MVIBL),SPRT(2,MSP,MSP),AJM(MSP),FP(MSP),    &
     ALOSS(MSP),EME(MSP),STAT=ERROR)
     //
     IF (ERROR /= 0) THEN
     WRITE (*,*)'PROGRAM COULD NOT ALLOCATE SPECIES VARIABLES',ERROR
     END IF
     //*/
    gas->i_allocate(gas->MMEX+1,gas->MSP+1,gas->MSP+1,gas->NEX);
    gas->i_allocate(gas->MSP+1,gas->MSP+1,gas->NSPEX);
    gas->d_allocate(7,gas->MMEX+1,gas->MSP+1,gas->MSP+1,gas->SPEX);
    gas->i_allocate(gas->MMEX+1,8,gas->MSP+1,gas->MSP+1,gas->ISPEX);
    gas->i_allocate(5,gas->MSP+1,gas->TREACG);
    gas->d_allocate(gas->MMEX+1,gas->PSF);
    gas->i_allocate(5,gas->MSP+1,gas->TREACL);
    gas->d_allocate(gas->MEX+1,gas->TNEX);
    gas->d_allocate(3,gas->MMEX+1,gas->MSP+1,gas->MSP+1,gas->MVIBL+1,gas->SPREX);
    gas->i_allocate(3,gas->MSP+1,gas->NSLEV);
    gas->d_allocate(gas->MSP+1,gas->SLER);
    // ALLOCATE (NEX(MMEX,MSP,MSP),NSPEX(MSP,MSP),SPEX(6,MMEX,MSP,MSP),ISPEX(MMEX,7,MSP,MSP),TREACG(4,MSP),         &
    //           PSF(MMEX),TREACL(4,MSP),TNEX(MEX),SPREX(2,MMEX,MSP,MSP,0:MVIBL),NSLEV(2,MSP),SLER(MSP),STAT=ERROR)
    // //
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE Q-K REACTION VARIABLES',ERROR
    // END IF
    // //
    if(gas->MMVM >= 0){
        gas->d_allocate(6,gas->MMVM+1,gas->MSP+1,gas->SPVM);
        gas->i_allocate(3,gas->MMVM+1,gas->MSP+1,gas->ISPVM);
        calc->d_allocate(gas->MSP+1,calc->TDISS);
        calc->d_allocate(gas->MSP+1,calc->TRECOMB);
        //ALLOCATE (SPVM(5,MMVM,MSP),ISPVM(2,MMVM,MSP),TDISS(MSP),TRECOMB(MSP),STAT=ERROR)
        // IF (ERROR /= 0) THEN
        //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE VIBRATION VARIABLES',ERROR
    }
    
    //N.B. surface reactions are not yet implemented
    if(gas->MNSR > 0){
        gas->d_allocate(gas->MNSR+1,gas->ERS);
        gas->i_allocate(3,gas->MNSR+1,gas->LIS);
        gas->i_allocate(7,gas->MNSR+1,gas->LRS);
        gas->i_allocate(gas->MNSR+1,gas->MSP+1,gas->ISRCD);
        //ALLOCATE (ERS(MNSR),LIS(2,MNSR),LRS(6,MNSR),ISRCD(MNSR,MSP),STAT=ERROR)
        // IF (ERROR /= 0) THEN
        //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE SURFACE REACTION VARIABLES',ERROR
    }
     //calc->AJM=0.e00;
    //memset(calc->AJM,0.e00,sizeof(*calc->AJM));
    for(int i=0;i<gas->MSP+1;i++){
        calc->AJM[i]=0.e00;
    }
    return;
    
}

void HARD_SPHERE()
{
    ////GAS gas;
    ////CALC calc;
    cout<<"Reading HARD_SPHERE Data"<<endl;
    gas->MSP=1;
    gas->MMRM=0;
    gas->MMVM=0;
    gas->MNSR=0;
    gas->MEX=0;
    gas->MMEX=0;
    gas->MELE=1;
    gas->MVIBL=0;
    
    ALLOCATE_GAS();
    
    gas->SP[1][1]=4.0e-10;    //reference diameter
    gas->SP[2][1]=273.0;       //reference temperature
    gas->SP[3][1]=0.5;        //viscosity-temperature index
    gas->SP[4][1]=1.0;         //reciprocal of VSS scattering parameter (1 for VHS)
    gas->SP[5][1]=5.e-26;     //mass
    gas->ISPR[1][1]=0;        //number of rotational degrees of freedom
    cout<<"Hard Sphere data done"<<endl;
    return;
}


void ARGON()
{
    // //GAS gas;
    // //CALC calc;
    cout<<"Reading Argon Data"<<endl;
    gas->MSP=1;
    gas->MMRM=0;
    gas->MMVM=0;
    gas->MNSR=0;
    gas->MEX=0;
    gas->MMEX=0;
    gas->MELE=1;
    gas->MVIBL=0;
    ALLOCATE_GAS();
    gas->SP[1][1]=4.17e-10;
    gas->SP[2][1]=273.15;
    gas->SP[3][1]=0.81;
    gas->SP[4][1]=1.0;
    gas->SP[5][1]=6.63e-26;
    gas->ISPR[1][1]=0;
    gas->ISPR[2][1]=0;
    cout<<"Argon Data done"<<endl;
    return;
}
//
void IDEAL_NITROGEN()
{
    // //GAS gas;
    // //CALC calc;
    cout<<"Reading IDEAL_NITROGEN data"<<endl;
    gas->MSP=1;
    gas->MMRM=1;
    gas->MMVM=0;
    gas->MNSR=0;
    gas->MEX=0;
    gas->MMEX=0;
    gas->MELE=0;
    gas->MVIBL=0;
    
    ALLOCATE_GAS();
    gas->SP[1][1]=4.17e-10;
    gas->SP[2][1]=273.0;
    gas->SP[3][1]=0.74;
    gas->SP[4][1]=1.0;
    gas->SP[5][1]=4.65e-26;
    gas->ISPR[1][1]=2;
    gas->ISPR[2][1]=0;
    gas->SPR[1][1]=5.0;
    return;
}
//
void REAL_OXYGEN()
{
    //
    //GAS gas;
    //CALC calc;
    cout<<"Reading Real_Oxygen data"<<endl;
    gas->MSP=2;
    gas->MMRM=1;
    gas->MMVM=1;
    gas->MNSR=0;
    gas->MEX=0;
    gas->MMEX=0;
    gas->MELE=5;
    gas->MVIBL=26;
    ALLOCATE_GAS();
    gas->SP[1][1]=4.07e-10;
    gas->SP[2][1]=273.00;
    gas->SP[3][1]=0.77e00;
    gas->SP[4][1]=1.e00;
    gas->SP[5][1]=5.312e-26;
    gas->SP[6][1]=0.e00;
    gas->ISPR[1][1]=2;
    gas->ISPR[2][1]=0 ;            //0,1 for constant,polynomial rotational relaxation collision number
    gas->SPR[1][1]=5.0;             // the collision number or the coefficient of temperature in the polynomial (if a polynomial, the coeff. of T^2 is in spr_db(3  )
    
    gas->ISPV[1]=1   ;            // the number of vibrational modes
    gas->SPVM[1][1][1]=2256.e00  ;        // the characteristic vibrational temperature
    gas->SPVM[2][1][1]=90000.e00;        // a constant Zv, or the reference Zv
    gas->SPVM[3][1][1]=2256.e00;        // -1 for a constant Zv, or the reference temperature
    gas->SPVM[5][1][1]=1.0;            //arbitrary reduction factor
    gas->ISPVM[1][1][1]=2;
    gas->ISPVM[2][1][1]=2;
    gas->NELL[1]=3;
    if(gas->MELE > 1){
        //******
        gas->QELC[1][1][1]=3.0;
        gas->QELC[2][1][1]=0.0;
        gas->QELC[3][1][1]=50.0;  //500.
        gas->QELC[1][2][1]=2.0;
        gas->QELC[2][2][1]=11393.0;
        gas->QELC[3][2][1]=50.0;  //500         //for equipartition, the cross-section ratios must be the same for all levels
        gas->QELC[1][3][1]=1.0;
        gas->QELC[2][3][1]=18985.0;
        gas->QELC[3][3][1]=50.0;  //500.
    }
    //
    //species 2 is atomic oxygen
    gas->SP[1][2]=3.e-10;
    gas->SP[2][2]=273.e00;
    gas->SP[3][2]=0.8e00;
    gas->SP[4][2]=1.e00;
    gas->SP[5][2]=2.656e-26;
    gas->SP[6][2]=4.099e-19;
    gas->ISPR[1][2]=0;
    gas->ISPV[2]=0;     //must be set//
    //set electronic information
    if(gas->MELE > 1){
        gas->NELL[2]=5;
        gas->QELC[1][1][2]=5.0;
        gas->QELC[2][1][2]=0.0;
        gas->QELC[3][1][2]=50.0;
        gas->QELC[1][2][2]=3.0;
        gas->QELC[2][2][2]=228.9;
        gas->QELC[3][2][2]=50.0;
        gas->QELC[1][3][2]=1.0;
        gas->QELC[2][3][2]=325.9;
        gas->QELC[3][3][2]=50.0;
        gas->QELC[1][4][2]=5.0;
        gas->QELC[2][4][2]=22830.0;
        gas->QELC[3][4][2]=50.0;
        gas->QELC[1][5][2]=1.0;
        gas->QELC[2][5][2]=48621.0;
        gas->QELC[3][5][2]=50.0;
    }
    //set data needed for recombination
    //
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->ISPRC[i][j]=0;
            gas->ISPRK[i][j]=0;
        }
    }
    // gas->ISPRC=0;
    // gas->ISPRK=0;
    gas->ISPRC[2][2]=1;    //O+O -> O2  recombined species code for an O+O recombination
    gas->ISPRK[2][2]=1 ;     //the relevant vibrational mode of this species
    gas->SPRC[1][2][2][1]=0.04;
    gas->SPRC[2][2][2][1]=-1.3;
    gas->SPRC[1][2][2][2]=0.05;
    gas->SPRC[2][2][2][2]=-1.1;
    gas->SPRT[1][2][2]=5000.e00;
    gas->SPRT[2][2][2]=15000.e00;
    //
    //memset(gas->NSPEX,0,sizeof(**gas->NSPEX));
    //memset(gas->SPEX,0.e00,sizeof(****gas->SPEX));
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->NSPEX[i][j]=0;
        }
    }
    for(int i=0;i<7;i++){
        for(int j=0;j<gas->MMEX+1;j++){
            for(int k=0;k<gas->MSP+1;k++){
                for(int l=0;l<gas->MSP+1;l++)
                    gas->SPEX[i][j][k][l]=0.e00;
            }
        }
    }
    //gas->SPEX=0.e00;
    gas->ISPEX=0;
    //
    DERIVED_GAS_DATA();
    //
    cout<<"Real_Oxygen data done"<<endl;
    return;
}
//
void IDEAL_AIR()
{
    //GAS gas;
    //CALC calc;
    cout<<"Reading IDEAL_AIR data"<<endl;
    gas->MSP=2;
    gas->MMRM=1;
    gas->MMVM=0;
    gas->MNSR=0;
    gas->MEX=0;
    gas->MMEX=0;
    gas->MELE=1;
    gas->MVIBL=0;
    //
    ALLOCATE_GAS();
    //
    gas->SP[1][1]=4.07e-10;
    gas->SP[2][1]=273.0;
    gas->SP[3][1]=0.77;
    gas->SP[4][1]=1.0;
    gas->SP[5][1]=5.312e-26;
    gas->ISPR[1][1]=2;
    gas->ISPR[2][1]=0;
    gas->SPR[1][1]=5.0;
    gas->SP[1][2]=4.17e-10;
    gas->SP[2][2]=273.0;
    gas->SP[3][2]=0.74;
    gas->SP[4][2]=1.0;
    gas->SP[5][2]=4.65e-26;
    gas->ISPR[1][2]=2;
    gas->ISPR[2][2]=0;
    gas->SPR[1][2]=5.0;
    cout<<"IDEAL_AIR data done"<<endl;
    return;
}
//
void REAL_AIR()
{
    //GAS gas;
    //CALC calc;
    cout<<"REAL_AIR data done"<<endl;
    gas->MSP=5;
    gas->MMRM=1;
    gas->MMVM=1;
    gas->MELE=5;
    gas->MVIBL=40;  //?
    //
    gas->MEX=4;
    gas->MMEX=1;
    //
    gas->MNSR=0;
    ALLOCATE_GAS();
    //species 1 is oxygen
    gas->SP[1][1]=4.07e-10;
    gas->SP[2][1]=273.e00;
    gas->SP[3][1]=0.77e00;
    gas->SP[4][1]=1.e00;
    gas->SP[5][1]=5.312e-26;
    gas->SP[6][1]=0.e00;
    gas->ISPR[1][1]=2;
    gas->ISPR[2][1]=0;
    gas->SPR[1][1]=5.e00;
    gas->ISPV[1]=1;               // the number of vibrational modes
    gas->SPVM[1][1][1]=2256.e00;          // the characteristic vibrational temperature
    gas->SPVM[2][1][1]=18000.e00;  //90000.D00        // a constant Zv, or the reference Zv
    gas->SPVM[3][1][1]=2256.e00;       // -1 for a constant Zv, or the reference temperature
    gas->SPVM[5][1][1]=1.0;
    gas->ISPVM[1][1][1]=3;
    gas->ISPVM[2][1][1]=3;
    gas->NELL[1]=3;
    gas->QELC[1][1][1]=3.0;
    gas->QELC[2][1][1]=0.0;
    gas->QELC[3][1][1]=50.0;
    gas->QELC[1][2][1]=2.0;
    gas->QELC[2][2][1]=11393.0;
    gas->QELC[3][2][1]=50.0;
    gas->QELC[1][3][1]=1.0;
    gas->QELC[2][3][1]=18985.0;
    gas->QELC[3][3][1]=50.0;
    //species 2 is nitrogen
    gas->SP[1][2]=4.17e-10;
    gas->SP[2][2]=273.e00;
    gas->SP[3][2]=0.74e00;
    gas->SP[4][2]=1.e00;
    gas->SP[5][2]=4.65e-26;
    gas->SP[6][2]=0.e00;
    gas->ISPR[1][2]=2;
    gas->ISPR[2][2]=0;
    gas->SPR[1][2]=5.e00;
    gas->ISPV[2]=1;
    gas->SPVM[1][1][2]=3371.e00;
    gas->SPVM[2][1][2]=52000.e00;     //260000.D00
    gas->SPVM[3][1][2]=3371.e00;
    gas->SPVM[5][1][2]=0.3;
    gas->ISPVM[1][1][2]=4;
    gas->ISPVM[2][1][2]=4;
    gas->NELL[2]=1;
    gas->QELC[1][1][2]=1.0;
    gas->QELC[2][1][2]=0.0;
    gas->QELC[3][1][2]=100.0;
    //species 3 is atomic oxygen
    gas->SP[1][3]=3.e-10;
    gas->SP[2][3]=273.e00;
    gas->SP[3][3]=0.8e00;
    gas->SP[4][3]=1.e00;
    gas->SP[5][3]=2.656e-26;
    gas->SP[6][3]=4.099e-19;
    gas->ISPR[1][3]=0;
    gas->ISPV[3]=0;
    gas->NELL[3]=5;
    gas->QELC[1][1][3]=5.0;
    gas->QELC[2][1][3]=0.0;
    gas->QELC[3][1][3]=50.0;
    gas->QELC[1][2][3]=3.0;
    gas->QELC[2][2][3]=228.9;
    gas->QELC[3][2][3]=50.0;
    gas->QELC[1][3][3]=1.0;
    gas->QELC[2][3][3]=325.9;
    gas->QELC[3][3][3]=50.0;
    gas->QELC[1][4][3]=5.0;
    gas->QELC[2][4][3]=22830.0;
    gas->QELC[3][4][3]=50.0;
    gas->QELC[1][5][3]=1.0;
    gas->QELC[2][5][3]=48621.0;
    gas->QELC[3][5][3]=50.0;
    //species 4 is atomic nitrogen
    gas->SP[1][4]=3.e-10;
    gas->SP[2][4]=273.e00;
    gas->SP[3][4]=0.8e00;
    gas->SP[4][4]=1.0e00;
    gas->SP[5][4]=2.325e-26;
    gas->SP[6][4]=7.849e-19;
    gas->ISPR[1][4]=0;
    gas->ISPV[4]=0;
    gas->NELL[4]=3;
    gas->QELC[1][1][4]=4.0;
    gas->QELC[2][1][4]=0.0;
    gas->QELC[3][1][4]=50.0;
    gas->QELC[1][2][4]=10.0;
    gas->QELC[2][2][4]=27658.0;
    gas->QELC[3][2][4]=50.0;
    gas->QELC[1][3][4]=6.0;
    gas->QELC[2][3][4]=41495.0;
    gas->QELC[3][3][4]=50.0;
    //species 5 is NO
    gas->SP[1][5]=4.2e-10;
    gas->SP[2][5]=273.e00;
    gas->SP[3][5]=0.79e00;
    gas->SP[4][5]=1.0e00;
    gas->SP[5][5]=4.98e-26;
    gas->SP[6][5]=1.512e-19;
    gas->ISPR[1][5]=2;
    gas->ISPR[2][5]=0;
    gas->SPR[1][5]=5.e00;
    gas->ISPV[5]=1;
    gas->SPVM[1][1][5]=2719.e00;
    gas->SPVM[2][1][5]=14000.e00;   //70000.D00
    gas->SPVM[3][1][5]=2719.e00;
    gas->SPVM[5][1][5]=0.2;
    gas->ISPVM[1][1][5]=3;
    gas->ISPVM[2][1][5]=4;
    gas->NELL[5]=2;
    gas->QELC[1][1][5]=2.0;
    gas->QELC[2][1][5]=0.0;
    gas->QELC[3][1][5]=50.0;
    gas->QELC[1][2][5]=2.0;
    gas->QELC[2][2][5]=174.2;
    gas->QELC[3][2][5]=50.0;
    //set the recombination data for the molecule pairs
    //memset(gas->ISPRC,0,sizeof(**gas->ISPRC));//gas->ISPRC=0;    //data os zero unless explicitly set
    //memset(gas->ISPRK,0,sizeof(**gas->ISPRK));//gas->ISPRK=0;
    //memset(gas->SPRC,0,sizeof(****gas->SPRC));//gas->SPRC=0.e00;
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->ISPRC[i][j]=0;
        }
    }
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->ISPRK[i][j]=0;
        }
    }
    for(int i=0;i<5;i++){
        for(int j=0;j<gas->MSP+1;j++){
            for(int k=0;k<gas->MSP+1;k++){
                for(int l=0;l<gas->MSP+1;l++)
                    gas->SPEX[i][j][k][l]=0.e00;
            }
        }
    }
    gas->ISPRC[3][3]=1; //O+O -> O2  recombined species code for an O+O recombination
    gas->ISPRK[3][3]=1;
    gas->SPRC[1][3][3][1]=0.04e00;
    gas->SPRC[2][3][3][1]=-1.3e00;
    gas->SPRC[1][3][3][2]=0.07e00;
    gas->SPRC[2][3][3][2]=-1.2e00;
    gas->SPRC[1][3][3][3]=0.08e00;
    gas->SPRC[2][3][3][3]=-1.2e00;
    gas->SPRC[1][3][3][4]=0.09e00;
    gas->SPRC[2][3][3][4]=-1.2e00;
    gas->SPRC[1][3][3][5]=0.065e00;
    gas->SPRC[2][3][3][5]=-1.2e00;
    gas->SPRT[1][3][3]=5000.e00;
    gas->SPRT[2][3][3]=15000.e00;
    gas->ISPRC[4][4]=2;  //N+N -> N2
    gas->ISPRK[4][4]=1;
    gas->SPRC[1][4][4][1]=0.15e00;
    gas->SPRC[2][4][4][1]=-2.05e00;
    gas->SPRC[1][4][4][2]=0.09e00;
    gas->SPRC[2][4][4][2]=-2.1e00;
    gas->SPRC[1][4][4][3]=0.16e00;
    gas->SPRC[2][4][4][3]=-2.0e00;
    gas->SPRC[1][4][4][4]=0.17e00;
    gas->SPRC[2][4][4][4]=-2.0e00;
    gas->SPRC[1][4][4][5]=0.17e00;
    gas->SPRC[2][4][4][5]=-2.1e00;
    gas->SPRT[1][4][4]=5000.e00;
    gas->SPRT[2][4][4]=15000.e00;
    gas->ISPRC[3][4]=5;
    gas->ISPRK[3][4]=1;
    gas->SPRC[1][3][4][1]=0.3e00;
    gas->SPRC[2][3][4][1]=-1.9e00;
    gas->SPRC[1][3][4][2]=0.4e00;
    gas->SPRC[2][3][4][2]=-2.0e00;
    gas->SPRC[1][3][4][3]=0.3e00;
    gas->SPRC[2][3][4][3]=-1.75e00;
    gas->SPRC[1][3][4][4]=0.3e00;
    gas->SPRC[2][3][4][4]=-1.75e00;
    gas->SPRC[1][3][4][5]=0.15e00;
    gas->SPRC[2][3][4][5]=-1.9e00;
    gas->SPRT[1][3][4]=5000.e00;
    gas->SPRT[2][3][4]=15000.e00;
    //set the exchange reaction data
    //memset(gas->SPEX,0,sizeof(****gas->SPEX));//gas->SPEX=0.e00;
    for(int i=0;i<7;i++){
        for(int j=0;j<gas->MMEX+1;j++){
            for(int k=0;k<gas->MSP+1;k++){
                for(int l=0;l<gas->MSP+1;l++)
                    gas->SPEX[i][j][k][l]=0.e00;
            }
        }
    }
    gas->ISPEX=0;
    gas->NSPEX=0;
    gas->NSPEX[2][3]=1;
    gas->NSPEX[4][5]=1;
    gas->NSPEX[3][5]=1;
    gas->NSPEX[1][4]=1;
    //N2+O->NO+N
    gas->ISPEX[1][1][2][3]=2;
    gas->ISPEX[1][2][2][3]=3;
    gas->ISPEX[1][3][2][3]=5;
    gas->ISPEX[1][4][2][3]=4;
    gas->ISPEX[1][5][2][3]=1;
    gas->ISPEX[1][6][2][3]=1;
    gas->SPEX[6][1][2][3]=0.e00;
    gas->NEX[1][2][3]=1;
    //NO+N->N2+0
    gas->ISPEX[1][1][4][5]=5;
    gas->ISPEX[1][2][4][5]=4;
    gas->ISPEX[1][3][4][5]=2;
    gas->ISPEX[1][4][4][5]=3;
    gas->ISPEX[1][5][4][5]=1;
    gas->ISPEX[1][6][4][5]=1;
    gas->ISPEX[1][7][4][5]=1;
    gas->SPEX[1][1][4][5]=0.8e00;
    gas->SPEX[2][1][4][5]=-0.75e00;
    gas->SPEX[4][1][4][5]=5000.e00;
    gas->SPEX[5][1][4][5]=15000.e00;
    gas->SPEX[6][1][4][5]=0.e00;
    gas->NEX[1][4][5]=2;
    //NO+O->O2+N
    gas->ISPEX[1][1][3][5]=5;
    gas->ISPEX[1][2][3][5]=3;
    gas->ISPEX[1][3][3][5]=1;
    gas->ISPEX[1][4][3][5]=4;
    gas->ISPEX[1][5][3][5]=1;
    gas->ISPEX[1][6][3][5]=1;
    gas->SPEX[6][1][3][5]=2.e-19;
    gas->NEX[1][3][5]=3;
    //O2+N->NO+O
    gas->ISPEX[1][1][1][4]=1;
    gas->ISPEX[1][2][1][4]=4;
    gas->ISPEX[1][3][1][4]=5;
    gas->ISPEX[1][4][1][4]=3;
    gas->ISPEX[1][5][1][4]=1;
    gas->ISPEX[1][6][1][4]=1;
    gas->ISPEX[1][7][1][4]=1 ;
    gas->SPEX[1][1][1][4]=7.e00;
    gas->SPEX[2][1][1][4]=-0.85e00;
    gas->SPEX[4][1][1][4]=5000.e00;
    gas->SPEX[5][1][1][4]=15000.e00;
    gas->SPEX[6][1][1][4]=0.e00;
    gas->NEX[1][1][4]=4;
    
    DERIVED_GAS_DATA();
    cout<<"REAL_AIR data done"<<endl;
    return;
}
//
void HELIUM_ARGON_XENON()
{
    //GAS gas;
    //CALC calc;
    cout<<"Reading HELIUM_ARGON_XENON data"<<endl;
    gas->MSP=3;
    gas->MMRM=0;
    gas->MMVM=0;
    gas->MNSR=0;
    gas->MEX=0;
    gas->MMEX=0;
    gas->MELE=1;
    gas->MVIBL=0;
    
    ALLOCATE_GAS();
    
    gas->SP[1][1]=2.30e-10;   //2.33D-10
    gas->SP[2][1]=273.0;
    gas->SP[3][1]=0.66;
    gas->SP[4][1]=0.794;   //1.
    gas->SP[5][1]=6.65e-27;
    gas->ISPR[1][1]=0;
    gas->ISPR[2][1]=0;
    //
    gas->SP[1][2]=4.11e-10;   //4.17D-10
    gas->SP[2][2]=273.15;
    gas->SP[3][2]=0.81;
    gas->SP[4][2]=0.714;    //1.
    gas->SP[5][2]=6.63e-26;
    gas->ISPR[1][2]=0;
    gas->ISPR[2][2]=0;
    //
    gas->SP[1][3]=5.65e-10;   //5.74D-10
    gas->SP[2][3]=273.0;
    gas->SP[3][3]=0.85;
    gas->SP[4][3]=0.694;   //1.
    gas->SP[5][3]=21.8e-26;
    gas->ISPR[1][3]=0;
    gas->ISPR[2][3]=0;
    cout<<"HELIUM_ARGON_XENON data done"<<endl;
    return;
}
//
void OXYGEN_HYDROGEN()
{
    //
    //GAS gas;
    //CALC calc;
    cout<<"Reading OXYGEN_HYDROGEN data"<<endl;
    gas->MSP=8;
    gas->MMRM=3;
    gas->MMVM=3;
    gas->MELE=1;
    gas->MVIBL=40;  //the maximum number of vibrational levels before a cumulative level reaches 1
    //
    gas->MEX=16;
    gas->MMEX=3;
    //
    gas->MNSR=0;
    //
    ALLOCATE_GAS();
    //
    //species 1 is hydrogen H2
    gas->SP[1][1]=2.92e-10;
    gas->SP[2][1]=273.e00;
    gas->SP[3][1]=0.67e00;
    gas->SP[4][1]=1.e00;
    gas->SP[5][1]=3.34e-27;
    gas->SP[6][1]=0.e00;
    gas->ISPR[1][1]=2;
    gas->ISPR[2][1]=0;
    gas->SPR[1][1]=5.e00;
    gas->ISPV[1]=1;         // the number of vibrational modes
    gas->SPVM[1][1][1]=6159.e00;          // the characteristic vibrational temperature
    gas->SPVM[2][1][1]=20000.e00;  //estimate
    gas->SPVM[3][1][1]=2000.e00; //estimate
    gas->SPVM[5][1][1]=1.0;
    gas->ISPVM[1][1][1]=2;
    gas->ISPVM[2][1][1]=2;
    //species 2 is atomic hydrogen H
    gas->SP[1][2]=2.5e-10;      //estimate
    gas->SP[2][2]=273.e00;
    gas->SP[3][2]=0.8e00;
    gas->SP[4][2]=1.e00;
    gas->SP[5][2]=1.67e-27;
    gas->SP[6][2]=3.62e-19;
    gas->ISPR[1][2]=0;
    gas->ISPV[2]=0;
    //species 3 is oxygen O2
    gas->SP[1][3]=4.07e-10;
    gas->SP[2][3]=273.e00;
    gas->SP[3][3]=0.77e00;
    gas->SP[4][3]=1.e00;
    gas->SP[5][3]=5.312e-26;
    gas->SP[6][3]=0.e00;
    gas->ISPR[1][3]=2;
    gas->ISPR[2][3]=0;
    gas->SPR[1][3]=5.e00;
    gas->ISPV[3]=1;               // the number of vibrational modes
    gas->SPVM[1][1][3]=2256.e00;          // the characteristic vibrational temperature
    gas->SPVM[2][1][3]=18000.e00;  //90000.D00        // a constant Zv, or the reference Zv
    gas->SPVM[3][1][3]=2256.e00;       // -1 for a constant Zv, or the reference temperature
    gas->SPVM[5][1][3]=1.e00;
    gas->ISPVM[1][1][3]=4;
    gas->ISPVM[2][1][3]=4;
    //species 4 is atomic oxygen O
    gas->SP[1][4]=3.e-10;    //estimate
    gas->SP[2][4]=273.e00;
    gas->SP[3][4]=0.8e00;
    gas->SP[4][4]=1.e00;
    gas->SP[5][4]=2.656e-26;
    gas->SP[6][4]=4.099e-19;
    gas->ISPR[1][4]=0;
    gas->ISPV[4]=0;
    //species 5 is hydroxy OH
    gas->SP[1][5]=4.e-10;       //estimate
    gas->SP[2][5]=273.e00;
    gas->SP[3][5]=0.75e00;      //-estimate
    gas->SP[4][5]=1.0e00;
    gas->SP[5][5]=2.823e-26;
    gas->SP[6][5]=6.204e-20;
    gas->ISPR[1][5]=2;
    gas->ISPR[2][5]=0;
    gas->SPR[1][5]=5.e00;
    gas->ISPV[5]=1;
    gas->SPVM[1][1][5]=5360.e00;
    gas->SPVM[2][1][5]=20000.e00;   //estimate
    gas->SPVM[3][1][5]=2500.e00;    //estimate
    gas->SPVM[5][1][5]=1.0e00;
    gas->ISPVM[1][1][5]=2;
    gas->ISPVM[2][1][5]=4;
    //species 6 is water vapor H2O
    gas->SP[1][6]=4.5e-10;      //estimate
    gas->SP[2][6]=273.e00;
    gas->SP[3][6]=0.75e00 ;     //-estimate
    gas->SP[4][6]=1.0e00;
    gas->SP[5][6]=2.99e-26;
    gas->SP[6][6]=-4.015e-19;
    gas->ISPR[1][6]=3;
    gas->ISPR[2][6]=0;
    gas->SPR[1][6]=5.e00;
    gas->ISPV[6]=3;
    gas->SPVM[1][1][6]=5261.e00;  //symmetric stretch mode
    gas->SPVM[2][1][6]=20000.e00;   //estimate
    gas->SPVM[3][1][6]=2500.e00;    //estimate
    gas->SPVM[5][1][6]=1.e00;
    gas->SPVM[1][2][6]=2294.e00;  //bend mode
    gas->SPVM[2][2][6]=20000.e00;   //estimate
    gas->SPVM[3][2][6]=2500.e00;    //estimate
    gas->SPVM[5][2][6]=1.0e00;
    gas->SPVM[1][3][6]=5432.e00;  //asymmetric stretch mode
    gas->SPVM[2][3][6]=20000.e00;   //estimate
    gas->SPVM[3][3][6]=2500.e00 ;   //estimate
    gas->SPVM[5][3][6]=1.e00;
    gas->ISPVM[1][1][6]=2;
    gas->ISPVM[2][1][6]=5;
    gas->ISPVM[1][2][6]=2;
    gas->ISPVM[2][2][6]=5;
    gas->ISPVM[1][3][6]=2;
    gas->ISPVM[2][3][6]=5;
    //species 7 is hydroperoxy HO2
    gas->SP[1][7]=5.5e-10;       //estimate
    gas->SP[2][7]=273.e00;
    gas->SP[3][7]=0.75e00 ;     //-estimate
    gas->SP[4][7]=1.0e00;
    gas->SP[5][7]=5.479e-26;
    gas->SP[6][7]=2.04e-20;
    gas->ISPR[1][7]=2;    //assumes that HO2 is linear
    gas->ISPR[2][7]=0;
    gas->SPR[1][7]=5.e00;
    gas->ISPV[7]=3;
    gas->SPVM[1][1][7]=4950.e00;
    gas->SPVM[2][1][7]=20000.e00;   //estimate
    gas->SPVM[3][1][7]=2500.e00  ;  //estimate
    gas->SPVM[5][1][7]=1.e00;
    gas->SPVM[1][2][7]=2000.e00;
    gas->SPVM[2][2][7]=20000.e00;   //estimate
    gas->SPVM[3][2][7]=2500.e00;    //estimate
    gas->SPVM[5][2][7]=1.e00;
    gas->SPVM[1][3][7]=1580.e00;
    gas->SPVM[2][3][7]=20000.e00;   //estimate
    gas->SPVM[3][3][7]=2500.e00;    //estimate
    gas->SPVM[5][3][7]=1.e00;
    gas->ISPVM[1][1][7]=2;
    gas->ISPVM[2][1][7]=3;
    gas->ISPVM[1][2][7]=2;
    gas->ISPVM[2][2][7]=3;
    gas->ISPVM[1][3][7]=2;
    gas->ISPVM[2][3][7]=3;
    //Species 8 is argon
    gas->SP[1][8]=4.17e-10;
    gas->SP[2][8]=273.15;
    gas->SP[3][8]=0.81   ;
    gas->SP[4][8]=1.0;
    gas->SP[5][8]=6.63e-26;
    gas->SP[6][8]=0.e00;
    gas->ISPR[1][8]=0;
    gas->ISPV[8]=0;
    //
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->ISPRC[i][j]=0;
        }
    }
    //gas->ISPRC=0;    //data is zero unless explicitly set
    //
    gas->ISPRC[4][4]=3;    //O+O+M -> O2+M  recombined species code for an O+O recombination
    gas->ISPRK[4][4]=1;
    gas->SPRC[1][4][4][1]=0.26e00;
    gas->SPRC[2][4][4][1]=-1.3e00;
    gas->SPRC[1][4][4][2]=0.29e00;
    gas->SPRC[2][4][4][2]=-1.3e00;
    gas->SPRC[1][4][4][3]=0.04e00;
    gas->SPRC[2][4][4][3]=-1.5e00;
    gas->SPRC[1][4][4][4]=0.1e00;
    gas->SPRC[2][4][4][4]=-1.4e00;
    gas->SPRC[1][4][4][5]=0.1e00;
    gas->SPRC[2][4][4][5]=-1.4e00;
    gas->SPRC[1][4][4][6]=0.1e00;
    gas->SPRC[2][4][4][6]=-1.4e00;
    gas->SPRC[1][4][4][7]=0.07e00;
    gas->SPRC[2][4][4][7]=-1.5e00;
    gas->SPRC[1][4][4][8]=0.07e00;
    gas->SPRC[2][4][4][8]=-1.5e00;
    gas->SPRT[1][4][4]=1000.e00;
    gas->SPRT[2][4][4]=3000.e00;
    //
    gas->ISPRC[2][2]=1;   //H+H+M -> H2+M
    gas->ISPRK[2][2]=1;
    gas->SPRC[1][2][2][1]=0.07e00;
    gas->SPRC[2][2][2][1]=-2.e00;
    gas->SPRC[1][2][2][2]=0.11e00;
    gas->SPRC[2][2][2][2]=-2.2e00;
    gas->SPRC[1][2][2][3]=0.052e00;
    gas->SPRC[2][2][2][3]=-2.5e00;
    gas->SPRC[1][2][2][4]=0.052e00;
    gas->SPRC[2][2][2][4]=-2.5e00;
    gas->SPRC[1][2][2][5]=0.052e00;
    gas->SPRC[2][2][2][5]=-2.5e00;
    gas->SPRC[1][2][2][6]=0.052e00;
    gas->SPRC[2][2][2][6]=-2.5e00;
    gas->SPRC[1][2][2][7]=0.052e00;
    gas->SPRC[2][2][2][7]=-2.5e00;
    gas->SPRC[1][2][2][8]=0.04e00;
    gas->SPRC[2][2][2][7]=-2.5e00;
    gas->SPRT[1][2][2]=1000.e00;
    gas->SPRT[2][2][2]=3000.e00;
    //
    gas->ISPRC[2][4]=5;    //H+0+M -> OH+M
    gas->ISPRK[2][4]=1;
    gas->SPRC[1][2][4][1]=0.15e00;
    gas->SPRC[2][2][4][1]=-2.e00;
    gas->SPRC[1][2][4][2]=0.04e00;
    gas->SPRC[2][2][4][2]=-1.3e00;
    gas->SPRC[1][2][4][3]=0.04e00;
    gas->SPRC[2][2][4][3]=-1.3e00;
    gas->SPRC[1][2][4][4]=0.04e00;
    gas->SPRC[2][2][4][4]=-1.3e00;
    gas->SPRC[1][2][4][5]=0.04e00;
    gas->SPRC[2][2][4][5]=-1.3e00;
    gas->SPRC[1][2][4][6]=0.21e00;
    gas->SPRC[2][2][4][6]=-2.1e00;
    gas->SPRC[1][2][4][7]=0.18e00;
    gas->SPRC[2][2][4][7]=-2.3e00;
    gas->SPRC[1][2][4][8]=0.16e00;
    gas->SPRC[2][2][4][8]=-2.3e00;
    gas->SPRT[1][2][4]=1000.e00;
    gas->SPRT[2][2][4]=3000.e00;
    //
    gas->ISPRC[2][5]=6;    //H+OH+M -> H2O+M
    gas->ISPRK[2][5]=1;
    gas->SPRC[1][2][5][1]=0.1e00;
    gas->SPRC[2][2][5][1]=-2.0e00;
    gas->SPRC[1][2][5][2]=0.1e00;
    gas->SPRC[2][2][5][2]=-2.0e00;
    gas->SPRC[1][2][5][3]=0.0025e00;
    gas->SPRC[2][2][5][3]=-2.2e00;
    gas->SPRC[1][2][5][4]=0.0025e00;
    gas->SPRC[2][2][5][4]=-2.2e00;
    gas->SPRC[1][2][5][5]=0.0025e00;
    gas->SPRC[2][2][5][5]=-2.2e00;
    gas->SPRC[1][2][5][6]=0.0015e00;
    gas->SPRC[2][2][5][6]=-2.2e00;
    gas->SPRC[1][2][5][7]=0.0027e00;
    gas->SPRC[2][2][5][7]=-2.e00;
    gas->SPRC[1][2][5][8]=0.0025e00;
    gas->SPRC[2][2][5][8]=-2.e00;
    gas->SPRT[1][2][5]=1000.e00;
    gas->SPRT[2][2][5]=3000.e00;
    //
    gas->ISPRC[2][3]=7;   //H+O2+M -> H02+M
    gas->ISPRK[2][3]=1;
    gas->SPRC[1][2][3][1]=0.0001e00;
    gas->SPRC[2][2][3][1]=-1.7e00;
    gas->SPRC[1][2][3][2]=0.0001e00;
    gas->SPRC[2][2][3][2]=-1.7e00;
    gas->SPRC[1][2][3][3]=0.00003e00;
    gas->SPRC[2][2][3][3]=-1.5e00;
    gas->SPRC[1][2][3][4]=0.00003e00;
    gas->SPRC[2][2][3][4]=-1.7e00;
    gas->SPRC[1][2][3][5]=0.00003e00;
    gas->SPRC[2][2][3][5]=-1.7e00;
    gas->SPRC[1][2][3][6]=0.00003e00;
    gas->SPRC[2][2][3][6]=-1.7e00;
    gas->SPRC[1][2][3][7]=0.000012e00;
    gas->SPRC[2][2][3][7]=-1.7e00;
    gas->SPRC[1][2][3][8]=0.00002e00;
    gas->SPRC[2][2][3][8]=-1.7e00;
    gas->SPRT[1][2][3]=1000.e00;
    gas->SPRT[2][2][3]=3000.e00;
    //
    //set the exchange reaction data
    //  memset(gas->SPEX,0,sizeof(****gas->SPEX));//gas->SPEX=0.e00;    //all activation energies and heats of reaction are zero unless set otherwise
    for(int i=0;i<7;i++){
        for(int j=0;j<gas->MMEX+1;j++){
            for(int k=0;k<gas->MSP+1;k++){
                for(int l=0;l<gas->MSP+1;l++)
                    gas->SPEX[i][j][k][l]=0.e00;
            }
        }
    }
    //gas->ISPEX=0;       // ISPEX is also zero unless set otherwise
    for(int i=0;i<gas->MMEX+1;i++){
        for(int j=0;j<8;j++){
            for(int k=0;k<gas->MSP+1;k++){
                for(int l=0;l<gas->MSP+1;l++)
                    gas->ISPEX[i][j][k][l]=0.e00;
            }
        }
    }
    //gas->NSPEX=0;
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->NSPEX[i][j]=0;
        }
    }
    //set the number of exchange reactions for each species pair
    gas->NSPEX[1][3]=1;
    gas->NSPEX[2][7]=3;
    gas->NSPEX[2][3]=1;
    gas->NSPEX[4][5]=1;
    gas->NSPEX[1][4]=1;
    gas->NSPEX[2][5]=1;
    gas->NSPEX[1][5]=1;
    gas->NSPEX[2][6]=1;
    gas->NSPEX[4][6]=2;
    gas->NSPEX[5][5]=2;
    gas->NSPEX[4][7]=1;
    gas->NSPEX[3][5]=1;
    //set the information on the chain reactions
    //
    //H2+O2 -> HO2+H
    gas->ISPEX[1][1][1][3]=1;
    gas->ISPEX[1][2][1][3]=3;
    gas->ISPEX[1][3][1][3]=7;
    gas->ISPEX[1][4][1][3]=2;
    gas->ISPEX[1][5][1][3]=1;
    gas->ISPEX[1][6][1][3]=1;
    gas->SPEX[6][1][1][3]=0.e00;
    gas->NEX[1][1][3]=1;
    //
    //HO2+H -> H2+02
    gas->ISPEX[1][1][2][7]=7;
    gas->ISPEX[1][2][2][7]=2;
    gas->ISPEX[1][3][2][7]=1;
    gas->ISPEX[1][4][2][7]=3;
    gas->ISPEX[1][5][2][7]=1;
    gas->ISPEX[1][6][2][7]=1;
    gas->ISPEX[1][7][2][7]=1;
    //H02 is H-O-O so that not all vibrational modes contribute to this reaction, but the numbers here are guesses//
    gas->SPEX[1][1][2][7]=20.e00;
    gas->SPEX[2][1][2][7]=0.4e00;
    gas->SPEX[4][1][2][7]=2000.e00;
    gas->SPEX[5][1][2][7]=3000.e00;
    gas->SPEX[6][1][2][7]=0.e00;
    gas->NEX[1][2][7]=2;
    //
    //O2+H -> OH+O
    gas->ISPEX[1][1][2][3]=3;
    gas->ISPEX[1][2][2][3]=2;
    gas->ISPEX[1][3][2][3]=5;
    gas->ISPEX[1][4][2][3]=4;
    gas->ISPEX[1][5][2][3]=1;
    gas->ISPEX[1][6][2][3]=1;
    gas->SPEX[6][1][2][3]=0.e00;
    gas->NEX[1][2][3]=3;
    //
    //OH+O -> O2+H
    gas->ISPEX[1][1][4][5]=5;
    gas->ISPEX[1][2][4][5]=4;
    gas->ISPEX[1][3][4][5]=3;
    gas->ISPEX[1][4][4][5]=2;
    gas->ISPEX[1][5][4][5]=1;
    gas->ISPEX[1][6][4][5]=1;
    gas->ISPEX[1][7][4][5]=1;
    gas->SPEX[1][1][4][5]=0.65e00;
    gas->SPEX[2][1][4][5]=-0.26;
    gas->SPEX[4][1][4][5]=2000.e00;
    gas->SPEX[5][1][4][5]=3000.e00;
    gas->SPEX[6][1][4][5]=0.e00;
    gas->NEX[1][4][5]=4;
    //
    //H2+O -> OH+H
    gas->ISPEX[1][1][1][4]=1;
    gas->ISPEX[1][2][1][4]=4;
    gas->ISPEX[1][3][1][4]=5;
    gas->ISPEX[1][4][1][4]=2;
    gas->ISPEX[1][5][1][4]=1;
    gas->ISPEX[1][6][1][4]=1;
    gas->SPEX[6][1][1][4]=0.e00;
    gas->NEX[1][1][4]=5;
    //
    //OH+H -> H2+O
    gas->ISPEX[1][1][2][5]=5;
    gas->ISPEX[1][2][2][5]=2;
    gas->ISPEX[1][3][2][5]=1;
    gas->ISPEX[1][4][2][5]=4;
    gas->ISPEX[1][5][2][5]=1;
    gas->ISPEX[1][6][2][5]=1;
    gas->ISPEX[1][7][2][5]=1;
    gas->SPEX[1][1][2][5]=0.5e00;
    gas->SPEX[2][1][2][5]=-0.2e00;
    gas->SPEX[4][1][2][5]=2000.e00;
    gas->SPEX[5][1][2][5]=3000.e00;
    gas->SPEX[6][1][2][5]=0.e00;
    gas->NEX[1][2][5]=6;
    //
    //H20+H -> OH+H2
    gas->ISPEX[1][1][2][6]=6;
    gas->ISPEX[1][2][2][6]=2;
    gas->ISPEX[1][3][2][6]=5;
    gas->ISPEX[1][4][2][6]=1;
    gas->ISPEX[1][5][2][6]=1;
    gas->ISPEX[1][6][2][6]=1;
    gas->SPEX[6][1][2][6]=2.0e-19;
    gas->NEX[1][2][6]=7;
    
    //OH+H2 -> H2O+H
    gas->ISPEX[1][1][1][5]=5;
    gas->ISPEX[1][2][1][5]=1;
    gas->ISPEX[1][3][1][5]=6;
    gas->ISPEX[1][4][1][5]=2;
    gas->ISPEX[1][5][1][5]=1;
    gas->ISPEX[1][6][1][5]=1;
    gas->ISPEX[1][7][1][5]=1;
    gas->SPEX[1][1][1][5]=0.5;
    gas->SPEX[2][1][1][5]=-0.2;
    gas->SPEX[4][1][1][5]=2000.e00;
    gas->SPEX[5][1][1][5]=3000.e00;
    gas->SPEX[6][1][1][5]=0.e00;
    gas->NEX[1][1][5]=8;
    //
    //H2O+O -> OH+OH
    gas->ISPEX[1][1][4][6]=6;
    gas->ISPEX[1][2][4][6]=4;
    gas->ISPEX[1][3][4][6]=5;
    gas->ISPEX[1][4][4][6]=5;
    gas->ISPEX[1][5][4][6]=1;
    gas->ISPEX[1][6][4][6]=1;
    gas->SPEX[6][1][4][6]=0.e00;
    gas->NEX[1][4][6]=9;
    //
    //0H+OH -> H2O+O
    gas->ISPEX[1][1][5][5]=5;
    gas->ISPEX[1][2][5][5]=5;
    gas->ISPEX[1][3][5][5]=6;
    gas->ISPEX[1][4][5][5]=4;
    gas->ISPEX[1][5][5][5]=1;
    gas->ISPEX[1][6][5][5]=1;
    gas->ISPEX[1][7][5][5]=1;
    gas->SPEX[1][1][5][5]=0.35;
    gas->SPEX[2][1][5][5]=-0.2 ;
    gas->SPEX[4][1][5][5]=2000.e00;
    gas->SPEX[5][1][5][5]=3000.e00;
    gas->SPEX[6][1][5][5]=0.e00;
    gas->NEX[1][5][5]=10;
    //
    //OH+OH  -> HO2+H
    //
    gas->ISPEX[2][1][5][5]=5;
    gas->ISPEX[2][2][5][5]=5;
    gas->ISPEX[2][3][5][5]=7;
    gas->ISPEX[2][4][5][5]=2;
    gas->ISPEX[2][5][5][5]=1;
    gas->ISPEX[2][6][5][5]=1;
    gas->SPEX[6][2][5][5]=0.e00;
    gas->NEX[2][5][5]=11;
    //
    //H02+H -> 0H+OH
    gas->ISPEX[2][1][2][7]=7;
    gas->ISPEX[2][2][2][7]=2;
    gas->ISPEX[2][3][2][7]=5;
    gas->ISPEX[2][4][2][7]=5;
    gas->ISPEX[2][5][2][7]=1;
    gas->ISPEX[2][6][2][7]=1;
    gas->ISPEX[2][7][2][7]=1;
    gas->SPEX[1][2][2][7]=120.e00;
    gas->SPEX[2][2][2][7]=-0.05e00;
    gas->SPEX[4][2][2][7]=2000.e00;
    gas->SPEX[5][2][2][7]=3000.e00;
    gas->SPEX[6][2][2][7]=0.e00;
    gas->NEX[2][2][7]=12;
    //
    //H2O+O -> HO2+H
    //
    gas->ISPEX[2][1][4][6]=6;
    gas->ISPEX[2][2][4][6]=4;
    gas->ISPEX[2][3][4][6]=7;
    gas->ISPEX[2][4][4][6]=2;
    gas->ISPEX[2][5][4][6]=1;
    gas->ISPEX[2][6][4][6]=1;
    gas->SPEX[6][2][4][6]=0.e00;
    gas->NEX[2][4][6]=13;
    //
    //H02+H -> H2O+O
    //
    gas->ISPEX[3][1][2][7]=7;
    gas->ISPEX[3][2][2][7]=2;
    gas->ISPEX[3][3][2][7]=6;
    gas->ISPEX[3][4][2][7]=4;
    gas->ISPEX[3][5][2][7]=1;
    gas->ISPEX[3][6][2][7]=1;
    gas->ISPEX[3][7][2][7]=1;
    gas->SPEX[1][3][2][7]=40.e00;
    gas->SPEX[2][3][2][7]=-1.e00;
    gas->SPEX[4][3][2][7]=2000.e00;
    gas->SPEX[5][3][2][7]=3000.e00;
    gas->SPEX[6][3][2][7]=0.e00;
    gas->NEX[3][2][7]=14;
    //
    //OH+O2 -> HO2+O
    //
    gas->ISPEX[1][1][3][5]=5;
    gas->ISPEX[1][2][3][5]=3;
    gas->ISPEX[1][3][3][5]=7;
    gas->ISPEX[1][4][3][5]=4;
    gas->ISPEX[1][5][3][5]=1;
    gas->ISPEX[1][6][3][5]=1;
    gas->SPEX[6][1][3][5]=0.e00;
    gas->NEX[1][3][5]=15;
    //
    //H02+0 -> OH+O2
    //
    gas->ISPEX[1][1][4][7]=7;
    gas->ISPEX[1][2][4][7]=4;
    gas->ISPEX[1][3][4][7]=5;
    gas->ISPEX[1][4][4][7]=3;
    gas->ISPEX[1][5][4][7]=1;
    gas->ISPEX[1][6][4][7]=1;
    gas->ISPEX[1][7][4][7]=1;
    gas->SPEX[1][1][4][7]=100.e00;
    gas->SPEX[2][1][4][7]=0.15e00;
    gas->SPEX[4][1][4][7]=2000.e00;
    gas->SPEX[5][1][4][7]=3000.e00;
    gas->SPEX[6][1][4][7]=0.e00;
    gas->NEX[1][4][7]=16;
    
    //
    DERIVED_GAS_DATA();
    //
    cout<<"OXYGEN_HYDROGEN data done"<<endl;
    return;
}
//***************************************************************************
//*************************END OF GAS DATABASE*******************************
//***************************************************************************
//
void DERIVED_GAS_DATA()
{
    //
    //GAS gas;
    //CALC calc;
    int I,II,J,JJ,K,L,M,MM,N,JMAX,MOLSP,MOLOF,NSTEP,IMAX;
    double A,B,BB,C,X,T,CUR,EAD,TVD,ZVT,ERD,PETD,DETD,PINT,ETD,SUMD,VAL;
    double **BFRAC,**TOT;
    double ****VRRD;
    double *****VRREX;
    //
    //VRRD(1,L,M,K) dissociation rate coefficient to species L,M for vibrational level K at 5,000 K
    //VRRD(2,L,M,K) similar for 15,000 K
    //VRREX(1,J,L,M,K)  Jth exchange rate coefficient to species L,M for vibrational level K at 1,000 K
    //VRREX(2,J,L,M,K) similar for 3,000 K
    //BFRAC(2,J) Boltzmann fraction
    //JMAX imax-1
    //T temperature
    //CUR sum of level resolved rates
    //
    
    VRRD = new double ***[3];
    for (int i = 0; i < 3; ++i)
    {
        VRRD[i] = new double **[gas->MSP+1];
        for (int j = 0; j < gas->MSP+1; ++j)
        {
            VRRD[i][j] = new double *[gas->MSP+1];
            for(int k=0; k<gas->MSP+1; ++k)
                VRRD[i][j][k]=new double [gas->MVIBL+1];
        }
    }
    
    BFRAC = new double*[gas->MVIBL+1];
    for(int i =0; i< (gas->MVIBL+1); ++i)
        BFRAC[i] = new double[3];
    
    VRREX = new double ****[3];
    for (int i = 0; i < 3; ++i)
    {
        VRREX[i] = new double ***[gas->MMEX+1];
        for (int j = 0; j < gas->MMEX+1; ++j)
        {
            VRREX[i][j] = new double **[gas->MSP+1];
            for(int k=0; k<gas->MSP+1; ++k)
            {
                VRREX[i][j][k]=new double *[gas->MSP+1];
                for(int l=0; l<gas->MSP+1; ++l)
                    VRREX[i][j][k][l]= new double[gas->MVIBL+1];
            }
        }
    }
    
    TOT = new double*[gas->MVIBL+1];
    for(int i =0; i< (gas->MVIBL+1); ++i)
        TOT[i] = new double[3];
    
    // ALLOCATE (VRRD(2,MSP,MSP,0:MVIBL),BFRAC(0:MVIBL,2),VRREX(2,MMEX,MSP,MSP,0:MVIBL),TOT(0:MVIBL,2),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE VIB. RES. DISS. RATES',ERROR
    // END IF
    //
    cout<<"Setting derived gas data"<<endl;
    //copy the L,M data that has been specified for L < M so that it applies also for M>L
    for(L=1;L<=gas->MSP;L++){
        for(M=1;M<=gas->MSP;M++){
            if(L > M){
                gas->NSPEX[L][M]=gas->NSPEX[M][L];
                gas->ISPRC[L][M]=gas->ISPRC[M][L];
                gas->ISPRK[L][M]=gas->ISPRK[M][L];
                for(K=1;K<=gas->MSP;K++){
                    gas->SPRT[1][L][M]=gas->SPRT[1][M][L];
                    gas->SPRT[2][L][M]=gas->SPRT[2][M][L];
                    gas->SPRC[1][L][M][K]=gas->SPRC[1][M][L][K];
                    gas->SPRC[2][L][M][K]=gas->SPRC[2][M][L][K];
                }
                for(K=1;K<=gas->MMEX;K++){
                    gas->NEX[K][L][M]=gas->NEX[K][M][L];
                    for(J=1;J<=6;J++){
                        gas->SPEX[J][K][L][M]=gas->SPEX[J][K][M][L];
                    }
                    for(J=1;J<=7;J++){
                        gas->ISPEX[K][J][L][M]=gas->ISPEX[K][J][M][L];
                    }
                }
            }
        }
    }
    //
    if(gas->MMVM > 0){
        //set the characteristic dissociation temperatures
        for(L=1;L<=gas->MSP;L++){
            if(gas->ISPV[L] > 0){
                for(K=1;K<=gas->ISPV[L];K++)
                {
                    I=gas->ISPVM[1][K][L];
                    J=gas->ISPVM[2][K][L];
                    gas->SPVM[4][K][L]=(gas->SP[6][I]+gas->SP[6][J]-gas->SP[6][L])/BOLTZ;
                    //WRITE (9,*) 'Char. Diss temp of species',L,' is',SPVM(4,K,L)
                    file_9<<"Char. Diss temp of species "<<L<<" is "<<gas->SPVM[4][K][L]<<endl;
                }
            }
        }
    }
    //
    if(gas->MMEX > 0){
        //set the heats of reaction of the exchange and chain reactions
        for(L=1;L<=gas->MSP;L++){
            for(M=1;M<=gas->MSP;M++){
                for(J=1;J<=gas->MMEX;J++){
                    if((gas->ISPEX[J][3][L][M]> 0) && (gas->ISPEX[J][4][L][M]>0) && (gas->ISPEX[J][1][L][M]>0) && (gas->ISPEX[J][2][L][M]>0)){
                        gas->SPEX[3][J][L][M]=gas->SP[6][gas->ISPEX[J][1][L][M]]+gas->SP[6][gas->ISPEX[J][2][L][M]]-gas->SP[6][gas->ISPEX[J][3][L][M]]-gas->SP[6][gas->ISPEX[J][4][L][M]];
                        // WRITE (9,*) 'Reaction',NEX(J,L,M),' heat of reaction',SPEX(3,J,L,M)
                        file_9<<"Reaction "<<gas->NEX[J][L][M]<<" heat of reaction"<<gas->SPEX[3][J][L][M]<<endl;
                    }
                }
            }
        }
    }
    //
    if(gas->MELE > 1){
        //set the electronic cross-section ratios to a mean electronic relaxation collision number
        //(equipartition is not achieved unless there is a single number)
        for(L=1;L<=gas->MSP;L++){
            A=0.e00;
            for(K=1;K<=gas->NELL[L];K++){
                A=A+gas->QELC[3][K][L];
            }
            gas->QELC[3][1][L]=A/double(gas->NELL[L]);
        }
    }
    //
    //set the cumulative distributions of the post-recombination vibrational distributions for establishment of detailed balance
    for(L=1;L<=gas->MSP;L++){
        for(M=1;M<=gas->MSP;M++){
            if(gas->ISPRC[L][M] > 0){
                N=gas->ISPRC[L][M];   //recombined species
                K=gas->ISPRK[L][M];   //relevant vibrational mode
                //WRITE (9,*) 'SPECIES',L,M,' RECOMBINE TO',N
                file_9<<"SPECIES "<<L<<" "<<M<<" RECOMBINE TO"<<N<<endl;
                JMAX=gas->SPVM[4][K][N]/gas->SPVM[1][K][N];
                if(JMAX > gas->MVIBL){
                    cout<<" The variable MVIBL="<<gas->MVIBL<<" in the gas database must be increased to"<<JMAX<<endl;
                    cout<<"Enter 0 ENTER to stop";
                    cin>> A;
                    return ;
                }
                A=2.5e00-gas->SP[3][N];
                for(I=1;I<=2;I++){
                    if(I == 1) T=gas->SPRT[1][L][M];
                    if(I == 2) T=gas->SPRT[2][L][M];
                    //WRITE (9,*) 'TEMPERATURE',T
                    file_9<<"TEMPERATURE "<<T<<endl;
                    CUR=0.e00;
                    for(J=0;J<=JMAX;J++){
                        X=double(JMAX+1-J)*gas->SPVM[1][K][N]/T;
                        CQAX(A,X,B);
                        VRRD[I][L][M][J]=B*exp(-double(J)*gas->SPVM[1][K][N]/T);
                        CUR=CUR+VRRD[I][L][M][J];
                    }
                    B=0.e00;
                    for(J=0;J<=JMAX;J++){
                        B=B+VRRD[I][L][M][J]/CUR;
                        gas->SPRP[I][L][M][J]=B;
                        //WRITE (9,*) 'CDF level dissoc',J,SPRP(I,L,M,J)
                        file_9<< "CDF level dissoc "<<J<<" "<<gas->SPRP[I][L][M][J];
                    }
                }
            }
        }
    }
    //
    //READ (*,*)  //optionally pause program to check cumulative distributions for exchange and chain reactions
    //
    //set the cumulative distributions of the post-reverse vibrational distributions for establishment of detailed balance
    for(L=1;L<=gas->MSP;L++){
        for(M=1;M<=gas->MSP;M++){
            if(gas->NSPEX[L][M] > 0){
                for(K=1;K<=gas->NSPEX[L][M];K++){
                    if(gas->SPEX[3][K][L][M] > 0.e00){         //exothermic (reverse) exchange reaction
                        //L,M are the species in the reverse reaction, E_a of forward reaction is SPEX(3,K,L,M)
                        //WRITE (9,*) 'SPECIES',L,M,' REVERSE REACTION'
                        file_9<<"SPECIES "<<L<<" "<<M<<" REVERSE REACTION"<<endl;
                        MOLSP=gas->ISPEX[K][3][L][M];  //molecuke that splits in the forward reaction
                        MOLOF=gas->ISPEX[K][4][L][M];
                        JMAX=(gas->SPEX[3][K][L][M]+gas->SPEX[6][K][MOLSP][MOLOF])/(BOLTZ*gas->SPVM[1][gas->ISPEX[K][5][L][M]][MOLSP])+15;   //should always be less than the JMAX set by dissociation reactions
                        for(I=1;I<=2;I++){
                            if(I == 1) T=gas->SPEX[4][K][L][M];
                            if(I == 2) T=gas->SPEX[5][K][L][M];
                            for(J=0;J<=JMAX;J++){
                                EAD=(gas->SPEX[3][K][L][M]+gas->SPEX[6][K][MOLSP][MOLOF])/(BOLTZ*T);
                                TVD=gas->SPVM[1][gas->ISPEX[K][5][L][M]][MOLSP]/T;
                                ZVT=1.e00/(1.e00-exp(-TVD));
                                C=ZVT/(tgamma(2.5e00-gas->SP[3][MOLSP])*exp(-EAD));  //coefficient of integral
                                ERD=EAD-double(J)*TVD;
                                if(ERD < 0.e00) ERD=0.e00;
                                PETD=ERD;
                                DETD=0.01e00;
                                PINT=0.e00;  //progressive value of integral
                                NSTEP=0;
                                A=1.e00;
                                while(A > 1.e-10){
                                    NSTEP=NSTEP+1;
                                    ETD=PETD+0.5e00*DETD;
                                    SUMD=0.e00;  //normalizing sum in the denominator
                                    IMAX=ETD/TVD+J;
                                    for(II=0;II<=IMAX;II++){
                                        SUMD=SUMD+pow((1.e00-double(II)*TVD/(ETD+double(J)*TVD)),(1.5e00-gas->SP[3][MOLSP]));
                                    }
                                    VAL=(pow((ETD*(1.e00-EAD/(ETD+double(J)*TVD))),(1.5e00-gas->SP[3][MOLSP]))/SUMD)*exp(-ETD);
                                    PINT=PINT+VAL*DETD;
                                    A=VAL/PINT;
                                    PETD=ETD+0.5e00*DETD;
                                }
                                VRREX[I][K][L][M][J]=C*PINT;
                                //              WRITE (*,*) 'Level ratio exch',I,J,VRREX(I,K,L,M,J)
                            }
                        }
                        //
                        //memset(TOT,0.e00,sizeof(**TOT));//TOT=0.e00;
                        for(int i=0;i<gas->MVIBL+1;i++){
                            for(int j=0;j<gas->MVIBL+1;j++){
                                TOT[i][j]=0;
                            }
                        }
                        for(I=1;I<=2;I++){
                            if(I == 1) T=gas->SPEX[4][K][L][M];
                            if(I == 2) T=gas->SPEX[5][K][L][M];
                            for(J=0;J<=JMAX;J++){
                                TVD=gas->SPVM[1][gas->ISPEX[K][5][L][M]][MOLSP]/T;
                                ZVT=1.e00/(1.e00-exp(-TVD));
                                BFRAC[J][I]=exp(-J*gas->SPVM[1][gas->ISPEX[K][5][L][M]][MOLSP]/T)/ZVT;    //Boltzmann fraction
                                VRREX[I][K][L][M][J]=VRREX[I][K][L][M][J]*BFRAC[J][I];
                                //              WRITE (*,*) 'Contribution',I,J,VRREX(I,K,L,M,J)
                                for(MM=0;MM<=J;MM++)
                                    TOT[J][I]=TOT[J][I]+VRREX[I][K][L][M][MM];
                            }
                        }
                        //
                        for(I=1;I<=2;I++){
                            for(J=0;J<=JMAX;J++){
                                gas->SPREX[I][K][L][M][J]=TOT[J][I];
                                if(J == JMAX) gas->SPREX[I][K][L][M][J]=1.e00;
                                //WRITE (9,*) 'Cumulative',I,J,SPREX(I,K,L,M,J)
                                file_9<<"Cumulative "<<I<<" "<<J<<" "<<gas->SPREX[I][K][L][M][J];
                            }
                        }
                    }
                }
                gas->NSLEV=0;
                //memset(gas->SLER,0.e00,sizeof(*gas->SLER));//gas->SLER=0.e00;
                for(int i=0;i<gas->MSP+1;i++)
                    gas->SLER[i]=0.e00;
            }
        }
    }
    //
    //READ (*,*)  //optionally pause program to check cumulative distributions for exchange abd chain reactions
    return;
}

void READ_DATA()
{
    //CALC calc;
    //MOLECS molecs;
    //GAS gas;
    //OUTPUT output;
    //GEOM_1D geom;
    fstream file_3;
    fstream file_4;
    
    int NVERD,MVERD,N,K;
    if(calc->ICLASS==0)
    {
        cout<<"Reading the data file DS0D.DAT"<<endl;
        file_4.open("DS0D.DAT", ios::in);
        file_3.open("DS0D.TXT", ios::out);
        file_3<<"Data summary for program DSMC"<<endl;
        
        // OPEN (4,FILE='DS0D.DAT')
        // OPEN (3,FILE='DS0D.TXT')
        // WRITE (3,*) 'Data summary for program DSMC'
    }
    if(calc->ICLASS==1)
    {
        cout<<"Reading the data file DS1D.DAT"<<endl;
        file_4.open("DS1D.DAT", ios::in);
        file_3.open("DS1D.TXT", ios::out );
        file_3<<"Data summary for program DSMC"<<endl;
        // OPEN (4,FILE='DS1D.DAT')
        // OPEN (3,FILE='DS1D.TXT')
        // WRITE (3,*) 'Data summary for program DSMC'
    }
    //the following items are common to all classes of flow
    file_4>>NVERD;
    file_3<<"The n in version number n.m is "<<NVERD<<endl;
    file_4>>MVERD;
    file_3<<"The m in version number n.m is "<<MVERD<<endl;
    file_4>>calc->IMEG;
    file_3<<"The approximate number of megabytes for the calculation is "<<calc->IMEG<<endl;
    file_4>>gas->IGAS;
    file_3<<gas->IGAS<<endl;//gas->IGAS=1;
    // READ (4,*) NVERD
    // WRITE (3,*) 'The n in version number n.m is',NVERD
    // READ (4,*) MVERD
    // WRITE (3,*) 'The m in version number n.m is',MVERD
    // READ (4,*) IMEG //calc->IMEG
    // WRITE (3,*) 'The approximate number of megabytes for the calculation is',IMEG //calc->IMEG
    // READ (4,*) IGAS //gas->IGAS
    // WRITE (3,*) IGAS //gas->IGAS
    if(gas->IGAS==1)
    {
        file_3<<" Hard sphere gas "<<endl;
        // WRITE (3,*) 'Hard sphere gas'
        HARD_SPHERE();
    }
    if(gas->IGAS==2)
    {
        file_3<<"Argon "<<endl;
        // WRITE (3,*) 'Argon'
        ARGON();
    }
    if(gas->IGAS==3)
    {
        file_3<<"Ideal nitrogen"<<endl;
        // WRITE (3,*) 'Ideal nitrogen'
        IDEAL_NITROGEN();
    }
    if(gas->IGAS==4)
    {
        file_3<<"Real oxygen "<<endl;
        // WRITE (3,*) 'Real oxygen'
        REAL_OXYGEN();
    }
    if(gas->IGAS==5)
    {
        file_3<<"Ideal air "<<endl;
        // TE (3,*) 'Ideal air'
        IDEAL_AIR();
    }
    if(gas->IGAS==6)
    {
        file_3<<"Real air @ 7.5 km/s "<<endl;
        // RITE (3,*) 'Real air @ 7.5 km/s'
        REAL_AIR();
    }
    if(gas->IGAS==7)
    {
        file_3<<"Helium-argon-xenon mixture "<<endl;
        // WRITE (3,*) 'Helium-argon-xenon mixture'
        HELIUM_ARGON_XENON();
    }
    if(gas->IGAS==8)
    {
        file_3<<"Oxygen-hydrogen "<<endl;
        // WRRITE (3,*) 'Oxygen-hydrogen'
        OXYGEN_HYDROGEN();
    }
    file_3<<"The gas properties are:- "<<endl;
    file_4>>gas->FND[1];
    file_3<<"The stream number density is "<<gas->FND[1]<<endl;
    file_4>>gas->FTMP[1];
    file_3<<"The stream temperature is "<<gas->FTMP[1]<<endl;
    // WRITE (3,*) 'The gas properties are:-'
    // READ (4,*) FND(1) //gas->FND[1]
    // WRITE (3,*) '    The stream number density is',FND(1) ////gas->FND[1]
    // READ (4,*) FTMP(1) //gas->FTMP[1]
    // WRITE (3,*) '    The stream temperature is',FTMP(1) //gas->FTMP[1]
    if(gas->MMVM>0)
    {
        file_4>>gas->FVTMP[1];
        file_3<<"The stream vibrational and electronic temperature is "<<gas->FVTMP[1]<<endl;
        // READ (4,*) FVTMP(1) //gas->FVTMP;
        // WRITE (3,*) '    The stream vibrational and electronic temperature is',FVTMP(1) //gas->FVTMP[1]
    }
    if(calc->ICLASS==1)
    {
        file_4>>gas->VFX[1];
        file_3<<"The stream velocity in the x direction is "<<gas->VFX[1]<<endl;
        file_4>>gas->VFY[1];
        file_3<<"The stream velocity in the y direction is "<<gas->VFY[1]<<endl;
        // READ (4,*) VFX(1) //gas->VFX[1]
        // WRITE (3,*) '    The stream velocity in the x direction is',VFX(1) //gas->VFX[1]
        // READ (4,*) VFY(1) ////gas->VFY[1]
        // WRITE (3,*) '    The stream velocity in the y direction is',VFY(1) ////gas->VFY[1]
    }
    if(gas->MSP>1)
    {
        for(N=1;N<=gas->MSP;N++)
        {
            file_4>>gas->FSP[N][1];
            file_3<<" The fraction of species "<<N<<" is "<<gas->FSP[N][1]<<endl;
            // READ (4,*) FSP(N,1) //gas->FSP[N][1]
            // WRITE (3,*) '    The fraction of species',N,' is',FSP(N,1) //gas->FSP[N][1]
        }
    }
    else
    {
        gas->FSP[1][1]=1.0; //simple gas
    }
    if(calc->ICLASS==0){
        //       !--a homogeneous gas case is calculated as a one-dimensional flow with a single sampling cell
        // !--set the items that are required in the DS1D.DAT specification
        geom->IFX=0;
        geom->JFX=1;
        geom->XB[1]=0.e00;
        geom->XB[2]=0.0001e00*1.e25/gas->FND[1];
        geom->ITYPE[1]=1;
        geom->ITYPE[2]=1;
        gas->VFX[1]=0.e00;
        calc->IGS=1;
        calc->ISECS=0;
        calc->IREM=0;
        calc->MOLSC=10000*calc->IMEG; //a single sampling cell
    }
    if(calc->ICLASS==1)
    {
        file_4>>geom->IFX;
        // READ (4,*) IFX //geom->IFX
        if(geom->IFX==0)
            file_3<<"Plane Flow"<<endl;
        // WRITE (3,*) 'Plane flow'
        if(geom->IFX==0)
            file_3<<"Cylindrical flow"<<endl;
        // WRITE (3,*) 'Cylindrical flow'
        if(geom->IFX==0)
            file_3<<"Spherical flow"<<endl;
        // WRITE (3,*) 'Spherical flow'
        geom->JFX=geom->IFX+1;
        file_4>>geom->XB[1];
        // READ (4,*) XB(1) //geom->XB[1]
        file_3<<"The minimum x coordinate is "<<geom->XB[1]<<endl;
        // WRITE (3,*) 'The minimum x coordinate is',XB(1) //geom->XB[1]
        file_4>>geom->ITYPE[1];
        // READ (4,*) ITYPE(1) //geom->ITYPE[1]
        if(geom->ITYPE[1]==0)
            file_3<<"The minimum x coordinate is a stream boundary"<<endl;
        // WRITE (3,*) 'The minimum x coordinate is a stream boundary'
        if(geom->ITYPE[1]==1)
            file_3<<"The minimum x coordinate is a plane of symmetry"<<endl;
        // WRITE (3,*) 'The minimum x coordinate is a plane of symmetry'
        if(geom->ITYPE[1]==2)
            file_3<<"The minimum x coordinate is a solid surface"<<endl;
        // WRITE (3,*) 'The minimum x coordinate is a solid surface'
        if(geom->ITYPE[1]==3)
            file_3<<"The minimum x coordinate is a vacuum"<<endl;
        // WRITE (3,*) 'The minimum x coordinate is a vacuum'
        if(geom->ITYPE[1]==4)
            file_3<<"The minimum x coordinate is an axis or center"<<endl;
        // WRITE (3,*) 'The minimum x coordinate is an axis or center'
        if(geom->ITYPE[1]==2)
        {
            file_3<<"The minimum x boundary is a surface with the following properties"<<endl;
            file_4>>gas->TSURF[1];
            file_3<<"The temperature of the surface is "<<gas->TSURF[1]<<endl;
            file_4>>gas->FSPEC[1];
            file_3<<"The fraction of specular reflection is "<<gas->FSPEC[1]<<endl;
            file_4>>gas->VSURF[1];
            file_3<<"The velocity in the y direction of this surface is "<<gas->VSURF[1];
            // WRITE (3,*) 'The minimum x boundary is a surface with the following properties'
            // READ (4,*) TSURF(1) //gas->TSURF[1]
            // WRITE (3,*) '     The temperature of the surface is',TSURF(1) //gas->TSURF[1]
            // READ (4,*) FSPEC(1) //gas->FSPEC[1]
            // WRITE (3,*) '     The fraction of specular reflection is',FSPEC(1) //gas->FSPEC[1]
            // READ (4,*) VSURF(1) //gas->VSURF[1]
            // WRITE (3,*) '     The velocity in the y direction of this surface is',VSURF(1) //gas->VSURF[1]
        }
        file_4>>geom->XB[2];
        file_3<<"The maximum x coordinate is "<<geom->XB[2]<<endl;
        file_4>>geom->ITYPE[2];
        // READ (4,*) XB(2) //geom->XB[2]
        // WRITE (3,*) 'The maximum x coordinate is',XB(2)//geom->XB[2]
        // READ (4,*) ITYPE(2)//geom->ITYPE[2]
        if(geom->ITYPE[2]==0)
            file_3<<"The mmaximum  x coordinate is a stream boundary"<<endl;
        // WRITE (3,*) 'The mmaximum  x coordinate is a stream boundary'
        if(geom->ITYPE[2]==1)
            file_3<<"The maximum x coordinate is a plane of symmetry"<<endl;
        // WRITE (3,*) 'The maximum x coordinate is a plane of symmetry'
        if(geom->ITYPE[2]==2)
            file_3<<"The maximum  x coordinate is a solid surface"<<endl;
        // WRITE (3,*) 'The maximum  x coordinate is a solid surface'
        if(geom->ITYPE[2]==3)
            file_3<<"The maximum  x coordinate is a vacuum"<<endl;
        // WRITE (3,*) 'The maximum  x coordinate is a vacuum'
        calc->ICN=0;
        if(geom->ITYPE[2]==4)
        {
            file_3<<"The maximum x coordinate is a stream boundary with a fixed number of simulated molecules"<<endl;
            // WRITE (3,*) 'The maximum x coordinate is a stream boundary with a fixed number of simulated molecules'
            if(gas->MSP==1)
                calc->ICN=1;
        }
        if(geom->ITYPE[2]==2)
        {
            file_3<<"The maximum  x boundary is a surface with the following properties"<<endl;
            file_4>>gas->TSURF[1];
            file_3<<"The temperature of the surface is "<<gas->TSURF[1]<<endl;
            file_4>>gas->FSPEC[1];
            file_3<<"The fraction of specular reflection is "<<gas->FSPEC[1]<<endl;
            file_4>>gas->VSURF[1];
            file_3<<"The velocity in the y direction of this surface is "<<gas->VSURF[1]<<endl;
            // WRITE (3,*) 'The maximum  x boundary is a surface with the following properties'
            // READ (4,*) TSURF(1) //gas->TSURF[1]
            // WRITE (3,*) '     The temperature of the surface is',TSURF(1) //gas->TSURF[1]
            // READ (4,*) FSPEC(1) //gas->FSPEC[1]
            // WRITE (3,*) '     The fraction of specular reflection is',FSPEC(1) //gas->FSPEC[1]
            // READ (4,*) VSURF(1) //gas->VSURF[1]
            // WRITE (3,*) '     The velocity in the y direction of this surface is',VSURF(1) //gas->VSURF[1]
        }
        if(geom->IFX>0)
        {
            file_4>>geom->IWF;
            // READ (4,*) READ (4,*) IWF //geom->IWF
            if(geom->IWF==0)
                file_3<<"There are no radial weighting factors"<<endl;
            // WRITE (3,*) 'There are no radial weighting factors'
            if(geom->IWF==0)
                file_3<<"There are radial weighting factors"<<endl;
            // WRITE (3,*) 'There are radial weighting factors'
            if(geom->IWF==0)
            {
                file_4>>geom->WFM;
                file_3<<"The maximum value of the weighting factor is  "<<geom->WFM<<endl;
                // READ (4,*) WFM //geom->WFM
                // WRITE (3,*) 'The maximum value of the weighting factor is ',WFM //geom->WFM
                geom->WFM=(geom->WFM-1)/geom->XB[2];
            }
        }
        file_4>>calc->IGS;
        // READ (4,*) IGS //calc->IGS
        if(calc->IGS==0)
            file_3<<"The flowfield is initially a vacuum "<<endl;
        // WRITE (3,*) 'The flowfield is initially a vacuum'
        if(calc->IGS==1)
            file_3<<"The flowfield is initially the stream(s) or reference gas"<<endl;
        // WRITE (3,*) 'The flowfield is initially the stream(s) or reference gas'
        file_4>>calc->ISECS;
        // READ (4,*) ISECS //calc->ISECS
        if(calc->ISECS==0)
            file_3<<"There is no secondary stream initially at x > 0"<<endl;
        // WRITE (3,*) 'There is no secondary stream initially at x > 0'
        if(calc->ISECS==1 && geom->IFX==0)
            file_3<<"There is a secondary stream applied initially at x = 0 (XB(2) must be > 0)"<<endl;
        // WRITE (3,*) 'There is a secondary stream applied initially at x = 0 (XB(2) must be > 0)'
        if(calc->ISECS==1 && geom->IFX>0)
        {
            if(geom->IWF==1)
            {
                file_3<<"There cannot be a secondary stream when weighting factors are present"<<endl;
                // WRITE (3,*) 'There cannot be a secondary stream when weighting factors are present'
                return;//STOP//dout
            }
            file_3<<"There is a secondary stream"<<endl;
            // WRITE (3,*) 'There is a secondary stream'
            file_4>>geom->XS;
            // READ (4,*) XS //geom->XS
            file_3<<"The secondary stream boundary is at r= "<<geom->XS<<endl;
            // WRITE (3,*) 'The secondary stream boundary is at r=',XS //geom->XS
        }
        if(calc->ISECS==1)
        {
            file_3<<"The secondary stream (at x>0 or X>XS) properties are:-"<<endl;
            file_4>>gas->FND[2];
            file_3<<"The stream number density is "<<gas->FND[2]<<endl;
            file_4>>gas->FTMP[2];
            file_3<<"The stream temperature is "<<gas->FTMP[2]<<endl;
            // WRITE (3,*) 'The secondary stream (at x>0 or X>XS) properties are:-'
            // READ (4,*) FND(2) //gas->FND
            // WRITE (3,*) '    The stream number density is',FND(2) //gas->FND
            // READ (4,*) FTMP(2) //gas->FTMP
            // WRITE (3,*) '    The stream temperature is',FTMP(2) //gas->FTMP
            if(gas->MMVM>0)
            {
                file_4>>gas->FVTMP[2];
                file_3<<"The stream vibrational and electronic temperature is "<<gas->FVTMP[2]<<endl;
                // READ (4,*) FVTMP(2) //gas->FVTMP[2]
                // WRITE (3,*) '    The stream vibrational and electronic temperature is',FVTMP(2) //gas->FVTMP[2]
            }
            file_4>>gas->VFX[2];
            file_3<<"The stream velocity in the x direction is "<<gas->VFX[2]<<endl;
            file_4>>gas->VFY[2];
            file_3<<"The stream velocity in the y direction is "<<gas->VFY[2]<<endl;
            // READ (4,*) VFX(2) //gas->VFX
            // WRITE (3,*) '    The stream velocity in the x direction is',VFX(2) //gas->VFX
            // READ (4,*) VFY(2) //gas->VFY
            // WRITE (3,*) '    The stream velocity in the y direction is',VFY(2) //gas->VFY
            if(gas->MSP>1)
            {
                for(N=1;N<=gas->MSP;N++)
                {
                    file_4>>gas->FSP[N][2];
                    file_3<<"The fraction of species "<<N<<" is "<<gas->FSP[N][2]<<endl;
                    // READ (4,*) FSP(N,2) //gas->FSP
                    // WRITE (3,*) '    The fraction of species',N,' is',FSP(N,2) //gas->FSP
                }
            }
            else
            {
                gas->FSP[1][2]=1;
            }
        }
        if(geom->IFX==0 && geom->ITYPE[1]==0)
        {
            file_4>>calc->IREM;
            // READ (4,*) IREM //calc->IREM
            if(calc->IREM==0)
            {
                file_3<<"There is no molecule removal"<<endl;
                // WRITE (3,*) 'There is no molecule removal'
                geom->XREM=geom->XB[1]-1.e00;
                geom->FREM=0.e00;
            }
            else if(calc->IREM==1)
            {
                file_4>>geom->XREM;
                file_3<<"There is full removal of the entering (at XB(1)) molecules between "<<geom->XREM<<" and "<<geom->XB[2]<<endl;
                // READ (4,*) XREM //geom->XREM
                // WRITE (3,*) ' There is full removal of the entering (at XB(1)) molecules between',XREM,' and',XB(2) //geom->XREM ,geom->XB[2]
                geom->FREM=1.e00;
            }
            else if(calc->IREM==2)
            {
                file_3<<"Molecule removal is specified whenever the program is restarted"<<endl;
                // WRITE (3,*) ' Molecule removal is specified whenever the program is restarted'
                geom->XREM=geom->XB[1]-1.e00;
                geom->FREM=0.e00;
            }
            else
            {
                geom->XREM=geom->XB[1]-1.e00;
                geom->FREM=0.e00;
            }
        }
        geom->IVB=0;
        geom->VELOB=0.e00;
        if(geom->ITYPE[2]==1)
        {
            file_4>>geom->IVB;
            // READ (4,*) IVB
            if(geom->IVB==0)
                file_3<<"The outer boundary is stationary"<<endl;
            // WRITE (3,*) ' The outer boundary is stationary'
            if(geom->IVB==1)
            {
                file_3<<"The outer boundary moves with a constant speed"<<endl;
                file_4>>geom->VELOB;
                file_3<<" The speed of the outer boundary is "<<geom->VELOB<<endl;
                // WRITE (3,*) ' The outer boundary moves with a constant speed'
                // READ (4,*) VELOB //geom->VELOB
                // WRITE (3,*) ' The speed of the outer boundary is',VELOB //geom->VELOB
            }
        }
        file_4>>calc->MOLSC;
        file_3<<"The desired number of molecules in a sampling cell is "<<calc->MOLSC<<endl;
        // READ (4,*) MOLSC //calc->MOLSC
        // WRITE (3,*) 'The desired number of molecules in a sampling cell is',MOLSC ////calc->MOLSC
    }
    //set the speed of the outer boundary
    file_3.close();
    file_4.close();
    // CLOSE (3)
    // CLOSE (4)
    // set the stream at the maximum x boundary if there is no secondary stream
    if(calc->ISECS==0 && geom->ITYPE[2]==0)
    {
        gas->FND[2]=gas->FND[1];
        gas->FTMP[2]=gas->FTMP[1];
        if(gas->MMVM>0)
            gas->FVTMP[2]=gas->FVTMP[1];
        gas->VFX[2]=gas->VFX[1];
        if(gas->MSP>1)
        {
            for(N=1;N<=gas->MSP;N++)
            {
                gas->FSP[N][2]=gas->FSP[N][1];
            }
        }
        else
            gas->FSP[1][2]=1;
    }
    //dout
    //1234   CONTINUE;
    return;
}

void INITIALISE_SAMPLES()
{
    //start a new sample for all classes of flow
    //CALC calc;
    //GEOM_1D geom;
    //GAS gas;
    //OUTPUT output;
    //MOLECS molecs;
    
    int N;
    //
    output->NSAMP=0.0;
    output->TISAMP=calc->FTIME;
    output->NMISAMP=molecs->NM;
    //memset(output->COLLS,0.e00,sizeof(*output->COLLS));memset(output->WCOLLS,0.e00,sizeof(*output->WCOLLS));memset(output->CLSEP,0.e00,sizeof(*output->CLSEP));
   
    for(int i=0;i<geom->NCELLS+1;i++)
        output->COLLS[i]=0.e00;
    for(int i=0;i<geom->NCELLS+1;i++)
       output->WCOLLS[i]=0.e00;
    for(int i=0;i<geom->NCELLS+1;i++)
        output->CLSEP[i]=0.e00;
    //output->COLLS=0.e00 ; output->WCOLLS=0.e00 ; output->CLSEP=0.e00;
    //memset(calc->TCOL,0.0,sizeof(**calc->TCOL));//calc->TCOL=0.0;
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            calc->TCOL[i][j]=0.0;
        }
    }
    //gas->TREACG=0;
    //gas->TREACL=0;
    for(int i=0;i<5;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->TREACG[i][j]=0;
        }
    }
    for(int i=0;i<5;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->TREACL[i][j]=0;
        }
    }
    //memset(output->CS,0.0,sizeof(***output->CS));memset(output->CSS,0.0,sizeof(****output->CSS));memset(output->CSSS,0.0,sizeof(**output->CSSS));
    for(int j=0;j<gas->MSP+10;j++){
        for(int k=0;k<geom->NCELLS+1;k++){
            for(int l=0;l<gas->MSP+1;l++)
                output->CS[j][k][l]=0.0;
        }
    }
    for(int i=0;i<9;i++){
        for(int j=0;j<3;j++){
            for(int k=0;k<gas->MSP+1;k++){
                for(int l=0;l<3;l++)
                    output->CSS[i][j][k][l]=0.0;
            }
        }
    }
    for(int k=0;k<7;k++){
        for(int l=0;l<3;l++)
            output->CSSS[k][l]=0.0;
    }
    //output->CS=0.0 ; output->CSS=0.0 ; output->CSSS=0.0;
    //memset(output->VIBFRAC,0.e00,sizeof(***output->VIBFRAC));//output->VIBFRAC=0.e00;
    //memset(output->SUMVIB,0.e00,sizeof(**output->SUMVIB));//output->SUMVIB=0.e00;
    for(int j=0;j<gas->MSP+1;j++){
        for(int k=0;k<gas->MMVM+1;k++){
            for(int l=0;l<151;l++)
                output->VIBFRAC[j][k][l]=0.0;
        }
    }
    for(int k=0;k<gas->MSP+1;k++){
        for(int l=0;l<gas->MMVM+1;l++)
            output->SUMVIB[k][l]=0.0;
    }
    
}
////
//
void SET_INITIAL_STATE_1D()
{
    //set the initial state of a homogeneous or one-dimensional flow
    //
    //MOLECS molecs;
    //GEOM_1D geom;
    //GAS gas;
    //CALC calc;
    //OUTPUT output;
    //
    //
    int J,L,K,KK,KN,II,III,INC,NSET,NSC;
    long long N,M;
    double A,B,AA,BB,BBB,SN,XMIN,XMAX,WFMIN,DENG,ELTI,EA,XPREV;
    double DMOM[4];
    double VB[4][3];
    double ROTE[3];
    //
    //NSET the alternative set numbers in the setting of exact initial state
    //DMOM(N) N=1,2,3 for x,y and z momentum sums of initial molecules
    //DENG the energy sum of the initial molecules
    //VB alternative sets of velocity components
    //ROTE alternative sets of rotational energy
    //EA entry area
    //INC counting increment
    //ELTI  initial electronic temperature
    //XPREV the pevious x coordinate
    //
    //memset(DMOM,0.e00,sizeof(DMOM));
    for(int i=0;i<4;i++)
        DMOM[i]=0.e00;
    DENG=0.e00;
    //set the number of molecules, divisions etc. based on stream 1
    //
    calc->NMI=10000*calc->IMEG+2;    //small changes in number for statistically independent runs
    geom->NDIV=calc->NMI/calc->MOLSC; //MOLSC molecules per division
    //WRITE (9,*) 'The number of divisions is',NDIV
    file_9<< "The number of divisions is "<<geom->NDIV<<endl;
    //
    geom->MDIV=geom->NDIV;
    geom->ILEVEL=0;
    //
    geom->i_allocate(geom->ILEVEL+1,geom->MDIV+1,geom->JDIV);
    // ALLOCATE (JDIV(0:ILEVEL,MDIV),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR JDIV ARRAY',ERROR
    // ENDIF
    //
    geom->DDIV=(geom->XB[2]-geom->XB[1])/double(geom->NDIV);
    geom->NCELLS=geom->NDIV;
    
    //WRITE (9,*) 'The number of sampling cells is',NCELLS
    file_9<<"The number of sampling cells is "<< geom->NCELLS<<endl;
    geom->NCIS=calc->MOLSC/calc->NMCC;
    geom->NCCELLS=geom->NCIS*geom->NDIV;
    //WRITE (9,*) 'The number of collision cells is',NCCELLS
    file_9<< "The number of collision cells is "<<geom->NCCELLS<<endl;
    //
    if(geom->IFX == 0) geom->XS=0.e00;
    //
    if(calc->ISECS == 0){
        if(geom->IFX == 0) calc->FNUM=((geom->XB[2]-geom->XB[1])*gas->FND[1])/double(calc->NMI);
        if(geom->IFX == 1) calc->FNUM=PI*(pow(geom->XB[2],2)-pow(geom->XB[1],2))*gas->FND[1]/double(calc->NMI);
        if(geom->IFX == 2) calc->FNUM=1.3333333333333333333333e00*PI*(pow(geom->XB[2],3)-pow(geom->XB[1],3))*gas->FND[1]/double(calc->NMI);
    }
    else{
        if(geom->IFX == 0) calc->FNUM=((geom->XS-geom->XB[1])*gas->FND[1]+(geom->XB[2]-geom->XS)*gas->FND[2])/double(calc->NMI);
        if(geom->IFX == 1) calc->FNUM=PI*((pow(geom->XS,2)-pow(geom->XB[1],2))*gas->FND[1]+(pow(geom->XB[2],2)-pow(geom->XS,2))*gas->FND[2])/double(calc->NMI);
        if(geom->IFX == 2) calc->FNUM=1.3333333333333333333333e00*PI*((pow(geom->XS,3)-pow(geom->XB[1],3))*gas->FND[1]+(pow(geom->XB[2],3)-pow(geom->XS,3))*gas->FND[2])/double(calc->NMI);
    }
    //
    calc->FNUM=calc->FNUM*calc->FNUMF;
    if(calc->FNUM < 1.e00) calc->FNUM=1.e00;
    //
    calc->FTIME=0.e00;
    //
    calc->TOTMOV=0.e00;
    calc->TOTCOL=0.e00;
    
    output->NDISSOC=0;
    //memset(calc->TCOL,0.e00,sizeof(**calc->TCOL));//calc->TCOL=0.e00;
    for(int i=0;i<gas->MSP+1;i++){
        for(int j=0;j<gas->MSP+1;j++){
            calc->TCOL[i][j]=0.e00;
        }
    }
    
    //memset(calc->TDISS,0.e00,sizeof(*calc->TDISS));//calc->TDISS=0.e00;
    //memset(calc->TRECOMB,0.e00,sizeof(*calc->TRECOMB));//calc->TRECOMB=0.e00;
    for(int i=0;i<gas->MSP+1;i++)
        calc->TDISS[i]=0.e00;
    for(int i=0;i<gas->MSP+1;i++)
        calc->TRECOMB[i]=0.e00;
    //gas->TREACG=0;
    //gas->TREACL=0;
    for(int i=0;i<5;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->TREACG[i][j]=0;
        }
    }
    for(int i=0;i<5;i++){
        for(int j=0;j<gas->MSP+1;j++){
            gas->TREACL[i][j]=0;
        }
    }
    //memset(gas->TNEX,0.e00,sizeof(*gas->TNEX));//gas->TNEX=0.e00;
    for(int i=0;i<gas->MEX+1;i++)
        gas->TNEX[i]= 0.e00;
    for(N=1;N<=geom->NDIV;N++){
        geom->JDIV[0][N]=-N;
    }
    
    //
    geom->d_allocate(5,geom->NCELLS+1,geom->CELL);
    geom->i_allocate(geom->NCELLS+1,geom->ICELL);
    geom->d_allocate(6,geom->NCCELLS+1,geom->CCELL);
    geom->i_allocate(4,geom->NCCELLS+1,geom->ICCELL);
    calc->d_allocate(geom->NCCELLS+1,calc->COLL_TOTCOL);
    // ALLOCATE (CELL(4,NCELLS),ICELL(NCELLS),CCELL(5,NCCELLS),ICCELL(3,NCCELLS),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR CELL ARRAYS',ERROR
    // ENDIF
    //
    output->d_allocate(geom->NCELLS+1,output->COLLS);
    output->d_allocate(geom->NCELLS+1,output->WCOLLS);
    output->d_allocate(geom->NCELLS+1,output->CLSEP);
    output->d_allocate(gas->MNSR+1,output->SREAC);
    output->d_allocate(24,geom->NCELLS+1,output->VAR);
    output->d_allocate(13,geom->NCELLS+1,gas->MSP+1,output->VARSP);
    output->d_allocate(36+gas->MSP,3,output->VARS);
    output->d_allocate(10+gas->MSP,geom->NCELLS+1,gas->MSP+1,output->CS);
    output->d_allocate(9,3,gas->MSP+1,3,output->CSS);
    output->d_allocate(7,3,output->CSSS);
    
    // ALLOCATE (COLLS(NCELLS),WCOLLS(NCELLS),CLSEP(NCELLS),SREAC(MNSR),VAR(23,NCELLS),VARSP(0:12,NCELLS,MSP),    &
    //           VARS(0:35+MSP,2),CS(0:9+MSP,NCELLS,MSP),CSS(0:8,2,MSP,2),CSSS(6,2),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR SAMPLING ARRAYS',ERROR
    // ENDIF
    //
    if(gas->MMVM >= 0){
        
        output->d_allocate(gas->MSP+1,gas->MMVM+1,151,output->VIBFRAC);
        output->d_allocate(gas->MSP+1,gas->MMVM+1,output->SUMVIB);
        // ALLOCATE (VIBFRAC(MSP,MMVM,0:150),SUMVIB(MSP,MMVM),STAT=ERROR)
        // IF (ERROR /= 0) THEN
        //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR RECOMBINATION ARRAYS',ERROR
        // ENDIF
    }
    //
    INITIALISE_SAMPLES();
    //
    //Set the initial cells
    
    for(N=1;N<=geom->NCELLS;N++){
        geom->CELL[2][N]=geom->XB[1]+double(N-1)*geom->DDIV;
        geom->CELL[3][N]=geom->CELL[2][N]+geom->DDIV;
        geom->CELL[1][N]=geom->CELL[2][N]+0.5e00*geom->DDIV;
        if(geom->IFX == 0) geom->CELL[4][N]=geom->CELL[3][N]-geom->CELL[2][N];    //calculation assumes unit cross-section
        if(geom->IFX == 1) geom->CELL[4][N]=PI*(pow(geom->CELL[3][N],2)-pow(geom->CELL[2][N],2));  //assumes unit length of full cylinder
        if(geom->IFX == 2) geom->CELL[4][N]=1.33333333333333333333e00*PI*(pow(geom->CELL[3][N],3)-pow(geom->CELL[2][N],3));    //flow is in the full sphere
        geom->ICELL[N]=geom->NCIS*(N-1);
        for(M=1;M<=geom->NCIS;M++){
            L=geom->ICELL[N]+M;
            XMIN=geom->CELL[2][N]+double(M-1)*geom->DDIV/double(geom->NCIS);
            XMAX=XMIN+geom->DDIV/double(geom->NCIS);
            if(geom->IFX == 0) geom->CCELL[1][L]=XMAX-XMIN;
            if(geom->IFX == 1) geom->CCELL[1][L]=PI*(pow(XMAX,2)-pow(XMIN,2));  //assumes unit length of full cylinder
            if(geom->IFX == 2) geom->CCELL[1][L]=1.33333333333333333333e00*PI*(pow(XMAX,3)-pow(XMIN,3));    //flow is in the full sphere
            geom->CCELL[2][L]=0.e00;
            geom->ICCELL[3][L]=N;
        }
        output->VAR[11][N]=gas->FTMP[1];
        output->VAR[8][N]=gas->FTMP[1];
    }
    //
    if(geom->IWF == 0) geom->AWF=1.e00;
    if(geom->IWF == 1){
        //FNUM must be reduced to allow for the weighting factors
        A=0.e00;
        B=0.e00;
        for(N=1;N<=geom->NCELLS;N++){
            A=A+geom->CELL[4][N];
            B=B+geom->CELL[4][N]/(1.0+geom->WFM*pow(geom->CELL[1][N],geom->IFX));
        }
        geom->AWF=A/B;
        calc->FNUM=calc->FNUM*B/A;
    }
    //
    //WRITE (9,*) 'FNUM is',FNUM
    file_9<<"FNUM is "<<calc->FNUM<<endl;
    //
    //set the information on the molecular species
    //
    A=0.e00;
    B=0.e00;
    for(L=1;L<=gas->MSP;L++){
        A=A+gas->SP[5][L]*gas->FSP[L][1];
        B=B+(3.0+gas->ISPR[1][L])*gas->FSP[L][1];
        gas->VMP[L][1]=sqrt(2.e00*BOLTZ*gas->FTMP[1]/gas->SP[5][L]);
        if((geom->ITYPE[2]== 0) || (calc->ISECS == 1)) gas->VMP[L][2]=sqrt(2.e00*BOLTZ*gas->FTMP[2]/gas->SP[5][L]);
        calc->VNMAX[L]=3.0*gas->VMP[L][1];
        if(L == 1)
            gas->VMPM=gas->VMP[L][1];
        else
            if(gas->VMP[L][1] > gas->VMPM) gas->VMPM=gas->VMP[L][1];
    }
    //WRITE (9,*) 'VMPM =',VMPM
    file_9<< "VMPM = "<<gas->VMPM<<endl;
    gas->FDEN=A*gas->FND[1];
    gas->FPR=gas->FND[1]*BOLTZ*gas->FTMP[1];
    gas->FMA=gas->VFX[1]/sqrt((B/(B+2.e00))*BOLTZ*gas->FTMP[1]/A);
    //set the molecular properties for collisions between unlike molecles
    //to the average of the molecules
    for(L=1;L<=gas->MSP;L++){
        for(M=1;M<=gas->MSP;M++){
            gas->SPM[4][L][M]=0.5e00*(gas->SP[1][L]+gas->SP[1][M]);
            gas->SPM[3][L][M]=0.5e00*(gas->SP[3][L]+gas->SP[3][M]);
            gas->SPM[5][L][M]=0.5e00*(gas->SP[2][L]+gas->SP[2][M]);
            gas->SPM[1][L][M]=gas->SP[5][L]*(gas->SP[5][M]/(gas->SP[5][L]+gas->SP[5][M]));
            gas->SPM[2][L][M]=0.25e00*PI*pow((gas->SP[1][L]+gas->SP[1][M]),2);
            AA=2.5e00-gas->SPM[3][L][M];
            A=tgamma(AA);
            gas->SPM[6][L][M]=1.e00/A;
            gas->SPM[8][L][M]=0.5e00*(gas->SP[4][L]+gas->SP[4][M]);
            if((gas->ISPR[1][L] > 0) && (gas->ISPR[1][M] > 0))
                gas->SPM[7][L][M]=(gas->SPR[1][L]+gas->SPR[1][M])*0.5e00;
            if((gas->ISPR[1][L] > 0) && (gas->ISPR[1][M] == 0))
                gas->SPM[7][L][M]=gas->SPR[1][L];
            if((gas->ISPR[1][M] > 0) && (gas->ISPR[1][L] == 0))
                gas->SPM[7][L][M]=gas->SPR[1][M];
        }
    }
    if(gas->MSP == 1){   //set unscripted variables for the simple gas case
        gas->RMAS=gas->SPM[1][1][1];
        gas->CXSS=gas->SPM[2][1][1];
        gas->RGFS=gas->SPM[6][1][1];
    }
    //
    for(L=1;L<=gas->MSP;L++){
        gas->CR[L]=0.e00;
        for(M=1;M<=gas->MSP;M++){   //set the equilibrium collision rates
            gas->CR[L]=gas->CR[L]+2.e00*SPI*pow(gas->SPM[4][L][M],2)*gas->FND[1]*gas->FSP[M][1]*pow((gas->FTMP[1]/gas->SPM[5][L][M]),(1.0-gas->SPM[3][L][M]))*sqrt(2.0*BOLTZ*gas->SPM[5][L][M]/gas->SPM[1][L][M]);
        }
    }
    A=0.e00;
    for(L=1;L<=gas->MSP;L++)
        A=A+gas->FSP[L][1]*gas->CR[L];
    gas->CTM=1.e00/A;
    //WRITE (9,*) 'Collision time in the stream is',CTM
    file_9<< "Collision time in the stream is "<<gas->CTM;
    //
    for(L=1;L<=gas->MSP;L++){
        gas->FP[L]=0.e00;
        for(M=1;M<=gas->MSP;M++){
            gas->FP[L]=gas->FP[L]+PI*pow(gas->SPM[4][L][M],2)*gas->FND[1]*gas->FSP[M][1]*pow((gas->FTMP[1]/gas->SPM[5][L][M]),(1.0-gas->SPM[3][L][M]))*sqrt(1.e00+gas->SP[5][L]/gas->SP[5][M]);
        }
        gas->FP[L]=1.e00/gas->FP[L];
    }
    gas->FPM=0.e00;
    for(L=1;L<=gas->MSP;L++)
        gas->FPM=gas->FPM+gas->FSP[L][1]*gas->FP[L];
    //WRITE (9,*) 'Mean free path in the stream is',FPM
    file_9<<"Mean free path in the stream is "<<gas->FPM<<endl;
    //
    calc->TNORM=gas->CTM;
    if(calc->ICLASS == 1) calc->TNORM= (geom->XB[2]-geom->XB[1])/gas->VMPM;     //there may be alternative definitions
    //
    //set the initial time step
    calc->DTM=gas->CTM*calc->CPDTM;
    //
    if(fabs(gas->VFX[1]) > 1.e-6)
        A=(0.5e00*geom->DDIV/gas->VFX[1])*calc->TPDTM;
    else
        A=0.5e00*geom->DDIV/gas->VMPM;
    
    if(geom->IVB == 1){
        B=0.25e00*geom->DDIV/(fabs(geom->VELOB)+gas->VMPM);
        if(B < A) A=B;
    }
    if(calc->DTM > A) calc->DTM=A;
    //
    calc->DTM=0.1e00*calc->DTM;   //OPTIONAL MANUAL ADJUSTMENT that is generally used with a fixed time step (e.g for making x-t diagram)
    //
    calc->DTSAMP=calc->SAMPRAT*calc->DTM;
    calc->DTOUT=calc->OUTRAT*calc->DTSAMP;
    calc->TSAMP=calc->DTSAMP;
    calc->TOUT=calc->DTOUT;
    calc->ENTMASS=0.0;
    //
    //WRITE (9,*) 'The initial value of the overall time step is',DTM
    file_9<< "The initial value of the overall time step is "<<calc->DTM<<endl;
    //
    //initialise cell quantities associated with collisions
    //
    for(N=1;N<=geom->NCCELLS;N++){
        geom->CCELL[3][N]=calc->DTM/2.e00;
        geom->CCELL[4][N]=2.e00*gas->VMPM*gas->SPM[2][1][1];
        calc->RANF=((double)rand()/(double)RAND_MAX);
        // RANDOM_NUMBER(RANF)
        geom->CCELL[2][N]=calc->RANF;
        geom->CCELL[5][N]=0.e00;
    }
    //
    //set the entry quantities
    //
    for(K=1;K<=2;K++){
        if((geom->ITYPE[K] == 0) || ((K == 2) && (geom->ITYPE[K] == 4))){
            if(geom->IFX == 0) EA=1.e00;
            if(geom->IFX == 1) EA=2.e00*PI*geom->XB[K];
            if(geom->IFX == 2) EA=4.e00*PI*pow(geom->XB[K],2);
            for(L=1;L<=gas->MSP;L++){
                if(K == 1) SN=gas->VFX[1]/gas->VMP[L][1];
                if(K == 2) SN=-gas->VFX[2]/gas->VMP[L][2];
                AA=SN;
                A=1.e00+erf(AA);
                BB=exp(-pow(SN,2));
                gas->ENTR[3][L][K]=SN;
                gas->ENTR[4][L][K]=SN+sqrt(pow(SN,2)+2.e00);
                gas->ENTR[5][L][K]=0.5e00*(1.e00+SN*(2.e00*SN-gas->ENTR[4][L][K]));
                gas->ENTR[6][L][K]=3.e00*gas->VMP[L][K];
                B=BB+SPI*SN*A;
                gas->ENTR[1][L][K]=EA*gas->FND[K]*gas->FSP[L][K]*gas->VMP[L][K]*B/(calc->FNUM*2.e00*SPI);
                gas->ENTR[2][L][K]=0.e00;
            }
        }
    }
    //
    //Set the uniform stream
    //
    molecs->MNM=1.1e00*calc->NMI;
    //
    if(gas->MMVM > 0){
        molecs->d_allocate(calc->NCLASS+1,molecs->MNM+1,molecs->PX);
        molecs->d_allocate(molecs->MNM+1,molecs->PTIM);
        molecs->d_allocate(molecs->MNM+1,molecs->PROT);
        molecs->i_allocate(molecs->MNM+1,molecs->IPCELL);
        molecs->i_allocate(molecs->MNM+1,molecs->IPSP);
        molecs->i_allocate(molecs->MNM+1,molecs->ICREF);
        molecs->i_allocate(molecs->MNM+1,molecs->IPCP);
        molecs->d_allocate(4,molecs->MNM+1,molecs->PV);
        molecs->i_allocate(gas->MMVM+1,molecs->MNM+1,molecs->IPVIB);
        molecs->d_allocate(molecs->MNM+1,molecs->PELE);
        // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),PROT(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),     &
        //      IPVIB(MMVM,MNM),PELE(MNM),STAT=ERROR)
    }
    
    else{
        if(gas->MMRM > 0){
            molecs->d_allocate(calc->NCLASS+1,molecs->MNM+1,molecs->PX);
            molecs->d_allocate(molecs->MNM+1,molecs->PTIM);
            molecs->d_allocate(molecs->MNM+1,molecs->PROT);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCELL);
            molecs->i_allocate(molecs->MNM+1,molecs->IPSP);
            molecs->i_allocate(molecs->MNM+1,molecs->ICREF);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCP);
            molecs->d_allocate(4,molecs->MNM+1,molecs->PV);
            molecs->d_allocate(molecs->MNM+1,molecs->PELE);
            // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),PROT(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),PELE(MNM),STAT=ERROR)
        }
        else{
            molecs->d_allocate(calc->NCLASS+1,molecs->MNM+1,molecs->PX);
            molecs->d_allocate(molecs->MNM+1,molecs->PTIM);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCELL);
            molecs->i_allocate(molecs->MNM+1,molecs->IPSP);
            molecs->i_allocate(molecs->MNM+1,molecs->ICREF);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCP);
            molecs->d_allocate(4,molecs->MNM+1,molecs->PV);
            molecs->d_allocate(molecs->MNM+1,molecs->PELE);
            // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),PELE(MNM),STAT=ERROR)
        }
    }
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR MOLECULE ARRAYS',ERROR
    // ENDIF
    //
    molecs->NM=0;
    if(calc->IGS == 1){
        cout<<"Setting the initial gas"<<endl;
        for(L=1;L<=gas->MSP;L++){
            //memset(ROTE,0.0,sizeof(ROTE));
            for(int i=0;i<3;i++)
                ROTE[i]=0.0;
            for(K=1;K<=calc->ISECS+1;K++){
                if(calc->ISECS == 0){         //no secondary stream
                    M=(double(calc->NMI)*gas->FSP[L][1]*geom->AWF);
                    XMIN=geom->XB[1];
                    XMAX=geom->XB[2];
                }
                else{
                    A=(pow(geom->XS,geom->JFX)-pow(geom->XB[1],geom->JFX))*gas->FND[1]+(pow(geom->XB[2],geom->JFX)-pow(geom->XS,geom->JFX))*gas->FND[2];
                    if(K == 1){
                        M=int(double(calc->NMI)*((pow(geom->XS,geom->JFX)-pow(geom->XB[1],geom->JFX))*gas->FND[1]/A)*gas->FSP[L][1]);
                        XMIN=geom->XB[1];
                        XMAX=geom->XS;
                    }
                    else{
                        M=int(double(calc->NMI)*((pow(geom->XB[2],geom->JFX)-pow(geom->XS,geom->JFX))*gas->FND[2]/A)*gas->FSP[L][2]);
                        XMIN=geom->XS;
                        XMAX=geom->XB[2];
                    }
                }
                if((K == 1) || (calc->ISECS == 1)){
                    III=0;
                    WFMIN=1.e00+geom->WFM*pow(geom->XB[1],geom->IFX);
                    N=1;
                    INC=1;
                    if((K== 2) && (geom->JFX > 1)){
                        BBB=(pow(XMAX,geom->JFX)-pow(XMIN,geom->JFX))/double(M);
                        XPREV=XMIN;
                    }
                    while(N < M){
                        if((geom->JFX == 1) || (K == 1))
                            A=pow((pow(XMIN,geom->JFX)+((double(N)-0.5e00)/double(M))*pow((XMAX-XMIN),geom->JFX)),(1.e00/double(geom->JFX)));
                        else{
                            A=pow((pow(XPREV,geom->JFX)+BBB),(1.e00/double(geom->JFX)));
                            XPREV=A;
                        }
                        if(geom->IWF == 0)
                            B=1.e00;
                        else{
                            B=WFMIN/(1.e00+geom->WFM*pow(A,geom->IFX));
                            if((B < 0.1e00) && (INC == 1)) INC=10;
                            if((B < 0.01e00) && (INC == 10)) INC=100;
                            if((B < 0.001e00) && (INC == 100)) INC=1000;
                            if((B < 0.0001e00) && (INC == 1000)) INC=10000;
                        }
                        calc->RANF=((double)rand()/(double)RAND_MAX);
                        // CALL RANDOM_NUMBER(RANF)
                        if(B*double(INC) > calc->RANF){
                            molecs->NM=molecs->NM+1;
                            molecs->PX[1][molecs->NM]=A;
                            molecs->IPSP[molecs->NM]=L;
                            molecs->PTIM[molecs->NM]=0.0;
                            if(geom->IVB == 0) FIND_CELL_1D(molecs->PX[1][molecs->NM],molecs->IPCELL[molecs->NM],KK);
                            if(geom->IVB == 1) FIND_CELL_MB_1D(molecs->PX[1][molecs->NM],molecs->IPCELL[molecs->NM],KK,molecs->PTIM[molecs->NM]);
                            //
                            for(NSET=1;NSET<=2;NSET++){
                                for(KK=1;KK<=3;KK++){
                                    RVELC(A,B,gas->VMP[L][K]);
                                    if(A < B){
                                        if(DMOM[KK] < 0.e00)
                                            BB=B;
                                        else
                                            BB=A;
                                    }           
                                    else{
                                        if(DMOM[KK] < 0.e00)
                                            BB=A;
                                        else
                                            BB=B;
                                    }
                                    VB[KK][NSET]=BB;
                                }
                                if(gas->ISPR[1][L] > 0) SROT(L,gas->FTMP[K],ROTE[NSET]);
                            }
                            A=(0.5e00*gas->SP[5][L]*(pow(VB[1][1],2)+pow(VB[2][1],2)+pow(VB[3][1],2))+ROTE[1])/(0.5e00*BOLTZ*gas->FTMP[K])-3.e00-double(gas->ISPR[1][L]);
                            B=(0.5e00*gas->SP[5][L]*(pow(VB[1][2],2)+pow(VB[2][2],2)+pow(VB[3][2],2))+ROTE[2])/(0.5e00*BOLTZ*gas->FTMP[K])-3.e00-double(gas->ISPR[1][L]);
                            if(A < B){
                                if(DENG < 0.e00)
                                    KN=2;
                                else
                                    KN=1;
                            }
                            else{
                                if(DENG < 0.e00)
                                    KN=1;
                                else
                                    KN=2;
                            }
                            
                            for(KK=1;KK<=3;KK++){
                                molecs->PV[KK][molecs->NM]=VB[KK][KN];
                                DMOM[KK]=DMOM[KK]+VB[KK][KN];
                            }
                            molecs->PV[1][molecs->NM]=molecs->PV[1][molecs->NM]+gas->VFX[K];
                            molecs->PV[2][molecs->NM]=molecs->PV[2][molecs->NM]+gas->VFY[K];
                            if(gas->ISPR[1][L] > 0) molecs->PROT[molecs->NM]=ROTE[KN];
                            //           PROT(NM)=0.d00       //uncomment for zero initial rotational temperature (Figs. 6.1 and 6.2)
                            if(KN == 1) DENG=DENG+A;
                            if(KN == 2) DENG=DENG+B;
                            if(gas->MMVM > 0){
                                if(gas->ISPV[L] > 0){
                                    for(J=1;J<=gas->ISPV[L];J++)
                                        SVIB(L,gas->FVTMP[K],molecs->IPVIB[J][molecs->NM],J);
                                }
                                ELTI=gas->FVTMP[K];
                                if(gas->MELE > 1) SELE(L,ELTI,molecs->PELE[molecs->NM]);
                            }
                        }
                        N=N+INC;
                    }
                }
            }
        }
        //
        //WRITE (9,*) 'DMOM',DMOM
        //WRITE (9,*) 'DENG',DENG
        file_9<<"DMOM "<<DMOM<<endl;
        file_9<<"DENG "<<DENG<<endl;
    }
    //
    calc->NMI=molecs->NM;
    //
    
    //SPECIAL CODING FOR INITIATION OF COMBUSION IN H2-02 MIXTURE (FORCED IGNITION CASES in section 6.7)
    //set the vibrational levels of A% random molecules to 5
    //  A=0.05D00
    //  M=0.01D00*A*NM
    //  DO N=1,M
    //    CALL RANDOM_NUMBER(RANF)
    //    K=INT(RANF*DFLOAT(NM))+1
    //    IPVIB(1,K)=5
    //  END DO
    //
    SAMPLE_FLOW();
    OUTPUT_RESULTS();
    calc->TOUT=calc->TOUT-calc->DTOUT;
    return;
}

void MOLECULES_ENTER_1D()
{
    //molecules enter boundary at XB(1) and XB(2) and may be removed behind a wave
    //MOLECS molecs;
    //GAS gas;
    //CALC calc;
    //GEOM_1D geom;
    //OUTPUT output;
    //
    int K,L,M,N,NENT,II,J,JJ,KK,NTRY;
    double A,B,AA,BB,U,VN,XI,X,DX,DY,DZ;
    //
    //NENT number to enter in the time step
    //
    calc->ENTMASS=0.e00;
    //
    for(J=1;J<=2;J++){     //J is the end
        if((geom->ITYPE[J] == 0) || (geom->ITYPE[J] == 4)){
            KK=1;//the entry surface will normally use the reference gas (main stream) properties
            if((J == 2) && (calc->ISECS == 1) && (geom->XB[2] > 0.e00)) KK=2;    //KK is 1 for reference gas 2 for the secondary stream
            for(L=1;L<=gas->MSP;L++){
                A=gas->ENTR[1][L][J]*calc->DTM+gas->ENTR[2][L][J];
                if((geom->ITYPE[2] == 4) && (calc->ICN == 1)){
                    NENT=A;
                    if(J == 1) calc->EME[L]=NENT;
                    if(J == 2) {
                        A=calc->ALOSS[L]-calc->EME[L]-calc->AJM[L];
                        calc->AJM[L]=0.e00;
                        if(A < 0.e00){
                            calc->AJM[L]=-A;
                            A=0.e00;
                        }
                    }
                }
                NENT=A;
                gas->ENTR[2][L][J]=A-NENT;
                if((geom->ITYPE[2] == 4) && (J == 2) && (calc->ICN == 1)) gas->ENTR[2][L][J]=0.e00;
                if(NENT > 0){
                    for(M=1;M<=NENT;M++){
                        if(molecs->NM >= molecs->MNM){
                            cout<< "EXTEND_MNM from MOLECULES_ENTER "<<endl;
                            EXTEND_MNM(1.1);
                        }
                        molecs->NM=molecs->NM+1;
                        AA=max(0.e00,gas->ENTR[3][L][J]-3.e00);
                        BB=max(3.e00,gas->ENTR[3][L][J]+3.e00);
                        II=0;
                        while(II == 0){
                            calc->RANF=((double)rand()/(double)RAND_MAX);
                            // CALL RANDOM_NUMBER(RANF)
                            B=AA+(BB-AA)*calc->RANF;
                            U=B-gas->ENTR[3][L][J];
                            A=(2.e00*B/gas->ENTR[4][L][J])*exp(gas->ENTR[5][L][J]-U*U);
                            calc->RANF=((double)rand()/(double)RAND_MAX);
                            // CALL RANDOM_NUMBER(RANF)
                            if(A > calc->RANF) II=1;
                        }
                        molecs->PV[1][molecs->NM]=B*gas->VMP[L][KK];
                        if(J == 2) molecs->PV[1][molecs->NM]=-molecs->PV[1][molecs->NM];
                        //
                        RVELC(molecs->PV[2][molecs->NM],molecs->PV[3][molecs->NM],gas->VMP[L][KK]);
                        molecs->PV[2][molecs->NM]=molecs->PV[2][molecs->NM]+gas->VFY[J];
                        //
                        if(gas->ISPR[1][L] > 0) SROT(L,gas->FTMP[KK],molecs->PROT[molecs->NM]);
                        //
                        if(gas->MMVM > 0){
                            for(K=1;K<=gas->ISPV[L];K++)
                                SVIB(L,gas->FVTMP[KK],molecs->IPVIB[K][molecs->NM],K);
                        }
                        if(gas->MELE > 1) SELE(L,gas->FTMP[KK],molecs->PELE[molecs->NM]);
                        //
                        if(molecs->PELE[molecs->NM] > 0.e00)
                            continue;                     //DEBUG
                        //
                        molecs->IPSP[molecs->NM]=L;
                        //advance the molecule into the flow
                        calc->RANF=((double)rand()/(double)RAND_MAX);
                        // CALL RANDOM_NUMBER(RANF)
                        XI=geom->XB[J];
                        DX=calc->DTM*calc->RANF*molecs->PV[1][molecs->NM];
                        if((geom->IFX == 0) || (J == 2)) X=XI+DX;
                        if(J == 1){   //1-D move at outer boundary so molecule remains in flow
                            if(geom->IFX > 0) DY=calc->DTM*calc->RANF*molecs->PV[2][molecs->NM];
                            DZ=0.e00;
                            if(geom->IFX == 2) DZ=calc->DTM*calc->RANF*molecs->PV[3][molecs->NM];
                            if(geom->IFX > 0) AIFX(XI,DX,DY,DZ,X,molecs->PV[1][molecs->NM],molecs->PV[2][molecs->NM],molecs->PV[3][molecs->NM]);
                        }
                        molecs->PX[calc->NCLASS][molecs->NM]=X;
                        molecs->PTIM[molecs->NM]=calc->FTIME;
                        if(geom->IVB == 0) FIND_CELL_1D(molecs->PX[calc->NCLASS][molecs->NM],molecs->IPCELL[molecs->NM],JJ);
                        if(geom->IVB == 1) FIND_CELL_MB_1D(molecs->PX[calc->NCLASS][molecs->NM],molecs->IPCELL[molecs->NM],JJ,molecs->PTIM[molecs->NM]);
                        molecs->IPCP[molecs->NM]=0;
                        if(geom->XREM > geom->XB[1]) calc->ENTMASS=calc->ENTMASS+gas->SP[5][L];
                    }
                }
            }
            if((geom->ITYPE[2] == 4) && (J==2) && (molecs->NM != calc->NMP) && (calc->ICN == 1))
                continue;
        }
    }
    //
    //stagnation streamline molecule removal
    if(geom->XREM > geom->XB[1]){
        calc->ENTMASS=geom->FREM*calc->ENTMASS;
        NTRY=0;
        calc->ENTMASS=calc->ENTMASS+calc->ENTREM;
        while((calc->ENTMASS > 0.e00) && (NTRY < 10000)){
            NTRY=NTRY+1;
            if(NTRY == 10000){
                cout<<"Unable to find molecule for removal"<<endl;
                calc->ENTMASS=0.e00;
                //memset(calc->VNMAX,0.e00,sizeof(*calc->VNMAX));//calc->VNMAX=0.e00;
                for(int i=0;i<gas->MSP+1;i++)
                    calc->VNMAX[i]=0.e00;
            }
            calc->RANF=((double)rand()/(double)RAND_MAX);
            // CALL RANDOM_NUMBER(RANF)
            N=molecs->NM*calc->RANF+0.9999999e00;
            if(molecs->PX[calc->NCLASS][N] > geom->XREM){
                // CALL RANDOM_NUMBER(RANF)
                calc->RANF=((double)rand()/(double)RAND_MAX);
                //IF (RANF < ((PX(N)-XREM)/(XB(2)-XREM))**2) THEN
                if(fabs(gas->VFY[1]) < 1.e-3)
                    VN=sqrt(molecs->PV[2][N]*molecs->PV[2][N]+molecs->PV[3][N]*molecs->PV[3][N]);   //AXIALLY SYMMETRIC STREAMLINE
                else
                    VN=fabs(molecs->PV[3][N]);   //TWO-DIMENSIONAL STREAMLINE
                
                L=molecs->IPSP[N];
                if(VN > calc->VNMAX[L]) calc->VNMAX[L]=VN;
                // CALL RANDOM_NUMBER(RANF)
                calc->RANF=((double)rand()/(double)RAND_MAX);
                if(calc->RANF < VN/calc->VNMAX[L]){
                    REMOVE_MOL(N);
                    calc->ENTMASS=calc->ENTMASS-gas->SP[5][L];
                    NTRY=0;
                }
                //END IF
            }
        }
        calc->ENTREM=calc->ENTMASS;
    }
}

void FIND_CELL_1D(double &X,int &NCC,int &NSC)
{
    //find the collision and sampling cells at a givem location in a 0D or 1D case
    //MOLECS molecs;
    //GEOM_1D geom;
    //CALC calc;
    
    int N,L,M,ND;
    double FRAC,DSC;
    //
    //NCC collision cell number
    //NSC sampling cell number
    //X location
    //ND division number
    //DSC the ratio of the sub-division width to the division width
    //
    ND=(X-geom->XB[1])/geom->DDIV+0.99999999999999e00;
    //
    if(geom->JDIV[0][ND] < 0){    //the division is a level 0 (no sub-division) sampling cell
        NSC=-geom->JDIV[0][ND];
        //  IF (IFX == 0)
        NCC=geom->NCIS*(X-geom->CELL[2][NSC])/(geom->CELL[3][NSC]-geom->CELL[2][NSC])+0.9999999999999999e00;
        NCC=NCC+geom->ICELL[NSC];
        //  IF (NCC == 0) NCC=1
        return;
    }
    else{  //the molecule is in a subdivided division
        FRAC=(X-geom->XB[1])/geom->DDIV-double(ND-1);
        M=ND;
        for(N=1;N<=geom->ILEVEL;N++){
            DSC=1.e00/double(N+1);
            for(L=1;L<=2;L++){  //over the two level 1 subdivisions
                if(((L == 1) && (FRAC < DSC)) || ((L == 2) || (FRAC >= DSC))){
                    M=geom->JDIV[N-1][M]+L;  //the address in JDIV
                    if(geom->JDIV[N][M] < 0){
                        NSC=-geom->JDIV[N][M];
                        NCC=geom->NCIS*(X-geom->CELL[2][NSC])/(geom->CELL[3][NSC]-geom->CELL[2][NSC])+0.999999999999999e00;
                        if(NCC == 0) NCC=1;
                        NCC=NCC+geom->ICELL[NSC];
                        return;
                    }
                }
            }
            FRAC=FRAC-DSC;
        }
    }
    file_9<<"No cell for molecule at x= "<<X<<endl;
    return ;
}

void FIND_CELL_MB_1D(double &X,int &NCC,int &NSC,double &TIM)
{
    //find the collision and sampling cells at a givem location in a 0D or 1D case
    //when there is a moving boundary
    //MOLECS molecs;
    //GEOM_1D geom;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int N,L,M,ND;
    double FRAC,DSC,A,B,C;
    //
    //NCC collision cell number
    //NSC sampling cell number
    //X location
    //ND division number
    //DSC the ratio of the sub-division width to the division width
    //TIM the time
    //
    A=(geom->XB[2]+geom->VELOB*TIM-geom->XB[1])/double(geom->NDIV);      //new DDIV
    ND=(X-geom->XB[1])/A+0.99999999999999e00;
    B=geom->XB[1]+double(ND-1)*A;
    //
    //the division is a level 0 sampling cell
    NSC=-geom->JDIV[0][ND];
    NCC=geom->NCIS*(X-B)/A+0.99999999999999e00;
    NCC=NCC+geom->ICELL[NSC];
    
    //WRITE (9,*) 'No cell for molecule at x=',X
    file_9<< "No cell for molecule at x= "<<X<<endl;
    return;
    //return ;
    //
}

void RVELC(double &U,double &V,double &VMP)
{
    //CALC calc;
    //generates two random velocity components U and V in an equilibrium
    //gas with most probable speed VMP
    //based on equations (4.4) and (4.5)
    double A,B;
    //
    // CALL RANDOM_NUMBER(RANF)
    calc->RANF=((double)rand()/(double)RAND_MAX);
    A=sqrt(-log(calc->RANF));
    // CALL RANDOM_NUMBER(RANF)
    calc->RANF=((double)rand()/(double)RAND_MAX);
    B=DPI*calc->RANF;
    U=A*sin(B)*VMP;
    V=A*cos(B)*VMP;
    return;
}

void SROT(int &L,double &TEMP,double &ROTE)
{
    //sets a typical rotational energy ROTE of species L
    //CALC calc;
    //GAS gas;
    //
    // IMPLICIT NONE
    //
    int I;
    double A,B,ERM;
    //
    if(gas->ISPR[1][L] == 2){
        // CALL RANDOM_NUMBER(RANF)
        calc->RANF=((double)rand()/(double)RAND_MAX);
        ROTE=-log(calc->RANF)*BOLTZ*TEMP;   //equation (4.8)
    }
    else{
        A=0.5e00*gas->ISPR[1][L]-1.e00;
        I=0;
        while(I == 0){
            // CALL RANDOM_NUMBER(RANF)
            calc->RANF=((double)rand()/(double)RAND_MAX);
            ERM=calc->RANF*10.e00;
            //there is an energy cut-off at 10 kT
            B=(pow((ERM/A),A))*exp(A-ERM);      //equation (4.9)
            // CALL RANDOM_NUMBER(RANF)
            calc->RANF=((double)rand()/(double)RAND_MAX);
            if(B > calc->RANF) I=1;
        }
        ROTE=ERM*BOLTZ*TEMP;
    }
    return;
}

void SVIB(int &L,double &TEMP,int &IVIB, int &K)
{
    //sets a typical vibrational state at temp. TEMP of mode K of species L
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int N;
    //    double TEMP;
    //    int IVIB;
    //
    // CALL RANDOM_NUMBER(RANF)
    calc->RANF=((double)rand()/(double)RAND_MAX);
    N=-log(calc->RANF)*TEMP/gas->SPVM[1][K][L];                 //eqn(4.10)
    //the state is truncated to an integer
    IVIB=N;
}

void SELE(int &L,double &TEMP, double &ELE)
{
    //sets a typical electronic energy at temp. TEMP of species L
    //employs direct sampling from the Boltzmann distribution
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int K,N;
    double EPF,A,B;
    double CTP[20];
    //
    //ELE electronic energy of a molecule
    //EPF electronic partition function
    //CTP(N) contribution of electronic level N to the electronic partition function
    //
    if(TEMP > 0.1){
        EPF=0.e00;
        for(N=1;N<=gas->NELL[L];N++)
            EPF=EPF+gas->QELC[1][N][L]*exp(-gas->QELC[2][N][L]/TEMP) ;
        //
        // CALL RANDOM_NUMBER(RANF)
        calc->RANF=((double)rand()/(double)RAND_MAX);
        //
        A=0.0;
        K=0; //becomes 1 when the energy is set
        N=0;  //level
        while(K == 0){
            N=N+1;
            A=A+gas->QELC[1][N][L]*exp(-gas->QELC[2][N][L]/TEMP);
            B=A/EPF;
            if(calc->RANF < B){
                K=1;
                ELE=BOLTZ*gas->QELC[2][N][L];
            }
        }
    }
    else
        ELE=0.e00;
    
    //
}

void CQAX(double &A,double &X,double &GAX)
{
    //calculates the function Q(a,x)=Gamma(a,x)/Gamma(a)
    //
    // IMPLICIT NONE
    double G,DT,T,PV,V;
    int NSTEP,N;
    //
    G=tgamma(A);
    //
    if(X < 10.e00){       //direct integration
        NSTEP=100000;
        DT=X/double(NSTEP);
        GAX=0.e00;
        PV=0.e00;
        for(N=1;N<=NSTEP;N++){
            T=double(N)*DT;
            V=exp(-T)*pow(T,(A-1));
            GAX=GAX+(PV+V)*DT/2.e00;
            PV=V;
        }
        GAX=1.e00-GAX/G;
    }
    else{      //asymptotic formula
        GAX=pow(X,(A-1.e00))*exp(-X)*(1.0+(A-1.e00)/X+(A-1.e00)*(A-2.e00)/pow(X,2)+(A-1.e00)*(A-2.e00)*(A-3.e00)/pow(X,3)+(A-1.e00)*(A-2.e00)*(A-3.e00)*(A-4.e00)/pow(X,4));
        GAX=GAX/G;
    }
    //
    return;
}
//*****************************************************************************
//
void LBS(double XMA,double XMB,double &ERM)
{
    //selects a Larsen-Borgnakke energy ratio using eqn (11.9)
    //
    double PROB,RANF;
    int I,N;
    //
    //I is an indicator
    //PROB is a probability
    //ERM ratio of rotational to collision energy
    //XMA degrees of freedom under selection-1
    //XMB remaining degrees of freedom-1
    //
    I=0;
    while(I == 0){
        // CALL RANDOM_NUMBER(RANF)
        RANF=((double)rand()/(double)RAND_MAX);
        ERM=RANF;
        if((XMA < 1.e-6) || (XMB < 1.e-6)){
            //    IF (XMA < 1.E-6.AND.XMB < 1.E-6) RETURN
            //above can never occur if one mode is translational
            if(XMA < 1.e-6) PROB=pow((1.e00-ERM),XMB);
            if(XMB < 1.e-6) PROB=pow((1.e00-ERM),XMA);
        }
        else
            PROB=pow(((XMA+XMB)*ERM/XMA),XMA)*pow(((XMA+XMB)*(1.e00-ERM)/XMB),XMB);
        
        // CALL RANDOM_NUMBER(RANF)
        RANF=((double)rand()/(double)RAND_MAX);
        if(PROB > RANF) I=1;
    }
    //
    return;
}

void REFLECT_1D(int &N,int J,double &X)
{
    //reflects molecule N and samples the surface J properties
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    int L,K,M;
    double A,B,VMPS,DTR,XI,DX,DY,DZ,WF;
    //
    //VMPS most probable velocity at the surface temperature
    //DTR time remaining after molecule hits a surface
    //
    L=molecs->IPSP[N];
    WF=1.e00;
    if(geom->IWF == 1) WF=1.e00+geom->WFM*pow(X,geom->IFX);
    output->CSS[0][J][L][1]=output->CSS[0][J][L][1]+1.e00;
    output->CSS[1][J][L][1]=output->CSS[1][J][L][1]+WF;
    output->CSS[2][J][L][1]=output->CSS[2][J][L][1]+WF*molecs->PV[1][N]*gas->SP[5][L];
    output->CSS[3][J][L][1]=output->CSS[3][J][L][1]+WF*(molecs->PV[2][N]-gas->VSURF[J])*gas->SP[5][L];
    output->CSS[4][J][L][1]=output->CSS[4][J][L][1]+WF*molecs->PV[3][N]*gas->SP[5][L];
    A=pow(molecs->PV[1][N],2)+pow((molecs->PV[2][N]-gas->VSURF[J]),2)+pow(molecs->PV[3][N],2);
    output->CSS[5][J][L][1]=output->CSS[5][J][L][1]+WF*0.5e00*gas->SP[5][L]*A;
    if(gas->ISPR[1][L] > 0) output->CSS[6][J][L][1]=output->CSS[6][J][L][1]+WF*molecs->PROT[N];
    if(gas->MELE > 1) output->CSS[8][J][L][1]=output->CSS[8][J][L][1]+WF*molecs->PELE[N];
    if(gas->MMVM > 0){
        if(gas->ISPV[L] > 0){
            for(K=1;K<=gas->ISPV[L];K++)
                output->CSS[7][J][L][1]=output->CSS[7][J][L][1]+WF*double(molecs->IPVIB[K][N])*BOLTZ*gas->SPVM[1][K][L];
        }
    }
    A=pow(molecs->PV[1][N],2)+pow(molecs->PV[2][N],2)+pow(molecs->PV[3][N],2);
    B=fabs(molecs->PV[1][N]);
    output->CSSS[1][J]=output->CSSS[1][J]+WF/B;
    output->CSSS[2][J]=output->CSSS[2][J]+WF*gas->SP[5][L]/B;
    output->CSSS[3][J]=output->CSSS[3][J]+WF*gas->SP[5][L]*molecs->PV[2][N]/B;
    //this assumes that any flow normal to the x direction is in the y direction
    output->CSSS[4][J]=output->CSSS[4][J]+WF*gas->SP[5][L]*A/B;
    if(gas->ISPR[1][L] > 0){
        output->CSSS[5][J]=output->CSSS[5][J]+WF*molecs->PROT[N]/B;
        output->CSSS[6][J]=output->CSSS[6][J]+WF*gas->ISPR[1][L]/B;
    }
    //
    // CALL RANDOM_NUMBER(RANF)
    calc->RANF=((double)rand()/(double)RAND_MAX);
    if(gas->FSPEC[J] > calc->RANF){      //specular reflection
        X=2.e00*geom->XB[J]-X;
        molecs->PV[1][N]=-molecs->PV[1][N];
        DTR=(X-geom->XB[J])/molecs->PV[1][N];
    }
    else{                         //diffuse reflection
        VMPS=sqrt(2.e00*BOLTZ*gas->TSURF[J]/gas->SP[5][L]);
        DTR=(geom->XB[J]-molecs->PX[1][N])/molecs->PV[1][N];
        // CALL RANDOM_NUMBER(RANF)
        calc->RANF=((double)rand()/(double)RAND_MAX);
        molecs->PV[1][N]=sqrt(-log(calc->RANF))*VMPS;
        if(J == 2) molecs->PV[1][N]=-molecs->PV[1][N];
        RVELC(molecs->PV[2][N],molecs->PV[3][N],VMPS);
        molecs->PV[2][N]=molecs->PV[2][N]+gas->VSURF[J];
        if(gas->ISPR[1][L] > 0) SROT(L,gas->TSURF[J],molecs->PROT[N]);
        if(gas->MMVM > 0){
            for(K=1;K<=gas->ISPV[L];K++)
                SVIB(L,gas->TSURF[J],molecs->IPVIB[K][N],K);
        }
        if(gas->MELE > 1) SELE(L,gas->TSURF[J],molecs->PELE[N]);
    }
    //
    output->CSS[2][J][L][2]=output->CSS[2][J][L][2]-WF*molecs->PV[1][N]*gas->SP[5][L];
    output->CSS[3][J][L][2]=output->CSS[3][J][L][2]-WF*(molecs->PV[2][N]-gas->VSURF[J])*gas->SP[5][L];
    output->CSS[4][J][L][2]=output->CSS[4][J][L][2]-WF*molecs->PV[3][N]*gas->SP[5][L];
    A=pow(molecs->PV[1][N],2)+pow((molecs->PV[2][N]-gas->VSURF[J]),2)+pow(molecs->PV[3][N],2);
    output->CSS[5][J][L][2]=output->CSS[5][J][L][2]-WF*0.5e00*gas->SP[5][L]*A;
    if(gas->ISPR[1][L] > 0) output->CSS[6][J][L][2]=output->CSS[6][J][L][2]-WF*molecs->PROT[N];
    if(gas->MELE > 1) output->CSS[8][J][L][2]=output->CSS[8][J][L][2]-WF*molecs->PELE[N];
    if(gas->MMVM > 0){
        if(gas->ISPV[L] > 0){
            for(K=1;K<=gas->ISPV[L];K++)
                output->CSS[7][J][L][2]=output->CSS[7][J][L][2]-WF*double(molecs->IPVIB[K][N])*BOLTZ*gas->SPVM[1][K][L];
        }
    }
    A=pow(molecs->PV[1][N],2)+pow(molecs->PV[2][N],2)+pow(molecs->PV[3][N],2);
    B=fabs(molecs->PV[1][N]);
    output->CSSS[1][J]=output->CSSS[1][J]+WF/B;
    output->CSSS[2][J]=output->CSSS[2][J]+WF*gas->SP[5][L]/B;
    output->CSSS[3][J]=output->CSSS[3][J]+WF*gas->SP[5][L]*molecs->PV[2][N]/B;
    //this assumes that any flow normal to the x direction is in the y direction
    output->CSSS[4][J]=output->CSSS[4][J]+WF*gas->SP[5][L]*A/B;
    if(gas->ISPR[1][L] > 0){
        output->CSSS[5][J]=WF*output->CSSS[5][J]+molecs->PROT[N]/B;
        output->CSSS[6][J]=output->CSSS[6][J]+WF*gas->ISPR[1][L]/B;
    }
    //
    XI=geom->XB[J];
    DX=DTR*molecs->PV[1][N];
    DZ=0.e00;
    if(geom->IFX > 0) DY=DTR*molecs->PV[2][N];
    if(geom->IFX == 2) DZ=DTR*molecs->PV[3][N];
    if(geom->IFX == 0) X=XI+DX;
    if(geom->IFX > 0) AIFX(XI,DX,DY,DZ,X,molecs->PV[1][N],molecs->PV[2][N],molecs->PV[3][N]);
    //
    return;
}

void RBC(double &XI, double &DX, double &DY,double &DZ, double &R,double &S)
{
    //calculates the trajectory fraction S from a point at radius XI with
    //note that the axis is in the y direction
    //--displacements DX, DY, and DZ to a possible intersection with a
    //--surface of radius R, IFX=1, 2 for cylindrical, spherical geometry
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    double A,B,C,DD,S1,S2;
    //
    DD=DX*DX+DZ*DZ;
    if(geom->IFX == 2) DD=DD+DY*DY;
    B=XI*DX/DD;
    C=(XI*XI-R*R)/DD;
    A=B*B-C;
    if(A >= 0.e00){
        //find the least positive solution to the quadratic
        A=sqrt(A);
        S1=-B+A;
        S2=-B-A;
        if(S2 < 0.e00){
            if(S1 > 0.e00)
                S=S1;
            else
                S=2.e00;
        }
        else if(S1 < S2)
            S=S1;
        else
            S=S2;
    }
    else
        S=2.e00;
    //setting S to 2 indicates that there is no intersection
    return;
    //
}

void AIFX(double &XI,double &DX, double &DY, double &DZ, double &X, double &U, double &V, double &W)
{
    //
    //calculates the new radius and realigns the velocity components in
    //--cylindrical and spherical flows
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    //INTEGER ::
    double A,B,C,DR,VR,S;
    //
    if(geom->IFX == 1){
        DR=DZ;
        VR=W;
    }
    else if(geom->IFX == 2){
        DR=sqrt(DY*DY+DZ*DZ);
        VR=sqrt(V*V+W*W);
    }
    A=XI+DX;
    X=sqrt(A*A+DR*DR);
    S=DR/X;
    C=A/X;
    B=U;
    U=B*C+VR*S;
    W=-B*S+VR*C;
    if(geom->IFX == 2){
        VR=W;
        // CALL RANDOM_NUMBER(RANF)
        calc->RANF=((double)rand()/(double)RAND_MAX);
        A=DPI*calc->RANF;
        V=VR*sin(A);
        W=VR*cos(A);
    }
    //
    return;
    //
}

void REMOVE_MOL(int &N)
{
    //remove molecule N and replaces it by NM
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    //GAS gas;
    // IMPLICIT NONE
    //
    int NC,M,K;
    
    //N the molecule number
    //M,K working integer
    //
    if(N != molecs->NM){
        for(M=1;M<=calc->NCLASS;M++)
            molecs->PX[M][N]=molecs->PX[M][molecs->NM];
        for(M=1;M<=3;M++)
            molecs->PV[M][N]=molecs->PV[M][molecs->NM];
        
        if(gas->MMRM > 0) molecs->PROT[N]=molecs->PROT[molecs->NM];
        molecs->IPCELL[N]=fabs(molecs->IPCELL[molecs->NM]);
        molecs->IPSP[N]=molecs->IPSP[molecs->NM];
        molecs->IPCP[N]=molecs->IPCP[molecs->NM];
        if(gas->MMVM > 0){
            for(M=1;M<=gas->MMVM;M++)
                molecs->IPVIB[M][N]=molecs->IPVIB[M][molecs->NM];
        }
        if(gas->MELE > 1) molecs->PELE[N]=molecs->PELE[molecs->NM];
        molecs->PTIM[N]=molecs->PTIM[molecs->NM];
    }
    molecs->NM=molecs->NM-1;
    //
    return;
    //
}

void INDEX_MOLS()
{
    //index the molecules to the collision cells
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    // IMPLICIT NONE
    //
    int N,M,K;
    //
    //N,M,K working integer
    //
    for(N=1;N<=geom->NCCELLS;N++)
        geom->ICCELL[2][N]=0;
    //
    if(molecs->NM != 0){
        for(N=1;N<=molecs->NM;N++){
            M=molecs->IPCELL[N];
            geom->ICCELL[2][M]=geom->ICCELL[2][M]+1;
        }
        //

        M=0;
        for(N=1;N<=geom->NCCELLS;N++){
            geom->ICCELL[1][N]=M;
            M=M+geom->ICCELL[2][N];
            geom->ICCELL[2][N]=0;
        }
        //

        for(N=1;N<=molecs->NM;N++){
            M=molecs->IPCELL[N];
            geom->ICCELL[2][M]=geom->ICCELL[2][M]+1;
            K=geom->ICCELL[1][M]+geom->ICCELL[2][M];
            molecs->ICREF[K]=N;
        }
        //cin.get();
        //
    }
    return;
}

void SAMPLE_FLOW()
{
    //sample the flow properties
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    //GAS gas;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    int NC,NCC,LS,N,M,K,L,I,KV;
    double A,TE,TT,WF;
    //
    //NC the sampling cell number
    //NCC the collision cell number
    //LS the species code
    //N,M,K working integers
    //TE total translational energy
    //
    output->NSAMP=output->NSAMP+1;
    cout<<"Sample \t"<<output->NSAMP<<endl<<endl;
    //WRITE (9,*) NM,'Mols. at sample',NSAMP
    file_9<<molecs->NM<<"  Mols. at sample  "<<output->NSAMP<<endl;
    //
    for(N=1;N<=molecs->NM;N++){
        
        NCC=molecs->IPCELL[N];
        NC=geom->ICCELL[3][NCC];
        WF=1.e00;
        if(geom->IWF == 1) WF=1.e00+geom->WFM*pow(molecs->PX[1][N],geom->IFX);
        if((NC > 0) && (NC <= geom->NCELLS)){
            if(gas->MSP > 1)
                LS=fabs(molecs->IPSP[N]);
            else
                LS=1;
            
            output->CS[0][NC][LS]=output->CS[0][NC][LS]+1.e00;
            output->CS[1][NC][LS]=output->CS[1][NC][LS]+WF;
            for(M=1;M<=3;M++){
                output->CS[M+1][NC][LS]=output->CS[M+1][NC][LS]+WF*molecs->PV[M][N];
                output->CS[M+4][NC][LS]=output->CS[M+4][NC][LS]+WF*pow(molecs->PV[M][N],2);
            }
            if(gas->MMRM > 0) output->CS[8][NC][LS]=output->CS[8][NC][LS]+WF*molecs->PROT[N];
            if(gas->MELE > 1) output->CS[9][NC][LS]=output->CS[9][NC][LS]+WF*molecs->PELE[N];
            if(gas->MMVM > 0){
                if(gas->ISPV[LS] > 0){
                    for(K=1;K<=gas->ISPV[LS];K++)
                        output->CS[K+9][NC][LS]=output->CS[K+9][NC][LS]+WF*double(molecs->IPVIB[K][N]);
                }
            }
        }
        else{
            cout<<"Illegal sampling cell  "<<NC<<"  "<<NCC<<"  for MOL  "<<N<<"  at  "<<molecs->PX[1][N]<<endl;
            return;
        }
        
    }
    //
    if(calc->FTIME > 0.5e00*calc->DTM) calc->TSAMP=calc->TSAMP+calc->DTSAMP;
    //
    return;
}

void ADAPT_CELLS_1D()
{
    //adapt the sampling cells through the splitting of the divisions into successive levels
    //the collision cells are divisions of the sampling cells
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    int M,N,L,K,KK,I,J,JJ,MSEG,NSEG,NSEG1,NSEG2,MLEVEL;
    double A,B,DDE,DCRIT;
    int *KDIV,*NC;
    int **ISD;
    double *XMIN,*XMAX,*DRAT;
    // INTEGER, ALLOCATABLE, DIMENSION(:) :: KDIV,NC
    // INTEGER, ALLOCATABLE, DIMENSION(:,:) :: ISD
    // REAL(KIND=8), ALLOCATABLE, DIMENSION(:) :: XMIN,XMAX,DRAT
    //
    //DCRIT  the number density ratio that causes a cell to be subdivided
    //KDIV(N) the number of divisions/subdivisions (cells or further subdivisions) at level N
    //DRAT(N) the contriburion to the density ratio of element N
    //NC(I) the number of sampling cells at level I
    //DDE the width of an element
    //MSEG the maximum number of segments (a segment is the size of the smallest subdivision
    //NSEG1 the (first segment-1) in the subdivision
    //NSEG2 the final segment in the subdivision
    //ISD(N,M) 0,1 for cell,subdivided for level N subdivision
    //MLEVEL The maximum desired level ILEVEL of subdivision (cellS are proportional to 2**ILEVEL)
    //
    DCRIT=1.5e00;    //may be altered
    MLEVEL=2;    //may be altered
    //
    //determine the level to which the divisions are to be subdivided
    //
    A=1.e00;
    for(N=1;N<=geom->NCELLS;N++)
        if(output->VAR[3][N]/gas->FND[1] > A) A=output->VAR[3][N]/gas->FND[1];
    
    geom->ILEVEL=0;
    while(A > DCRIT){
        geom->ILEVEL=geom->ILEVEL+1;
        A=A/2.e00;
    }
    if(geom->ILEVEL > MLEVEL) geom->ILEVEL=MLEVEL;
    //WRITE (9,*) 'ILEVEL =',ILEVEL
    file_9<<"ILEVEL = "<<geom->ILEVEL<<endl;
    NSEG=pow(2,geom->ILEVEL);
    MSEG=geom->NDIV*NSEG;
    //
    
    KDIV = new int[geom->ILEVEL+1];
    DRAT = new double[MSEG+1];
    NC = new int[geom->ILEVEL+1];
    
    ISD = new int*[geom->ILEVEL+1];
    for(int i =0; i< (geom->ILEVEL+1); ++i)
        ISD[i] = new int[MSEG+1];
    
    
    // ALLOCATE (KDIV(0:ILEVEL),DRAT(MSEG),NC(0:ILEVEL),ISD(0:ILEVEL,MSEG),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR KDIV ARRAY',ERROR
    // ENDIF
    //
    DDE=(geom->XB[2]-geom->XB[1])/double(MSEG);
    for(N=1;N<=MSEG;N++){
        A=geom->XB[1]+(double(N)-0.5e00)*DDE;
        FIND_CELL_1D(A,M,L);
        DRAT[N]=output->VAR[3][L]/(gas->FND[1]*double(NSEG));
    }
    //
    //calculate the number of subdivisions at the various levels of subdivision
    KDIV=0;
    //also the number of sampling cells at each level
    NC=0;
    //
    for(N=1;N<=geom->NDIV;N++){  //divisions
        ISD=0;
        ISD[0][1]=1;
        KDIV[0]=KDIV[0]+1;
        //  WRITE (9,*) 'DIVISION',N
        for(I=0;I<=geom->ILEVEL;I++){  //level of subdivision
            //    WRITE (9,*) 'LEVEL',I
            J=pow(2,I);  //number of possible subdivisions at this level
            JJ=NSEG/J;  //number of segments in a subdivision
            for(M=1;M<=J;M++){
                //      WRITE (9,*) 'SUBDIVISION',M
                if(ISD[I][M] == 1){
                    NSEG1=(N-1)*NSEG+(M-1)*JJ+1;
                    NSEG2=NSEG1+JJ-1;
                    A=0.e00;
                    //        WRITE (9,*) 'NSEG RANGE',NSEG1,NSEG2
                    for(L=NSEG1;L<=NSEG2;L++)
                        A=A+DRAT[L];
                    
                    //        WRITE (9,*) 'DENS CONTRIB',A
                    if(A < DCRIT){
                        NC[I]=NC[I]+1;
                        //          WRITE (9,*) 'LEVEL',I,' CELLS TO', NC(I)
                    }
                    else{
                        KDIV[I+1]=KDIV[I+1]+2;
                        //          WRITE (9,*) 'LEVEL',I+1,' SUBDIVISIONS TO',KDIV(I+1)
                        for(L=NSEG1-(N-1)*NSEG;L<=NSEG2-(N-1)*NSEG;L++)
                            ISD[I+1][L]=1;
                        
                    }
                }
            }
        }
    }
    //
    //WRITE (9,*) 'KDIV',KDIV
    file_9<<"KDIV "<<KDIV<<endl;
    //
    //WRITE (9,*) 'NC',NC
    file_9<< "NC "<<NC<<endl;
    cin.get();
    //WRITE (9,*) 'Number of divisions',NDIV
    file_9<<"Number of divisions "<<geom->NDIV<<endl;
    A=0;
    geom->NCELLS=0;
    for(N=0;N<=geom->ILEVEL;N++){
        A=A+double(NC[N])/(pow(2.e00,N));
        geom->NCELLS=geom->NCELLS+NC[N];
    }
    //WRITE (9,*) 'Total divisions from sampling cells',A
    //WRITE (9,*) 'Adapted sampling cells',NCELLS
    file_9<< "Total divisions from sampling cells "<<A<<endl;
    file_9<< "Adapted sampling cells "<<geom->NCELLS<<endl;
    geom->NCCELLS=geom->NCELLS*geom->NCIS;
    //WRITE (9,*) 'Adapted collision cells',NCCELLS
    file_9<< "Adapted collision cells "<<geom->NCCELLS<<endl;
    //
    
    for (int i = 0; i < geom->ILEVEL+1; i++) {
        hipFree(geom->JDIV[i]); //delete [] geom->JDIV[i];
    }
    hipFree(geom->JDIV); //delete [] geom->JDIV;  // <- because they won't exist anymore after this
    
    for (int i = 0; i < 5; i++) {
        hipFree(geom->CELL[i]); //delete [] geom->CELL[i];
    }
    hipFree(geom->CELL); //delete [] geom->CELL;  // <- because they won't exist anymore after this
    
    
    hipFree(geom->ICELL); //delete[] geom->ICELL;
    
    for (int i = 0; i < 6; i++) {
        hipFree(geom->CCELL[i]); //delete [] geom->CCELL[i];
    }
    hipFree(geom->CCELL); //delete [] geom->CCELL;  // <- because they won't exist anymore after this
    
    for (int i = 0; i < 4; i++) {
        hipFree(geom->ICCELL[i]); //delete [] geom->ICCELL[i];
    }
    hipFree(geom->ICCELL); //delete [] geom->ICCELL;  // <- because they won't exist anymore after this
    
    hipFree(output->COLLS);  //delete[] output->COLLS;
    
    hipFree(output->WCOLLS); //delete[] output->WCOLLS;
    
    hipFree(output->CLSEP); //delete[] output->CLSEP;
    
    for (int i = 0; i < 24; i++) {
        hipFree(output->VAR[i]); //delete [] output->VAR[i];
    }
    hipFree(output->VAR); //delete [] output->VAR;  // <- because they won't exist anymore after this
    
    
    for(int i = 0; i < 13; i++)
    {
        for(int j = 0; j < geom->NCELLS+1; j++)
        {
            hipFree(output->VARSP[i][j]); //delete [] output->VARSP[i][j];
        }
        hipFree(output->VARSP[i]); //delete [] output->VARSP[i];
    }
    hipFree(output->VARSP); //delete [] output->VARSP;
    
    for(int i = 0; i < (10+gas->MSP); i++)
    {
        for(int j = 0; j < geom->NCELLS+1; j++)
        {
            hipFree(output->CS[i][j]); //delete [] output->CS[i][j];
        }
        hipFree(output->CS[i]); //delete [] output->CS[i];
    }
    hipFree(output->CS); //delete [] output->CS;
    /*DEALLOCATE (JDIV,CELL,ICELL,CCELL,ICCELL,COLLS,WCOLLS,CLSEP,VAR,VARSP,CS,STAT=ERROR)
     IF (ERROR /= 0) THEN
     WRITE (*,*)'PROGRAM COULD NOT DEALLOCATE ARRAYS IN ADAPT',ERROR
     END IF*/
    //
    for(N=0;N<=geom->ILEVEL;N++)
        if(KDIV[N] > geom->MDIV) geom->MDIV=KDIV[N];
    //
    
    geom->i_allocate(geom->ILEVEL+1,geom->MDIV, geom->JDIV);
    //    ALLOCATE (JDIV(0:ILEVEL,MDIV),STAT=ERROR)
    //    IF (ERROR /= 0) THEN
    //    WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR JDIV ARRAY IN ADAPT',ERROR
    //    ENDIF
    //
    
    geom->d_allocate(5,geom->NCELLS+1, geom->CELL);
    geom->i_allocate(geom->NCELLS+1, geom->ICELL);
    geom->d_allocate(6, geom->NCCELLS+1, geom->CCELL);
    geom->i_allocate(4, geom->NCCELLS+1,geom->ICCELL);
    
    XMIN= new double[geom->NCCELLS+1];
    XMAX = new double[geom->NCCELLS+1];
    //
    //    ALLOCATE (CELL(4,NCELLS),ICELL(NCELLS),CCELL(5,NCCELLS),ICCELL(3,NCCELLS),XMIN(NCCELLS),XMAX(NCCELLS),STAT=ERROR)
    //    IF (ERROR /= 0) THEN
    //    WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR CELL ARRAYS IN ADAPT',ERROR
    //    ENDIF
    //
    
    output->d_allocate(geom->NCELLS+1,output->COLLS);
    output->d_allocate(geom->NCELLS+1, output->WCOLLS);
    output->d_allocate(geom->NCELLS+1, output->CLSEP);
    output->d_allocate(24, geom->NCELLS+1, output->VAR);
    output->d_allocate(13,geom->NCELLS+1,gas->MSP+1, output->VARSP);
    output->d_allocate(10+gas->MSP+1,geom->NCELLS+1,gas->MSP+1,output->CS);
    
    
    //    ALLOCATE (COLLS(NCELLS),WCOLLS(NCELLS),CLSEP(NCELLS),VAR(23,NCELLS),VARSP(0:12,NCELLS,MSP),CS(0:9+MSP,NCELLS,MSP),STAT=ERROR)
    //    IF (ERROR /= 0) THEN
    //    WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR SAMPLING ARRAYS IN ADAPT',ERROR
    //    ENDIF
    //
    geom->NCCELLS=0;
    geom->NCELLS=0;
    //
    //set the JDIV arrays and the sampling cells at the various levels of subdivision
    KDIV=0;
    geom->JDIV=0;
    //
    for(N=1;N<=geom->NDIV;N++){    //divisions
        ISD=0;
        ISD[0][1]=1;
        KDIV[0]=KDIV[0]+1;
        for(I=0;I<=geom->ILEVEL;I++){  //level of subdivision
            J=pow(2,I);  //number of possible subdivisions at this level
            JJ=NSEG/J;  //number of segments in a subdivision
            for(M=1;M<=J;M++){
                if(ISD[I][M] == 1){
                    NSEG1=(N-1)*NSEG+(M-1)*JJ+1;
                    NSEG2=NSEG1+JJ-1;
                    A=0.e00;
                    for(L=NSEG1;L<=NSEG2;L++)
                        A=A+DRAT[L];
                    
                    if(A < DCRIT){
                        geom->NCELLS=geom->NCELLS+1;
                        output->VAR[11][geom->NCELLS]=gas->FTMP[1];
                        XMIN[geom->NCELLS]=geom->XB[1]+double(NSEG1-1)*DDE;
                        XMAX[geom->NCELLS]=XMIN[geom->NCELLS]+double(NSEG2-NSEG1+1)*DDE;
                        //WRITE (9,*) NCELLS,I,' XMIN,XMAX',XMIN(NCELLS),XMAX(NCELLS)
                        file_9<< geom->NCELLS<<" "<<I<<" XMIN,XMAX "<<XMIN[geom->NCELLS]<<" , "<<XMAX[geom->NCELLS]<<endl;
                        geom->JDIV[I][KDIV[I]-(J-M)]=-geom->NCELLS;
                        //          WRITE (9,*) 'JDIV(',I,',',KDIV(I)-(J-M),')=',-NCELLS
                    }
                    else{
                        geom->JDIV[I][KDIV[I]-(J-M)]=KDIV[I+1];
                        //          WRITE (9,*) 'JDIV(',I,',',KDIV(I)-(J-M),')=',KDIV(I+1)
                        KDIV[I+1]=KDIV[I+1]+2;
                        for(L=NSEG1-(N-1)*NSEG;L<=NSEG2-(N-1)*NSEG;L++)
                            ISD[I+1][L]=1;
                    }
                }
            }
        }
    }
    //
    //set the other quantities associated with the sampling cells and the collision cells
    //
    geom->NCCELLS=0;
    for(N=1;N<=geom->NCELLS;N++){
        geom->CELL[1][N]=(XMIN[N]+XMAX[N])/2.e00;
        geom->CELL[2][N]=XMIN[N];
        geom->CELL[3][N]=XMAX[N];
        if(geom->IFX == 0) geom->CELL[4][N]=XMAX[N]-XMIN[N];    //calculation assumes unit cross-section
        if(geom->IFX == 1) geom->CELL[4][N]=PI*(pow(XMAX[N],2)-pow(XMIN[N],2));
        if(geom->IFX == 2) geom->CELL[4][N]=1.33333333333333333333e00*PI*(pow(XMAX[N],3)-pow(XMIN[N],3));
        geom->ICELL[N]=geom->NCCELLS;
        for(M=1;M<=geom->NCIS;M++){
            geom->NCCELLS=geom->NCCELLS+1;
            geom->ICCELL[3][geom->NCCELLS]=N;
            geom->CCELL[1][geom->NCCELLS]=geom->CELL[4][N]/double(geom->NCIS);
            geom->CCELL[3][geom->NCCELLS]=calc->DTM/2.e00;
            geom->CCELL[4][geom->NCCELLS]=2.e00*gas->VMPM*gas->SPM[2][1][1];
            // CALL RANDOM_NUMBER(RANF)
            calc->RANF=((double)rand()/(double)RAND_MAX);
            geom->CCELL[2][geom->NCCELLS]=calc->RANF;
            geom->CCELL[5][geom->NCCELLS]=calc->FTIME;
        }
    }
    //
    //assign the molecules to the cells
    //
    for(N=1;N<=molecs->NM;N++){
        FIND_CELL_1D(molecs->PX[1][N],molecs->IPCELL[N],JJ);
        M=molecs->IPCELL[N];
    }
    //
    //deallocate the local variables
    for (int i = 0; i < geom->ILEVEL+1; i++) {
        delete [] ISD[i];
    }
    delete [] ISD;
    delete [] NC;
    delete[] KDIV;
    delete [] XMAX;
    delete [] XMIN;
    delete [] DRAT;
    /*DEALLOCATE (KDIV,NC,ISD,XMIN,XMAX,DRAT,STAT=ERROR)
     IF (ERROR /= 0) THEN
     WRITE (*,*)'PROGRAM COULD NOT DEALLOCATE LOCAL ARRAYS IN ADAPT',ERROR
     END IF*/
    //
    return;
}

void EXTEND_MNM(double FAC)
{  //
    //the maximum number of molecules is increased by a specified factor
    //the existing molecules are copied TO disk storage
    //MOLECS molecs;
    //CALC calc;
    //GAS gas;
    //
    // IMPLICIT NONE
    //
    int M,N,MNMN;
    fstream file_7;
    // REAL :: FAC
    //
    //M,N working integers
    //MNMN extended value of MNM
    //FAC the factor for the extension
    MNMN=FAC*molecs->MNM;
    cout<< "Maximum number of molecules is to be extended from "<<molecs->MNM<<" to "<<MNMN<<endl;
    cout<< "( if the additional memory is available //// )"<<endl;
    
    file_7.open("EXTMOLS.SCR", ios::binary | ios::out);
    if(file_7.is_open()){
        cout<<"EXTMOLS.SCR is opened"<<endl;
    }
    else{
        cout<<"EXTMOLS.SCR not opened"<<endl;
    }
    cout<<"Start write to disk storage"<<endl;
    //OPEN (7,FILE='EXTMOLS.SCR',FORM='BINARY')
    //WRITE (*,*) 'Start write to disk storage'
    
    for(N=1;N<=molecs->MNM;N++){
        if(gas->MMVM > 0){
            file_7<<molecs->PX[calc->NCLASS][N]<<endl<<molecs->PTIM[N]<<endl<<molecs->PROT[N]<<endl;
            for(M=1;M<=3;M++)
                file_7<<molecs->PV[M][N]<<endl;
            file_7<<molecs->IPSP[N]<<endl<<molecs->IPCELL[N]<<endl<<molecs->ICREF[N]<<endl<<molecs->IPCP[N]<<endl;
            for(M=1;M<=gas->MMVM;M++)
                file_7<<molecs->IPVIB[M][N]<<endl;
            file_7<<molecs->PELE[N]<<endl;//WRITE (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),(IPVIB(M,N),M=1,MMVM),PELE(N)
        }
        else{
            if(gas->MMRM > 0){
                file_7<<molecs->PX[calc->NCLASS][N]<<endl<<molecs->PTIM[N]<<endl<<molecs->PROT[N]<<endl;
                for(M=1;M<=3;M++)
                    file_7<<molecs->PV[M][N]<<endl;
                file_7<<molecs->IPSP[N]<<endl<<molecs->IPCELL[N]<<endl<<molecs->ICREF[N]<<endl<<molecs->IPCP[N]<<endl<<molecs->PELE[N]<<endl;//WRITE (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
            else{
                file_7<<molecs->PX[calc->NCLASS][N]<<endl<<molecs->PTIM[N]<<endl;
                for(M=1;M<=3;M++)
                    file_7<<molecs->PV[M][N]<<endl;
                file_7<<molecs->IPSP[N]<<endl<<molecs->IPCELL[N]<<endl<<molecs->ICREF[N]<<endl<<molecs->IPCP[N]<<endl<<molecs->PELE[N]<<endl;//WRITE (7) PX(NCLASS,N),PTIM(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
            
        }
    }
    cout<<"Disk write completed"<<endl;
    // WRITE (*,*) 'Disk write completed'
    // CLOSE (7)
    file_7.close();
    if(gas->MMVM > 0){
        for(int i=0;i<calc->NCLASS+1;i++){
                hipFree(molecs->PX[i]); //delete [] molecs->PX[i];
        }
        hipFree(molecs->PX); //delete [] molecs->PX;

        hipFree(molecs->PTIM); //delete [] molecs->PTIM;

        hipFree(molecs->PROT);

        for(int i=0;i<4;i++){
            hipFree(molecs->PV[i]); //delete [] molecs->PV[i];
        }
        hipFree(molecs->PV); //delete [] molecs->PV;

        hipFree(molecs->IPSP);
        hipFree(molecs->IPCELL);
        hipFree(molecs->ICREF);
        hipFree(molecs->IPCP);
        hipFree(molecs->PELE);
        for(int i=0;i<gas->MMVM;i++){
            hipFree(molecs->IPVIB[i]); //delete [] molecs->IPVIB[i];
        }
        hipFree(molecs->IPVIB); //delete molecs->IPVIB;
        // for(int i=0;i<calc->NCLASS+1;i++){
        //     delete [] molecs->PX[i];
        // }
        // delete [] molecs->PX;
        // delete [] molecs->PTIM;
        // delete [] molecs->PROT;
        // for(int i=0;i<4;i++){
        //     delete [] molecs->PV[i];
        // }
        // delete [] molecs->PV;
        // delete [] molecs->IPSP;
        // delete [] molecs->IPCELL;
        // delete [] molecs->ICREF;
        // delete [] molecs->IPCP;
        // delete [] molecs->PELE;
        // for(int i=0;i<gas->MMVM;i++){
        //     delete [] molecs->IPVIB[i];
        // }
        // delete molecs->IPVIB;
        //DEALLOCATE (PX,PTIM,PROT,PV,IPSP,IPCELL,ICREF,IPCP,IPVIB,PELE,STAT=ERROR)
    }
    else{
        if(gas->MMRM > 0){
            for(int i=0;i<calc->NCLASS+1;i++){
                hipFree(molecs->PX[i]); //delete [] molecs->PX[i];
            }
            hipFree(molecs->PX); //delete [] molecs->PX;

            hipFree(molecs->PTIM); //delete [] molecs->PTIM;

            hipFree(molecs->PROT);

            for(int i=0;i<4;i++){
                hipFree(molecs->PV[i]); //delete [] molecs->PV[i];
            }
            hipFree(molecs->PV); //delete [] molecs->PV;

            hipFree(molecs->IPSP);
            hipFree(molecs->IPCELL);
            hipFree(molecs->ICREF);
            hipFree(molecs->IPCP);
            hipFree(molecs->PELE);
            // delete [] molecs->IPSP;
            // delete [] molecs->IPCELL;
            // delete [] molecs->ICREF;
            // delete [] molecs->IPCP;
            // delete [] molecs->PELE;//DEALLOCATE (PX,PTIM,PV,IPSP,IPCELL,ICREF,IPCP,PELE,STAT=ERROR)
            // for(int i=0;i<calc->NCLASS+1;i++){
            //     delete [] molecs->PX[i];
            // }
            // delete [] molecs->PX;
            // delete [] molecs->PTIM;
            // delete [] molecs->PROT;
            // for(int i=0;i<4;i++){
            //     delete [] molecs->PV[i];
            // }
            // delete [] molecs->PV;
            // delete [] molecs->IPSP;
            // delete [] molecs->IPCELL;
            // delete [] molecs->ICREF;
            // delete [] molecs->IPCP;
            // delete [] molecs->PELE;
            //DEALLOCATE (PX,PTIM,PROT,PV,IPSP,IPCELL,ICREF,IPCP,PELE,STAT=ERROR)
        }
        else{
            for(int i=0;i<calc->NCLASS+1;i++){
                hipFree(molecs->PX[i]); //delete [] molecs->PX[i];
            }
            hipFree(molecs->PX); //delete [] molecs->PX;

            hipFree(molecs->PTIM); //delete [] molecs->PTIM;

            for(int i=0;i<4;i++){
                hipFree(molecs->PV[i]); //delete [] molecs->PV[i];
            }
            hipFree(molecs->PV); //delete [] molecs->PV;

            hipFree(molecs->IPSP);
            hipFree(molecs->IPCELL);
            hipFree(molecs->ICREF);
            hipFree(molecs->IPCP);
            hipFree(molecs->PELE);
            // delete [] molecs->IPSP;
            // delete [] molecs->IPCELL;
            // delete [] molecs->ICREF;
            // delete [] molecs->IPCP;
            // delete [] molecs->PELE;//DEALLOCATE (PX,PTIM,PV,IPSP,IPCELL,ICREF,IPCP,PELE,STAT=ERROR)
        }
    }
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT DEALLOCATE MOLECULES',ERROR
    // !  STOP
    // END IF
    // !
    
    if(gas->MMVM > 0){
        molecs->d_allocate(calc->NCLASS+1,MNMN+1,molecs->PX);
        molecs->d_allocate(MNMN+1,molecs->PTIM);
        molecs->d_allocate(MNMN+1,molecs->PROT);
        molecs->d_allocate(4,MNMN+1,molecs->PV);
        molecs->i_allocate(MNMN+1,molecs->IPSP);
        molecs->i_allocate(MNMN+1,molecs->IPCELL);
        molecs->i_allocate(MNMN+1,molecs->ICREF);
        molecs->i_allocate(MNMN+1,molecs->IPCP);
        molecs->i_allocate(gas->MMVM+1,MNMN+1,molecs->IPVIB);
        molecs->d_allocate(MNMN+1,molecs->PELE);
        // ALLOCATE (PX(NCLASS,MNMN),PTIM(MNMN),PROT(MNMN),PV(3,MNMN),IPSP(MNMN),IPCELL(MNMN),ICREF(MNMN),IPCP(MNMN),IPVIB(MMVM,MNMN),PELE(MNMN),STAT=ERROR)
    }
    else{
        if(gas->MMRM > 0){
            molecs->d_allocate(calc->NCLASS+1,MNMN+1,molecs->PX);
            molecs->d_allocate(MNMN+1,molecs->PTIM);
            molecs->d_allocate(MNMN+1,molecs->PROT);
            molecs->d_allocate(4,MNMN+1,molecs->PV);
            molecs->i_allocate(MNMN+1,molecs->IPSP);
            molecs->i_allocate(MNMN+1,molecs->IPCELL);
            molecs->i_allocate(MNMN+1,molecs->ICREF);
            molecs->i_allocate(MNMN+1,molecs->IPCP);
            molecs->d_allocate(MNMN+1,molecs->PELE);
            // ALLOCATE (PX(NCLASS,MNMN),PTIM(MNMN),PROT(MNMN),PV(3,MNMN),IPSP(MNMN),IPCELL(MNMN),ICREF(MNMN),IPCP(MNMN),PELE(MNMN),STAT=ERROR)
        }
        else{
            molecs->d_allocate(calc->NCLASS+1,MNMN+1,molecs->PX);
            molecs->d_allocate(MNMN+1,molecs->PTIM);
            molecs->d_allocate(4,MNMN+1,molecs->PV);
            molecs->i_allocate(MNMN+1,molecs->IPSP);
            molecs->i_allocate(MNMN+1,molecs->IPCELL);
            molecs->i_allocate(MNMN+1,molecs->ICREF);
            molecs->i_allocate(MNMN+1,molecs->IPCP);
            molecs->d_allocate(MNMN+1,molecs->PELE);
            // ALLOCATE (PX(NCLASS,MNMN),PTIM(MNMN),PV(3,MNMN),IPSP(MNMN),IPCELL(MNMN),ICREF(MNMN),IPCP(MNMN),PELE(MNMN),STAT=ERROR)
        }
    }
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*)'PROGRAM COULD NOT ALLOCATE SPACE FOR EXTEND_MNM',ERROR
    // !  STOP
    // END IF
    // !
    //memset(molecs->PX,0.0,sizeof(**molecs->PX)); memset(molecs->PTIM,0.0,sizeof(*molecs->PTIM)); memset(molecs->PV,0.0,sizeof(**molecs->PV)); memset(molecs->IPSP,0,sizeof(*molecs->IPSP)); memset(molecs->IPCELL,0,sizeof(*molecs->IPCELL)); memset(molecs->ICREF,0,sizeof(*molecs->ICREF)); memset(molecs->IPCP,0,sizeof(*molecs->IPCP)); memset(molecs->PELE,0,sizeof(*molecs->PELE));
    
    for(int i=0;i<calc->NCLASS+1;i++){
        for(int j=0;j<MNMN+1;j++)
            molecs->PX[i][j]=0.0;
    }
    
    for(int i=0;i<4;i++){
        for(int j=0;j<MNMN+1;j++)
            molecs->PV[i][j]=0.0;
    }
    for(int i=0;i<MNMN+1;i++){
        molecs->PTIM[i]=0.0;
        molecs->IPSP[i]=0;
        molecs->IPCELL[i]=0;
        molecs->ICREF[i]=0;
        molecs->IPCP[i]=0;
        molecs->PELE[i]=0;
    }
        
    
    if(gas->MMRM > 0) {
        for(int i=0;i<MNMN+1;i++)
            molecs->PROT[i]=0.0;
        //memset(molecs->PROT,0.0,sizeof(*molecs->PROT));
    }
    if(gas->MMVM > 0) {
        for(int i=0;i<gas->MMVM+1;i++){
            for(int j=0;j<MNMN+1;j++)
                molecs->IPVIB[i][j]=0;
        }
        //memset(molecs->IPVIB,0,sizeof(**molecs->IPVIB));
    }
    //restore the original molecules
    // OPEN (7,FILE='EXTMOLS.SCR',FORM='BINARY')
    // WRITE (*,*) 'Start read back from disk storage'
    file_7.open("EXTMOLS.SCR", ios::binary | ios::in);
    if(file_7.is_open()){
        cout<<"EXTMOLS.SCR is opened"<<endl;
    }
    else{
        cout<<"EXTMOLS.SCR not opened"<<endl;
    }
    for(N=1;N<=molecs->MNM;N++){
        if(gas->MMVM > 0){
            file_7>>molecs->PX[calc->NCLASS][N]>>molecs->PTIM[N]>>molecs->PROT[N];
            for(M=1;M<=3;M++)
                file_7>>molecs->PV[M][N];
            file_7>>molecs->IPSP[N]>>molecs->IPCELL[N]>>molecs->ICREF[N]>>molecs->IPCP[N];
            for(M=1;M<=gas->MMVM;M++)
                file_7>>molecs->IPVIB[M][N];
            file_7>>molecs->PELE[N];//READ (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),(IPVIB(M,N),M=1,MMVM),PELE(N)
        }
        else{
            if(gas->MMRM > 0){
                file_7>>molecs->PX[calc->NCLASS][N]>>molecs->PTIM[N]>>molecs->PROT[N];
                for(M=1;M<=3;M++)
                    file_7>>molecs->PV[M][N];
                file_7>>molecs->IPSP[N]>>molecs->IPCELL[N]>>molecs->ICREF[N]>>molecs->IPCP[N]>>molecs->PELE[N];//READ (7) PX(NCLASS,N),PTIM(N),PROT(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
            else{
                file_7>>molecs->PX[calc->NCLASS][N]>>molecs->PTIM[N];
                for(M=1;M<=3;M++)
                    file_7>>molecs->PV[M][N];
                file_7>>molecs->IPSP[N]>>molecs->IPCELL[N]>>molecs->ICREF[N]>>molecs->IPCP[N]>>molecs->PELE[N];//READ (7) PX(NCLASS,N),PTIM(N),(PV(M,N),M=1,3),IPSP(N),IPCELL(N),ICREF(N),IPCP(N),PELE(N)
            }
        }
    }
    cout<<"Disk read completed"<<endl;
    // WRITE (*,*) 'Disk read completed'
    // CLOSE (7,STATUS='DELETE')
    file_7.close();
    //
    molecs->MNM=MNMN;
    //
    return;
}

void DISSOCIATION()
{
    //dissociate diatomic molecules that have been marked for dissociation by -ve level or -99999 for ground state
    //MOLECS molecs;
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int K,KK,L,N,M,LS,MS,KV,IDISS;
    double A,B,C,EA,VRR,VR,RMM,RML;
    double VRC[4],VCM[4],VRCP[4];
    //
    N=0;
    while(N < molecs->NM){
        N=N+1;
        IDISS=0;
        L=molecs->IPSP[N];
        if(gas->ISPV[L] > 0){
            for(K=1;K<=gas->ISPV[L];K++){
                M=molecs->IPVIB[K][N];
                if(M < 0){
                    //dissociation
                    calc->TDISS[L]=calc->TDISS[L]+1.e00;
                    IDISS=1;
                }
            }
            if(IDISS == 1){
                EA=molecs->PROT[N];    //EA is energy available for relative translational motion of atoms
                if(gas->MELE > 1) EA=EA+molecs->PELE[N];
                if(molecs->NM >= molecs->MNM) EXTEND_MNM(1.1);
                molecs->NM=molecs->NM+1;
                //set center of mass velocity as that of molecule
                VCM[1]=molecs->PV[1][N];
                VCM[2]=molecs->PV[2][N];
                VCM[3]=molecs->PV[3][N];
                molecs->PX[calc->NCLASS][molecs->NM]=molecs->PX[calc->NCLASS][N];
                molecs->IPCELL[molecs->NM]=molecs->IPCELL[N];
                LS=molecs->IPSP[N];
                gas->TREACL[1][LS]=gas->TREACL[1][LS]-1;
                molecs->IPSP[molecs->NM]=gas->ISPVM[1][1][L];
                MS=molecs->IPSP[molecs->NM];
                molecs->IPSP[N]=gas->ISPVM[2][1][L];
                LS=molecs->IPSP[N];
                gas->TREACG[1][LS]=gas->TREACG[1][LS]+1;
                gas->TREACG[1][MS]=gas->TREACG[1][MS]+1;
                molecs->PTIM[molecs->NM]=molecs->PTIM[N];
                VRR=2.e00*EA/gas->SPM[1][LS][MS];
                VR=sqrt(VRR);
                RML=gas->SPM[1][LS][MS]/gas->SP[5][MS];
                RMM=gas->SPM[1][LS][MS]/gas->SP[5][LS];
                // CALL RANDOM_NUMBER(RANF)
                calc->RANF=((double)rand()/(double)RAND_MAX);
                B=2.e00*calc->RANF-1.e00;
                A=sqrt(1.e00-B*B);
                VRCP[1]=B*VR;
                // CALL RANDOM_NUMBER(RANF)
                calc->RANF=((double)rand()/(double)RAND_MAX);
                C=2.e00*PI*calc->RANF;
                VRCP[2]=A*cos(C)*VR;
                VRCP[3]=A*sin(C)*VR;
                for(KK=1;KK<=3;KK++){
                    molecs->PV[KK][N]=VCM[KK]+RMM*VRCP[KK];
                    molecs->PV[KK][molecs->NM]=VCM[KK]-RML*VRCP[KK];
                }
                
                if((fabs(molecs->PV[1][N]) > 100000.e00) || (fabs(molecs->PV[1][molecs->NM]) > 100000.e00)) {
                    cout<< "EXCESSIVE SPEED, DISS "<< N<< " "<<molecs->PV[1][N]<<" "<<molecs->NM<<" "<<molecs->PV[1][molecs->NM]<<endl;
                   
                }
                
                
                
                //set any internal modes to the ground state
                if(gas->ISPV[LS] > 0){
                    for(KV=1;KV<=gas->ISPV[LS];KV++)
                        molecs->IPVIB[KV][N]=0;
                }
                if(gas->ISPR[1][LS] > 0) molecs->PROT[N]=0.e00;
                if(gas->MELE > 1) molecs->PELE[N]=0.e00;
                if(gas->ISPV[MS] > 0){
                    for(KV=1;KV<=gas->ISPV[MS];KV++)
                        molecs->IPVIB[KV][molecs->NM]=0;
                }
                if(gas->ISPR[1][MS] > 0) molecs->PROT[molecs->NM]=0.0;
                if(gas->MELE > 1) molecs->PELE[molecs->NM]=0.e00;
            }
        }
    }
    return;
}
//************************************************************************************
//

void ENERGY(int I,double &TOTEN)
{
    //calculate the total energy (all molecules if I=0, otherwise molecule I)
    //I>0 used for dianostic purposes only
    //MOLECS molecs;
    //GAS gas;
    //CALC calc;
    //
    // IMPLICIT NONE
    //
    int K,L,N,II,M,IV,KV,J;
    double TOTENI,TOTELE;
    //
    TOTEN=0.0;
    TOTELE=0;
    //
    
    if(I == 0){
        for(N=1;N<=molecs->NM;N++){
            if(molecs->IPCELL[N] > 0){
                L=molecs->IPSP[N];
                TOTENI=TOTEN;
                TOTEN=TOTEN+gas->SP[6][L];
                TOTEN=TOTEN+0.5e00*gas->SP[5][L]*(pow(molecs->PV[1][N],2)+pow(molecs->PV[2][N],2)+pow(molecs->PV[3][N],2));
                if(gas->ISPR[1][L] > 0) TOTEN=TOTEN+molecs->PROT[N];
                if(gas->ISPV[L] > 0){
                    for(KV=1;KV<=gas->ISPV[L];KV++){
                        J=molecs->IPVIB[KV][N];
                        //         IF (J <0) THEN
                        //           J=-J
                        //           IF (J == 99999) J=0
                        //         END IF
                        TOTEN=TOTEN+double(J)*BOLTZ*gas->SPVM[1][KV][L];
                    }
                }
            }
            if(gas->MELE > 1){
                TOTEN=TOTEN+molecs->PELE[N];
                TOTELE=TOTELE+molecs->PELE[N];
            }
            if((TOTEN-TOTENI) > 1.e-16) cout<<"MOL "<<N<<" ENERGY "<<TOTEN-TOTENI<<endl;
        }
        //
        //WRITE (9,*) 'Total Energy =',TOTEN,NM
        //WRITE (*,*) 'Total Energy =',TOTEN,NM
        file_9<<"Total Energy =  "<<setprecision(25)<<TOTEN<<"\t"<<molecs->NM<<endl;
        cout<<"Total Energy =  "<<setprecision(20)<<TOTEN<<"\t"<<molecs->NM<<endl;
        //  WRITE (*,*) 'Electronic Energy =',TOTELE
    }
    else{
        N=I;
        if(molecs->IPCELL[N] > 0){
            L=molecs->IPSP[N];
            TOTEN=TOTEN+gas->SP[6][L];
            TOTEN=TOTEN+0.5e00*gas->SP[5][L]*(pow(molecs->PV[1][N],2)+pow(molecs->PV[2][N],2)+pow(molecs->PV[3][N],2));
            if(gas->ISPR[1][L] > 0) TOTEN=TOTEN+molecs->PROT[N];
            if(gas->ISPV[L] > 0){
                for(KV=1;KV<=gas->ISPV[L];KV++){
                    J=molecs->IPVIB[KV][N];
                    //         IF (J <0) THEN
                    //           J=-J
                    //           IF (J == 99999) J=0
                    //         END IF
                    TOTEN=TOTEN+double(J)*BOLTZ*gas->SPVM[1][KV][L];
                }
            }
        }
    }
    
    //
    return;   //
}



void SETXT()
{
    //generate TECPLOT files for displaying an x-t diagram of an unsteady flow
    //this employs ordered data, therefore the cells MUST NOT BE ADAPTED
    //N.B. some custom coding for particular problems
    //
    //
    //MOLECS molecs;
    //CALC calc;
    //GEOM_1D geom;
    //GAS gas;
    //OUTPUT output;
    //
    
    // IMPLICIT NONE
    //
    int N,M,IOUT;
    double A,C;
    double **VALINT;
    // REAL(KIND=8), ALLOCATABLE, DIMENSION(:,:) :: VALINT
    //
    //VALINT(N,M) the interpolated values at sampling cell M boundaries and extrapolated values at boundaries
    //    N=1 distance
    //    N=2 time
    //    N=3 number density
    //    N=4 radial velocity
    //    N=5 pressure (nkT)
    //    N=6 temperature
    //    N=7 h2o fraction (Sec. 7.9 only)
    //
    //the variables in VALINT may be altered for particular problems
    //
    VALINT = new double*[7];
    for(int i =0; i< 7; ++i)
        VALINT[i] = new double[geom->NCELLS+2];
    
    // ALLOCATE (VALINT(6,NCELLS+1),STAT=ERROR)
    //
    //777 FORMAT(12G14.6)
    //24[]
    
    //Internal options
    IOUT=0;    //0 for dimensioned output, 1 for non-dimensional output
    //
    A=1.e00;   //dt/dt for selection of v velocity component in TECPLOT to draw particle paths as "streamlines"
    //
    if(calc->FTIME < 0.5e00*calc->DTM){
        //Headings and zero time record
        //        IF (ERROR /= 0) THEN
        //        WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR VALINT ARRAY',ERROR
        //        ENDIF
        calc->NLINE=1;
        file_9<< "J in tecplot file = "<<calc->NLINE*(geom->NCELLS+1)<<endl;
        //  WRITE (18,*) 'VARIABLES = "Distance","Time","n","u","p","T","H2O","A"'   //for combustion wave output(Sec. 7.9)
        
        file_18<<"VARIABLES = 'Distance','Time','n','u','p','T','A' "<<endl;
        file_18<<"ZONE I= "<<geom->NCELLS+1<<", J=  (set to number of output intervals+1), F=POINT"<<endl;
        //
        for(N=1;N<=geom->NCELLS+1;N++){
            VALINT[1][N]=geom->XB[1]+(N-1)*geom->DDIV;    //distance
            VALINT[1][N]=VALINT[1][N];         //time
            VALINT[2][N]=0.0;
            VALINT[3][N]=gas->FND[1];
            VALINT[4][N]=0;
            VALINT[5][N]=gas->FND[1]*BOLTZ*gas->FTMP[1];
            VALINT[6][N]=gas->FTMP[1];
            //   VALINT(7,N)=FSP(6,1)   //FSP(6 for combustion wave
            if((VALINT[1][N] > geom->XS) && (calc->ISECS == 1)){
                VALINT[3][N]=gas->FND[2];
                VALINT[5][N]=gas->FND[2]*BOLTZ*gas->FTMP[2];
                VALINT[6][N]=gas->FTMP[2];
                //      VALINT(7,N)=FSP(6,2)
            }
            if(IOUT == 1){
                VALINT[3][N]=1.e00;
                VALINT[5][N]=1.e00;
                VALINT[6][N]=1.e00;
            }
            for(M=1;M<=6;M++)
                file_18<<VALINT[M][N]<<"\t";//WRITE (18,777) (VALINT(M,N),M=1,6),A
            file_18<<A<<endl;
        }
    }
    else{
        calc->NLINE=calc->NLINE+1;
        cout<<"J in tecplot file = "<<calc->NLINE<<endl;
        if(geom->IVB == 0) C=geom->DDIV;
        if(geom->IVB == 1) C=(geom->XB[2]+geom->VELOB*calc->FTIME-geom->XB[1])/double(geom->NDIV);
        for(N=1;N<=geom->NCELLS+1;N++){
            VALINT[1][N]=geom->XB[1]+(N-1)*C;
            VALINT[2][N]=calc->FTIME;
            if((N > 1) && (N < geom->NCELLS+1)){
                VALINT[3][N]=0.5e00*(output->VAR[3][N]+output->VAR[3][N-1]);
                VALINT[4][N]=0.5e00*(output->VAR[5][N]+output->VAR[5][N-1]);
                VALINT[5][N]=0.5e00*(output->VAR[18][N]+output->VAR[18][N-1]);
                VALINT[6][N]=0.5e00*(output->VAR[11][N]+output->VAR[11][N-1]);
                //     VALINT(7,N)=0.5D00*(VARSP(1,N,6)+VARSP(1,N-1,6))   //H2O fraction for Sec 7.9
            }
        }
        for(N=3;N<=6;N++)
            VALINT[N][1]=0.5e00*(3.e00*VALINT[N][2]-VALINT[N][3]);
        
        //
        for(N=3;N<=6;N++)
            VALINT[N][geom->NCELLS+1]=0.5e00*(3.e00*VALINT[N][geom->NCELLS]-VALINT[N][geom->NCELLS-1]);
        
        //
        for(N=1;N<=geom->NCELLS+1;N++){
            if(IOUT == 1){
                VALINT[1][N]=(VALINT[1][N]-geom->XB[1])/(geom->XB[2]-geom->XB[1]);
                VALINT[2][N]=VALINT[2][N]/calc->TNORM;
                VALINT[3][N]=VALINT[3][N]/gas->FND[1];
                VALINT[4][N]=VALINT[4][N]/gas->VMPM;
                VALINT[5][N]=VALINT[5][N]/(gas->FND[1]*BOLTZ*gas->FTMP[1]);
                VALINT[6][N]=VALINT[6][N]/gas->FTMP[1];
            }
            for(M=1;M<=6;M++)
                file_18<<VALINT[M][N]<<"\t";//WRITE (18,777) (VALINT[M][N],M=1,6),A       //
            file_18<<A<<endl;
        }
    }
    //
    return;
}


void MOLECULES_MOVE_1D()
{//
    //molecule moves appropriate to the time step
    //for homogeneous and one-dimensional flows
    //(homogeneous flows are calculated as one-dimensional)
    //MOLECS molecs;
    //GAS gas;
    //GEOM_1D geom;
    //CALC calc;
    //OUTPUT output;
    //
    // IMPLICIT NONE
    //
    int N,L,M,K,NCI,J,II,JJ;
    double A,B,X,XI,XC,DX,DY,DZ,DTIM,S1,XM,R,TI,DTC,POB,UR,WFI,WFR,WFRI;
    //
    //N working integer
    //NCI initial cell time
    //DTIM time interval for the move
    //POB position of the outer boundary
    //TI initial time
    //DTC time interval to collision with surface
    //UR radial velocity component
    //WFI initial weighting factor
    //WFR weighting factor radius
    //WFRI initial weighting factor radius
    //
    if((geom->ITYPE[2] == 4) && (calc->ICN == 1)){
        //memset(calc->ALOSS,0.e00,sizeof(*calc->ALOSS));//calc->ALOSS=0.e00;
        for(int i=0;i<gas->MSP+1;i++)
            calc->ALOSS[i]=0.e00;
        
        calc->NMP=molecs->NM;
    }
    //
    N=1;

    while(N <= molecs->NM){
        //
        NCI=molecs->IPCELL[N];
        if((calc->IMTS == 0) || (calc->IMTS == 2)) DTIM=calc->DTM;
        if(calc->IMTS == 1) DTIM=2.e00*geom->CCELL[3][NCI];
        if(calc->FTIME-molecs->PTIM[N] > 0.5*DTIM){
            WFI=1.e00;
            if(geom->IWF == 1) WFI=1.e00+geom->WFM*pow(molecs->PX[1][N],geom->IFX);
            II=0; //becomes 1 if a molecule is removed
            TI=molecs->PTIM[N];
            molecs->PTIM[N]=TI+DTIM;
            calc->TOTMOV=calc->TOTMOV+1;
            //
            XI=molecs->PX[1][N];
            DX=DTIM*molecs->PV[1][N];
            X=XI+DX;
            //
            if(geom->IFX > 0){
                DY=0.e00;
                DZ=DTIM*molecs->PV[3][N];
                if(geom->IFX == 2) DY=DTIM*molecs->PV[2][N];
                R=sqrt(X*X+DY*DY+DZ*DZ);
            }
            //
            if(geom->IFX == 0){
                for(J=1;J<=2;J++){    // 1 for minimum x boundary, 2 for maximum x boundary
                    if(II == 0){
                        if(((J == 1) && (X < geom->XB[1])) || ((J == 2) && (X > (geom->XB[2]+geom->VELOB*molecs->PTIM[N])))){  //molecule crosses a boundary
                            if((geom->ITYPE[J] == 0) || (geom->ITYPE[J] == 3) || (geom->ITYPE[J] == 4)){
                                if(geom->XREM > geom->XB[1]){
                                    L=molecs->IPSP[N];
                                    calc->ENTMASS=calc->ENTMASS-gas->SP[5][L];
                                }
                                if((geom->ITYPE[2] == 4) && (calc->ICN == 1)){
                                    L=molecs->IPSP[N];
                                    calc->ALOSS[L]=calc->ALOSS[L]+1.e00;
                                }
                                REMOVE_MOL(N);
                                N=N-1;
                                II=1;
                            }
                            //
                            if(geom->ITYPE[J] == 1){
                                if((geom->IVB == 0) || (J == 1)){
                                    X=2.e00*geom->XB[J]-X;
                                    molecs->PV[1][N]=-molecs->PV[1][N];
                                }
                                else if((J == 2) && (geom->IVB == 1)){
                                    DTC=(geom->XB[2]+TI*geom->VELOB-XI)/(molecs->PV[1][N]-geom->VELOB);
                                    XC=XI+molecs->PV[1][N]*DTC;
                                    molecs->PV[1][N]=-molecs->PV[1][N]+2.*geom->VELOB;
                                    X=XC+molecs->PV[1][N]*(DTIM-DTC);
                                }
                            }
                            //
                            if(geom->ITYPE[J] == 2)
                                REFLECT_1D(N,J,X);
                            // END IF
                        }
                    }
                }
            }
            else{         //cylindrical or spherical flow
                //check boundaries
                if((X <geom-> XB[1]) && (geom->XB[1] > 0.e00)){
                    RBC(XI,DX,DY,DZ,geom->XB[1],S1);
                    if(S1 < 1.e00){     //intersection with inner boundary
                        if(geom->ITYPE[1] == 2){//solid surface
                            DX=S1*DX;
                            DY=S1*DY;
                            DZ=S1*DZ;
                            AIFX(XI,DX,DY,DZ,X,molecs->PV[1][N],molecs->PV[2][N],molecs->PV[3][N]);
                            REFLECT_1D(N,1,X);
                        }
                        else{
                            REMOVE_MOL(N);
                            N=N-1;
                            II=1;
                        }
                    }
                }
                else if((geom->IVB == 0) && (R > geom->XB[2])){
                    RBC(XI,DX,DY,DZ,geom->XB[2],S1);
                    if(S1 < 1.e00){     //intersection with outer boundary
                        if(geom->ITYPE[2] == 2){ //solid surface
                            DX=S1*DX;
                            DY=S1*DY;
                            DZ=S1*DZ;
                            AIFX(XI,DX,DY,DZ,X,molecs->PV[1][N],molecs->PV[2][N],molecs->PV[3][N]);
                            X=1.001e00*geom->XB[2];
                            while(X > geom->XB[2])
                                REFLECT_1D(N,2,X);
                            // END DO
                        }
                        else{
                            REMOVE_MOL(N);
                            N=N-1;
                            II=1;
                        }
                    }
                }
                else if((geom->IVB == 1) && (R > (geom->XB[2]+molecs->PTIM[N]*geom->VELOB))){
                    if(geom->IFX == 1) UR=sqrt(pow(molecs->PV[1][N],2)+pow(molecs->PV[2][N],2));
                    if(geom->IFX == 2) UR=sqrt(pow(molecs->PV[1][N],2)+pow(molecs->PV[2][N],2)+pow(molecs->PV[3][N],2));
                    DTC=(geom->XB[2]+TI*geom->VELOB-XI)/(UR-geom->VELOB);
                    S1=DTC/DTIM;
                    DX=S1*DX;
                    DY=S1*DY;
                    DZ=S1*DZ;
                    AIFX(XI,DX,DY,DZ,X,molecs->PV[1][N],molecs->PV[2][N],molecs->PV[3][N]);
                    molecs->PV[1][N]=-molecs->PV[1][N]+2.0*geom->VELOB;
                    X=X+molecs->PV[1][N]*(DTIM-DTC);
                }
                else
                    AIFX(XI,DX,DY,DZ,X,molecs->PV[1][N],molecs->PV[2][N],molecs->PV[3][N]);
                
                
                //DIAGNOSTIC
                if(II == 0){
                    if(X > geom->XB[2]+molecs->PTIM[N]*geom->VELOB){
                        //WRITE (*,*) N,calc->FTIME,X,geom->XB[2]+molecs->PTIM[N]*geom->VELOB;
                        cout<<N<<" "<<calc->FTIME<<" "<<X<<" "<<(geom->XB[2]+molecs->PTIM[N]*geom->VELOB)<<endl;
                    }
                }
                
                //Take action on weighting factors
                if((geom->IWF == 1) && (II == 0)){
                    WFR=WFI/(1.e00+geom->WFM*pow(X,geom->IFX));
                    L=0;
                    WFRI=WFR;
                    if(WFR >= 1.e00){
                        while(WFR >= 1.e00){
                            L=L+1;
                            WFR=WFR-1.e00;
                        }
                    }
                    // CALL RANDOM_NUMBER(RANF)
                    calc->RANF=((double)rand()/(double)RAND_MAX);
                    if(calc->RANF <= WFR) L=L+1;
                    if(L == 0){
                        REMOVE_MOL(N);
                        N=N-1;
                        II=1;
                    }
                    L=L-1;
                    if(L > 0){
                        for(K=1;K<=L;K++){
                            if(molecs->NM >= molecs->MNM) EXTEND_MNM(1.1);
                            molecs->NM=molecs->NM+1;
                            molecs->PX[1][molecs->NM]=X;
                            for(M=1;M<=3;M++)
                                molecs->PV[M][molecs->NM]=molecs->PV[M][N];
                            
                            if(gas->MMRM > 0) molecs->PROT[molecs->NM]=molecs->PROT[N];
                            molecs->IPCELL[molecs->NM]=fabs(molecs->IPCELL[N]);
                            molecs->IPSP[molecs->NM]=molecs->IPSP[N];
                            molecs->IPCP[molecs->NM]=molecs->IPCP[N];
                            if(gas->MMVM > 0){
                                for(M=1;M<=gas->MMVM;M++)
                                    molecs->IPVIB[M][molecs->NM]=molecs->IPVIB[M][N];
                                
                            }
                            molecs->PTIM[molecs->NM]=molecs->PTIM[N];    //+5.D00*DFLOAT(K)*DTM
                            //note the possibility of a variable time advance that may take the place of the duplication buffer in earlier programs
                            
                            if(molecs->PX[1][molecs->NM] > geom->XB[2]+molecs->PTIM[molecs->NM]*geom->VELOB)
                                //WRITE (*,*) 'DUP',NM,FTIME,PX(1,NM),XB(2)+PTIM(NM)*VELOB
                                cout<<"DUP "<<molecs->NM<<" "<<calc->FTIME<<" "<<molecs->PX[1][molecs->NM]<<" "<<(geom->XB[2]+molecs->PTIM[molecs->NM]*geom->VELOB)<<endl;
                            
                        }
                    }
                }
            }
            //
            if(II == 0){
                molecs->PX[1][N]=X;
                
                if(molecs->PX[1][N] > geom->XB[1] && (molecs->PX[1][N] < geom->XB[2]))
                    continue;
                else{
                    cout<< N<<" OUTSIDE FLOWFIELD AT "<<molecs->PX[1][N]<<" VEL "<<molecs->PV[1][N]<<endl;
                    REMOVE_MOL(N);
                    N=N-1;
                    II=1;
                }
            }
            //
            if(II == 0){
                if(geom->IVB == 0) FIND_CELL_1D(molecs->PX[1][N],molecs->IPCELL[N],JJ);
                if(geom->IVB == 1) FIND_CELL_MB_1D(molecs->PX[1][N],molecs->IPCELL[N],JJ,molecs->PTIM[N]);
            }
            //
        }
        //
        N=N+1;
    }
    //
    return;
}



void READ_RESTART()
{
    //MOLECS molecs;
    //GEOM_1D geom;
    //GAS gas;
    //CALC calc;
    //OUTPUT output;
    // IMPLICIT NONE
    //
    fstream file_7;
    int ZCHECK;
    //
    //    101 CONTINUE
    _101:
    file_7.open("PARAMETERS.DAT", ios::in | ios::binary);
    if(file_7.is_open()){
        cout<<"PARAMETERS.DAT opened successfully"<<endl;
        file_7>>geom->NCCELLS>>geom->NCELLS>>gas->MMRM>>gas->MMVM>>molecs->MNM>>gas->MNSR>>gas->MSP>>geom->ILEVEL>>geom->MDIV>>gas->MMEX>>gas->MEX>>gas->MELE>>gas->MVIBL>>calc->NCLASS;
        file_7.close();
    }
    else{
        cout<<"PARAMETERS.DAT not opening"<<endl;
        goto _101;
    }
    //cout<<geom->NCCELLS<<endl<<geom->NCELLS<<endl<<gas->MMRM<<endl<<gas->MMVM<<endl<<molecs->MNM<<endl;
    // OPEN (7,FILE='PARAMETERS.DAT',FORM='BINARY',ERR=101)
    // READ (7) NCCELLS,NCELLS,MMRM,MMVM,MNM,MNSR,MSP,ILEVEL,MDIV,MMEX,MEX,MELE,MVIBL,NCLASS
    // CLOSE(7)
    //
    if(gas->MMVM > 0){
        
        molecs->d_allocate(calc->NCLASS+1,molecs->MNM+1,molecs->PX);
        molecs->d_allocate(molecs->MNM+1,molecs->PTIM);
        molecs->d_allocate(molecs->MNM+1,molecs->PROT);
        molecs->i_allocate(molecs->MNM+1,molecs->IPCELL);
        molecs->i_allocate(molecs->MNM+1,molecs->IPSP);
        molecs->i_allocate(molecs->MNM+1,molecs->ICREF);
        molecs->i_allocate(molecs->MNM+1,molecs->IPCP);
        molecs->d_allocate(4,molecs->MNM+1,molecs->PV);
        molecs->i_allocate(gas->MMVM+1,molecs->MNM+1,molecs->IPVIB);
        molecs->d_allocate(molecs->MNM+1,molecs->PELE);
        // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),PROT(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),      &
        //      IPVIB(MMVM,MNM),PELE(MNM),STAT=ERROR)
    }
    else{
        if(gas->MMRM > 0){
            molecs->d_allocate(calc->NCLASS+1,molecs->MNM+1,molecs->PX);
            molecs->d_allocate(molecs->MNM+1,molecs->PTIM);
            molecs->d_allocate(molecs->MNM+1,molecs->PROT);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCELL);
            molecs->i_allocate(molecs->MNM+1,molecs->IPSP);
            molecs->i_allocate(molecs->MNM+1,molecs->ICREF);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCP);
            molecs->d_allocate(4,molecs->MNM+1,molecs->PV);
            molecs->d_allocate(molecs->MNM+1,molecs->PELE);
            // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),PROT(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),PELE(MNM),STAT=ERROR)
        }
        else{
            molecs->d_allocate(calc->NCLASS+1,molecs->MNM+1,molecs->PX);
            molecs->d_allocate(molecs->MNM+1,molecs->PTIM);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCELL);
            molecs->i_allocate(molecs->MNM+1,molecs->IPSP);
            molecs->i_allocate(molecs->MNM+1,molecs->ICREF);
            molecs->i_allocate(molecs->MNM+1,molecs->IPCP);
            molecs->d_allocate(4,molecs->MNM+1,molecs->PV);
            molecs->d_allocate(molecs->MNM+1,molecs->PELE);
            // ALLOCATE (PX(NCLASS,MNM),PTIM(MNM),IPCELL(MNM),IPSP(MNM),ICREF(MNM),IPCP(MNM),PV(3,MNM),PELE(MNM),STAT=ERROR)
        }
        
    }
     
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR MOLECULE ARRAYS',ERROR
    // ENDIF
    //
    geom->i_allocate(geom->ILEVEL+1,geom->MDIV+1,geom->JDIV);
    // ALLOCATE (JDIV(0:ILEVEL,MDIV),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR JDIV ARRAY',ERROR
    // ENDIF
    geom->d_allocate(5,geom->NCELLS+1,geom->CELL);
    geom->i_allocate(geom->NCELLS+1,geom->ICELL);
    geom->d_allocate(6,geom->NCCELLS+1,geom->CCELL);
    geom->i_allocate(4,geom->NCCELLS+1,geom->ICCELL);
    // ALLOCATE (CELL(4,NCELLS),ICELL(NCELLS),CCELL(5,NCCELLS),ICCELL(3,NCCELLS),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR CELL ARRAYS',ERROR
    // ENDIF
    
    output->d_allocate(geom->NCELLS+1,output->COLLS);
    output->d_allocate(geom->NCELLS+1,output->WCOLLS);
    output->d_allocate(geom->NCELLS+1,output->CLSEP);
    output->d_allocate(gas->MNSR+1,output->SREAC);
    output->d_allocate(24,geom->NCELLS+1,output->VAR);
    output->d_allocate(13,geom->NCELLS+1,gas->MSP+1,output->VARSP);
    output->d_allocate(36+gas->MSP,3,output->VARS);
    output->d_allocate(10+gas->MSP,geom->NCELLS+1,gas->MSP+1,output->CS);
    output->d_allocate(9,3,gas->MSP+1,3,output->CSS);
    output->d_allocate(7,3,output->CSSS);
    // ALLOCATE (COLLS(NCELLS),WCOLLS(NCELLS),CLSEP(NCELLS),SREAC(MNSR),VAR(23,NCELLS),    &
    //           VARSP(0:12,NCELLS,MSP),VARS(0:35+MSP,2),CS(0:9+MSP,NCELLS,MSP),CSS(0:8,2,MSP,2),CSSS(6,2),STAT=ERROR)
    // IF (ERROR /= 0) THEN
    //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR SAMPLING ARRAYS',ERROR
    // ENDIF
    //

    if(gas->MMVM >= 0){
        output->d_allocate(gas->MSP+1,gas->MMVM+1,151,output->VIBFRAC);
        output->d_allocate(gas->MSP+1,gas->MMVM+1,output->SUMVIB);
        // ALLOCATE (VIBFRAC(MSP,MMVM,0:150),SUMVIB(MSP,MMVM),STAT=ERROR)
        // IF (ERROR /= 0) THEN
        //   WRITE (*,*) 'PROGRAM COULD NOT ALLOCATE SPACE FOR RECOMBINATION ARRAYS',ERROR
        // ENDIF
    }
    //
    ALLOCATE_GAS();
    //
    //102 CONTINU
_102:
    file_7.open("RESTART.DAT", ios::in | ios::binary);
    if(file_7.is_open()){
        cout<<"RESTART.DAT opened successfully"<<endl;
        /*file_7>>calc->AJM>>calc->ALOSS>>output->AVDTM>>BOLTZ>>geom->CCELL>>geom->CELL>>output->CLSEP>>output->COLLS>>calc->CPDTM>>gas->CR>>output->CS>>output->CSS>>output->CSSS>>gas->CTM>>gas->CXSS>>geom->DDIV>>DPI>>calc->DTM>>calc->DTSAMP>>calc->DTOUT>>calc->EME>>calc->ENTMASS>>gas->ENTR>>calc->ENTREM>>calc->ERROR>>gas->ERS>>gas->FDEN>>gas->FMA>>gas->FND>>calc->FNUM>>calc->FRACSAM>>gas->FSP>>gas->FP>>gas->FPM>>gas->FPR>>geom->FREM>>gas->FSPEC>>gas->FTMP>>calc->FTIME>>gas->FVTMP>>geom->ICCELL>>geom->ICELL>>calc->ICLASS>>calc->ICN>>molecs->ICREF>>geom->IFX>>gas->IGAS>>calc->IMTS>>molecs->IPCELL>>molecs->IPCP>>molecs->IPSP>>molecs->IPVIB>>calc->IREM>>calc->ISAD>>calc->ISECS>>calc->ISF>>gas->ISPEX>>gas->ISPR>>gas->ISPRC>>gas->ISPRK>>gas->ISPV>>gas->ISPVM>>gas->ISRCD>>geom->ITYPE>>geom->IVB>>geom->IWF>>geom->JDIV>>gas->LIS>>gas->LRS>>calc->MOLSC>>calc->MVER>>geom->NCCELLS>>geom->NCELLS>>geom->NCIS>>geom->NDIV>>gas->NELL>>gas->NEX>>calc->NLINE>>molecs->NM>>output->NMISAMP>>calc->NNC>>output->NOUT>>output->NSAMP>>gas->NSLEV>>gas->NSPEX>>calc->NREL>>calc->NVER>>molecs->PELE>>PI>>molecs->PROT>>molecs->PTIM>>molecs->PV>>molecs->PX>>gas->QELC>>gas->RGFS>>gas->RMAS>>gas->SLER>>gas->SP>>gas->SPEX>>SPI>>gas->SPM>>gas->SPR>>gas->SPRC>>gas->SPREX>>gas->SPRP>>gas->SPRT>>gas->SPV>>gas->SPVM>>output->SREAC>>output->SUMVIB>>calc->TCOL>>calc->TDISS>>calc->TRECOMB>>output->TISAMP>>calc->TPOUT>>calc->TREF>>calc->TLIM>>calc->TOTCOL>>calc->TOTMOV>>gas->TREACG>>gas->TREACL>>calc->TOUT>>calc->TPDTM>>calc->TREF>>calc->TSAMP>>gas->TSURF>>output->VAR>>output->VARS>>output->VARSP>>geom->VELOB>>gas->VFX>>gas->VFY>>output->VIBFRAC>>gas->VMP>>gas->VMPM>>calc->VNMAX>>gas->VSURF>>output->WCOLLS>>geom->WFM>>geom->XB>>geom->XREM>>output->XVELS>>output->YVELS>>gas->TNEX>>ZCHECK>>endl;*/
        file_7.read((char*)&calc,sizeof(calc));
        file_7.read((char*)&molecs,sizeof(molecs));
        file_7.read((char*)&gas,sizeof(gas));
        file_7.read((char*)&geom,sizeof(geom));
        file_7.read((char*)&output,sizeof(output));
        file_7.close();
    }
    else{
        cout<<"Restart.DAT not opening"<<endl;
        goto _102;
    }
    // OPEN (7,FILE='RESTART.DAT',FORM='BINARY',ERR=102)
    // READ (7) AJM,ALOSS,AVDTM,BOLTZ,CCELL,CELL,CLSEP,COLLS,    &
    //          CPDTM,CR,CS,CSS,CSSS,CTM,CXSS,DDIV,DPI,DTM,DTSAMP,DTOUT,EME,      &
    //          ENTMASS,ENTR,ENTREM,ERROR,ERS,FDEN,FMA,FND,FNUM,FRACSAM,FSP,FP,FPM,FPR,FREM,FSPEC,     &
    //          FTMP,FTIME,FVTMP,ICCELL,ICELL,ICLASS,ICN,ICREF,IFX,IGAS,IMTS,IPCELL,IPCP,     &
    //          IPSP,IPVIB,IREM,ISAD,ISECS,ISF,ISPEX,ISPR,ISPRC,ISPRK,ISPV,ISPVM,ISRCD,ITYPE,IVB,IWF,     &
    //          JDIV,LIS,LRS,MOLSC,MVER,NCCELLS,NCELLS,    &
    //          NCIS,NDIV,NELL,NEX,NLINE,NM,NMISAMP,NNC,NOUT,NSAMP,NSLEV,NSPEX,NREL,NVER,PELE,PI,PROT,PTIM,PV,PX,     &
    //          QELC,RGFS,RMAS,SLER,SP,SPEX,SPI,SPM,SPR,SPRC,SPREX,SPRP,SPRT,SPV,SPVM,SREAC,SUMVIB,    &
    //          TCOL,TDISS,TRECOMB,TISAMP,TPOUT,TREF,TLIM,TOTCOL,TOTMOV,     &
    //          TREACG,TREACL,TOUT,TPDTM,TREF,TSAMP,TSURF,VAR,VARS,VARSP,VELOB,VFX,VFY,VIBFRAC,VMP,     &
    //          VMPM,VNMAX,VSURF,WCOLLS,WFM,XB,XREM,XVELS,YVELS,TNEX,ZCHECK
    // //
    // CLOSE(7)
    //
    if(ZCHECK != 1234567){
        file_9<<molecs->NM<<" Molecules, Check integer = "<<ZCHECK<<endl;
        //WRITE (9,*) NM,' Molecules, Check integer =',ZCHECK
        return ;
    }
    else
        file_9<<"Restart file read, Check integer= "<<ZCHECK<<endl;
    //WRITE (9,*) 'Restart file read, Check integer=',ZCHECK
    
    //
    return;
    
    //
}
//*****************************************************************************

void WRITE_RESTART()
{
    //MOLECS molecs;
    //GEOM_1D geom;
    //GAS gas;
    //CALC calc;
    //OUTPUT output;
    // IMPLICIT NONE
    //
    int ZCHECK;
    //
    fstream file_7;
    ZCHECK=1234567;
    //
    //101 CONTINUE
_101:
    file_7.open("PARAMETERS.DAT", ios::out | ios::binary);
    if(file_7.is_open()){
        file_7<<geom->NCCELLS<<endl<<geom->NCELLS<<endl<<gas->MMRM<<endl<<gas->MMVM<<endl<<molecs->MNM<<endl<<gas->MNSR<<endl<<gas->MSP<<endl<<geom->ILEVEL<<endl<<geom->MDIV<<endl<<gas->MMEX<<endl<<gas->MEX<<endl<<gas->MELE<<endl<<gas->MVIBL<<endl<<calc->NCLASS<<endl;
        file_7.close();
    }
    else{
        cout<<"Parameters.DAT file not opening(write)"<<endl;
        goto _101;
    }
    // OPEN (7,FILE='PARAMETERS.DAT',FORM='BINARY',ERR=101)
    // WRITE (7) NCCELLS,NCELLS,MMRM,MMVM,MNM,MNSR,MSP,ILEVEL,MDIV,MMEX,MEX,MELE,MVIBL,NCLASS
    // CLOSE(7)
    //
    //    102 CONTINUE
_102:
    file_7.open("RESTART.DAT", ios::out | ios::binary);
    if(file_7.is_open()){
        /*file_7<<calc->AJM<<calc->ALOSS<<output->AVDTM<<BOLTZ<<geom->CCELL<<geom->CELL<<output->CLSEP<<output->COLLS<<calc->CPDTM<<gas->CR<<output->CS<<output->CSS<<output->CSSS<<gas->CTM<<gas->CXSS<<geom->DDIV<<DPI<<calc->DTM<<calc->DTSAMP<<calc->DTOUT<<calc->EME<<calc->ENTMASS<<gas->ENTR<<calc->ENTREM<<calc->ERROR<<gas->ERS<<gas->FDEN<<gas->FMA<<gas->FND<<calc->FNUM<<calc->FRACSAM<<gas->FSP<<gas->FP<<gas->FPM<<gas->FPR<<geom->FREM<<gas->FSPEC<<gas->FTMP<<calc->FTIME<<gas->FVTMP<<geom->ICCELL<<geom->ICELL<<calc->ICLASS<<calc->ICN<<molecs->ICREF<<geom->IFX<<gas->IGAS<<calc->IMTS<<molecs->IPCELL<<molecs->IPCP<<molecs->IPSP<<molecs->IPVIB<<calc->IREM<<calc->ISAD<<calc->ISECS<<calc->ISF<<gas->ISPEX<<gas->ISPR<<gas->ISPRC<<gas->ISPRK<<gas->ISPV<<gas->ISPVM<<gas->ISRCD<<geom->ITYPE<<geom->IVB<<geom->IWF<<geom->JDIV<<gas->LIS<<gas->LRS<<calc->MOLSC<<calc->MVER<<geom->NCCELLS<<geom->NCELLS<<geom->NCIS<<geom->NDIV<<gas->NELL<<gas->NEX<<calc->NLINE<<molecs->NM<<output->NMISAMP<<calc->NNC<<output->NOUT<<output->NSAMP<<gas->NSLEV<<gas->NSPEX<<calc->NREL<<calc->NVER<<molecs->PELE<<PI<<molecs->PROT<<molecs->PTIM<<molecs->PV<<molecs->PX<<gas->QELC<<gas->RGFS<<gas->RMAS<<gas->SLER<<gas->SP<<gas->SPEX<<SPI<<gas->SPM<<gas->SPR<<gas->SPRC<<gas->SPREX<<gas->SPRP<<gas->SPRT<<gas->SPV<<gas->SPVM<<output->SREAC<<output->SUMVIB<<calc->TCOL<<calc->TDISS<<calc->TRECOMB<<output->TISAMP<<calc->TPOUT<<calc->TREF<<calc->TLIM<<calc->TOTCOL<<calc->TOTMOV<<gas->TREACG<<gas->TREACL<<calc->TOUT<<calc->TPDTM<<calc->TREF<<calc->TSAMP<<gas->TSURF<<output->VAR<<output->VARS<<output->VARSP<<geom->VELOB<<gas->VFX<<gas->VFY<<output->VIBFRAC<<gas->VMP<<gas->VMPM<<calc->VNMAX<<gas->VSURF<<output->WCOLLS<<geom->WFM<<geom->XB<<geom->XREM<<output->XVELS<<output->YVELS<<gas->TNEX<<ZCHECK<<endl;*/
        file_7.write((char*)&calc,sizeof(calc));
        file_7.write((char*)&molecs,sizeof(molecs));
        file_7.write((char*)&gas,sizeof(gas));
        file_7.write((char*)&geom,sizeof(geom));
        file_7.write((char*)&output,sizeof(output));
        file_7.close();
    }
    else{
        cout<<"Restart.DAT file not opening(write)"<<endl;
        goto _101;
    }
    // OPEN (7,FILE='RESTART.DAT',FORM='BINARY',ERR=102)
    // WRITE (7)AJM,ALOSS,AVDTM,BOLTZ,CCELL,CELL,CLSEP,COLLS,    &
    //          CPDTM,CR,CS,CSS,CSSS,CTM,CXSS,DDIV,DPI,DTM,DTSAMP,DTOUT,EME,      &
    //          ENTMASS,ENTR,ENTREM,ERROR,ERS,FDEN,FMA,FND,FNUM,FRACSAM,FSP,FP,FPM,FPR,FREM,FSPEC,     &
    //          FTMP,FTIME,FVTMP,ICCELL,ICELL,ICLASS,ICN,ICREF,IFX,IGAS,IMTS,IPCELL,IPCP,     &
    //          IPSP,IPVIB,IREM,ISAD,ISECS,ISF,ISPEX,ISPR,ISPRC,ISPRK,ISPV,ISPVM,ISRCD,ITYPE,IVB,IWF,     &
    //          JDIV,LIS,LRS,MOLSC,MVER,NCCELLS,NCELLS,    &
    //          NCIS,NDIV,NELL,NEX,NLINE,NM,NMISAMP,NNC,NOUT,NSAMP,NSLEV,NSPEX,NREL,NVER,PELE,PI,PROT,PTIM,PV,PX,     &
    //          QELC,RGFS,RMAS,SLER,SP,SPEX,SPI,SPM,SPR,SPRC,SPREX,SPRP,SPRT,SPV,SPVM,SREAC,SUMVIB,    &
    //          TCOL,TDISS,TRECOMB,TISAMP,TPOUT,TREF,TLIM,TOTCOL,TOTMOV,     &
    //          TREACG,TREACL,TOUT,TPDTM,TREF,TSAMP,TSURF,VAR,VARS,VARSP,VELOB,VFX,VFY,VIBFRAC,VMP,     &
    //          VMPM,VNMAX,VSURF,WCOLLS,WFM,XB,XREM,XVELS,YVELS,TNEX,ZCHECK
    // //
    // CLOSE(7)
    //
    file_9<<"Restart files written"<<endl;
    //WRITE (9,*) 'Restart files written'
    //
    return;
}

void OUTPUT_RESULTS()
{
    //--calculate the surface and flowfield properties
    //--generate TECPLOT files for displaying these properties
    //--calculate collisiion rates and flow transit times and reset time intervals
    //--add molecules to any flow plane molecule output files
    //CALC calc;
    //MOLECS molecs;
    //GAS gas;
    //OUTPUT output;
    //GEOM_1D geom;
    
    fstream file_3;
    fstream file_10;
    fstream file_7;
    
    int IJ,J,JJ,K,L,LL,M,N,NN,NMCR,CTIME,II;
    long long NNN;
    double AS,AT,C1,C2,C3,C4,C5,C6,C7,C8,C9;
    double A,B,C,SDTM,SMCR,DOF,AVW,UU,VDOFM,TVIBM,VEL,DTMI,TT;
    //dout
    double SUM[14];
    double SUMS[10][3];
    double *TVIB,*VDOF,*PPA,*TEL,*ELDOF,*SDOF,*CDTM;
    double **TV,**THCOL;
    double ***DF;
    int *NMS;
    //    REAL(KIND=8), ALLOCATABLE, DIMENSION(:) :: TVIB,VDOF,PPA,TEL,ELDOF,SDOF,CDTM
    //    REAL(KIND=8), ALLOCATABLE, DIMENSION(:,:) :: TV,THCOL
    //    REAL(KIND=8), ALLOCATABLE, DIMENSION(:,:,:) :: DF
    //    INTEGER, ALLOCATABLE, DIMENSION(:) :: NMS
    //INTEGER, ALLOCATABLE, DIMENSION(:,:) ::
    string F,E;
    //--CTIME  computer time (microseconds)
    //--SUMS(N,L) sum over species of CSS(N,J,L,M) for surface properties
    //
    //--For flowfield properties,where <> indicates sampled sum
    //--SUM(0) the molecular number sum over all species
    //--SUM(1) the weighted number sum over all species
    //--SUM(2) the weighted sum of molecular masses
    //--SUM(3),(4),(5) the weighted sum over species of m*<u>,<v>,<w>
    //--SUM(6) the weighted sum over species of m*(<u**2>+<v**2>+<w**2>)
    //--SUM(7) the weighted sum over species of <u**2>+<v**2>+<w**2>
    //--SUM(8) the weighted sum of rotational energy
    //--SUM(9) the weighted sum of rotational degrees of freedom
    //--SUM(10) the weighted sum over species of m*<u**2>
    //--SUM(11) the weighted sum over species of m*<v**2>
    //--SUM(12) sum over species of m*<w**2>
    //--SUM(13) the weighted sum of electronic energy
    //--UU velocity squared
    //--DOF degrees of freedom
    //--AVW the average value of the viscosity-temperature exponent
    //--DVEL velocity difference
    //--TVEL thermal speed
    //--SMCR sum of mcs/mfp over cells
    //--NMCR number in the sum
    //--VDOFM effective vibrational degrees of freedom of mixture
    //--TVIB(L)
    //--VDOF(L)
    //--TV(K,L) the temperature of vibrational mode K of species L
    //--PPA particles per atom
    //--NMS number per species
    //--SDOF(L) total degrees of freedom for species L
    //
    //
    //--calculate the flowfield properties in the cells
    //dout
    
    
    TV = new double*[gas->MMVM+1];
    for(int i =0; i< gas->MMVM+1; ++i)
        TV[i] = new double[gas->MSP+1];
    
    TVIB = new double[gas->MSP+1];
    
    DF = new double **[geom->NCELLS+1];
    for (int i = 0; i < geom->NCELLS+1; ++i)
    {
        DF[i] = new double *[gas->MMVM+1];
        for (int j = 0; j < gas->MMVM+1; ++j)
            DF[i][j] = new double [gas->MSP+1];
    }
    
    VDOF= new double[gas->MSP+1];
    
    TEL = new double[gas->MSP+1];
    
    ELDOF = new double[gas->MSP+1];
    
    PPA = new double[gas->MSP+1];
    
    NMS = new int[gas->MSP+1];
    
    THCOL = new double*[gas->MSP+1];
    for(int i =0; i< gas->MSP+1; ++i)
        THCOL[i] = new double[gas->MSP+1];
    
    SDOF = new double[gas->MSP+1];
    
    CDTM = new double[geom->NCELLS+1];
    
    
    //    ALLOCATE (TV(MMVM,MSP),TVIB(MSP),DF(NCELLS,MMVM,MSP),VDOF(MSP),TEL(MSP),ELDOF(MSP),PPA(MSP),NMS(MSP),THCOL(MSP,MSP)    &
    //              ,SDOF(MSP),CDTM(NCELLS),STAT=ERROR)
    //    if(calc->ERROR!=0)
    //    {
    //        cout<<"ROGRAM COULD NOT ALLOCATE OUTPUT VARIABLES"<<calc->ERROR<<endl;
    //    }
    if(calc->FTIME>0.5e00*calc->DTM)
    {
        output->NOUT+=1;
        if(output->NOUT>9999)
            output->NOUT=output->NOUT-9999;
        cout<<"Generating files for output interval"<<output->NOUT<<endl;
        if(calc->ISF==0)
        {
            //dout
            //OPEN (3,FILE='DS1OUT.DAT')
            file_3.open("DS1OUT.DAT" , ios::out);
            if(file_3.is_open()){
                cout<<"DS1OUT.DAT is opened"<<endl;
            }
            else{
                cout<<"DS1OUT.DAT not opened"<<endl;
            }
            //F='DS';//E//'.OUT'
        }
        else
        {
            //--the files are DS1n.DAT, where n is a four digit integer equal to NOUT
            //dout
            //500 FORMAT(I5)
            //ENCODE(5,500,E) 10000+NOUT
            int a=output->NOUT+10000;
            E=to_string(a);
            F="DS" + E + "OUT.DAT";
            //dout
            file_3.open(F.c_str(), ios::out);
            if(file_3.is_open()){
                cout<<F<<" is opened"<<endl;
            }
            else{
                cout<<F<<" not opened"<<endl;
            }
            //OPEN (3,FILE=F)
        }
    }
    //dout
    //memset(output->VAR,0.e00,sizeof(**output->VAR));
    for(int i=0;i<24;i++){
        for(int j=0;j<geom->NCELLS+1;j++)
            output->VAR[i][j]=0.e00;
    }
    if(geom->IFX==0)
        A=calc->FNUM/(calc->FTIME-output->TISAMP);
    for(JJ=1;JJ<=2;JJ++)
    {
        if(geom->IFX==1)
            A=calc->FNUM/(2.e00*PI*geom->XB[JJ])*(calc->FTIME-output->TISAMP);
        if(geom->IFX==2)
            A=calc->FNUM/(4.e00*PI*geom->XB[JJ])*geom->XB[JJ]*(calc->FTIME-output->TISAMP);
        //--JJ=1 for surface at XB(1), JJ=2 for surface at XB(2)
        if(geom->ITYPE[JJ]==2)
        {
            //dout
            //memset(SUMS,0.e00,sizeof(SUMS));
            for(int i=0;i<10;i++){
                for(int j=0;j<3;j++)
                    SUMS[i][j]=0.e00;
            }
            for( L=1;L<=gas->MSP;L++)
            {
                for(J=0;J<=8;J++)
                {
                    for(IJ=1;IJ<=2;IJ++)
                    {
                        SUMS[J][IJ]=SUMS[J][IJ]+output->CSS[J][JJ][L][IJ];
                    }
                }
            }
            output->VARS[0][JJ]=SUMS[0][1];
            output->VARS[1][JJ]=SUMS[1][1];
            output->VARS[2][JJ]=SUMS[1][2];
            output->VARS[3][JJ]=SUMS[1][1]*A;
            output->VARS[4][JJ]=SUMS[1][2]*A;
            output->VARS[5][JJ]=SUMS[2][1]*A;
            output->VARS[6][JJ]=SUMS[2][2]*A;
            output->VARS[7][JJ]=SUMS[3][1]*A;
            output->VARS[8][JJ]=SUMS[3][2]*A;
            output->VARS[9][JJ]=SUMS[4][1]*A;
            output->VARS[10][JJ]=SUMS[4][2]*A;
            output->VARS[11][JJ]=SUMS[5][1]*A;
            output->VARS[12][JJ]=SUMS[5][2]*A;
            output->VARS[13][JJ]=SUMS[6][1]*A;
            output->VARS[14][JJ]=SUMS[6][2]*A;
            output->VARS[15][JJ]=SUMS[7][1]*A;
            output->VARS[16][JJ]=SUMS[7][2]*A;
            output->VARS[33][JJ]=SUMS[8][1]*A;
            output->VARS[34][JJ]=SUMS[8][2]*A;
            //   VARS(17,JJ)=SUMS(9,1)*A        //--SURFACE REACTIONS NOT YET IMPLEMENTED
            //   VARS(18,JJ)=SUMS(9,2)*A
            if(output->CSSS[1][JJ]>1.e-6)
            {
                output->VARS[19][JJ]=output->CSSS[3][JJ]/output->CSSS[2][JJ]; ////--n.b. must be modified to include second component in 3D
                output->VARS[20][JJ]=(output->CSSS[4][JJ]-output->CSSS[2][JJ]*output->VARS[19][JJ]*output->VARS[19][JJ])/(output->CSSS[1][JJ]*3.e00*BOLTZ)-gas->TSURF[JJ];
                output->VARS[19][JJ]=output->VARS[19][JJ]-gas->VSURF[JJ];
                if(output->CSSS[6][JJ]>1.e-6)
                {
                    output->VARS[21][JJ]=(2.e000/BOLTZ)*(output->CSSS[5][JJ]/output->CSSS[6][JJ])-gas->TSURF[JJ];
                }
                else
                {
                    output->VARS[21][JJ]=0.e00;
                }
            }
            else
            {
                output->VARS[19][JJ]=0.e00;
                output->VARS[20][JJ]=0.e00;
                output->VARS[21][JJ]=0.e00;
            }
            output->VARS[22][JJ]=(SUMS[2][1]+SUMS[2][2])*A;
            output->VARS[23][JJ]=(SUMS[3][1]+SUMS[3][2])*A;
            output->VARS[24][JJ]=(SUMS[4][1]+SUMS[4][2])*A;
            output->VARS[25][JJ]=(SUMS[5][1]+SUMS[5][2])*A;
            output->VARS[26][JJ]=(SUMS[6][1]+SUMS[6][2])*A;
            output->VARS[27][JJ]=(SUMS[7][1]+SUMS[7][2])*A;
            output->VARS[28][JJ]=(SUMS[9][1]+SUMS[9][2])*A;
            output->VARS[29][JJ]=output->VARS[11][JJ]+output->VARS[13][JJ]+output->VARS[15][JJ]+output->VARS[33][JJ];
            output->VARS[30][JJ]=output->VARS[12][JJ]+output->VARS[14][JJ]+output->VARS[16][JJ]+output->VARS[34][JJ];
            output->VARS[31][JJ]=output->VARS[29][JJ]+output->VARS[30][JJ];
            output->VARS[35][JJ]=output->VARS[33][JJ]+output->VARS[34][JJ];
            for(L=1;gas->MSP;L++)
            {
                if(SUMS[1][1]>0)
                {
                    output->VARS[35+L][JJ]=100*output->CSS[1][JJ][L][1]/SUMS[1][1];
                }
                else
                {
                    output->VARS[35+L][JJ]=0.0;
                }
            }
        }
    }
    //output->VARSP=0;
    for(int i=0;i<13;i++){
        for(int j=0;j<geom->NCELLS+1;j++){
            for(int k=0;k<gas->MSP+1;k++)
                output->VARSP[i][j][k]=0;
        }
    }
    SMCR=0;
    NMCR=0;
    for(N=1;N<=geom->NCELLS;N++)
    {
        if(N==120)
        {
            continue;
        }
        A=calc->FNUM/(geom->CELL[4][N])*output->NSAMP;
        if(geom->IVB==1)
            A=A*pow((geom->XB[2]-geom->XB[1])/(geom->XB[2]+geom->VELOB*0.5e00*(calc->FTIME-output->TISAMP)-geom->XB[1]),geom->IFX+1);
        //--check the above for non-zero XB(1)
        //dout
        //memset(SUM,0,sizeof(SUM));
        for(int i=0;i<14;i++)
            SUM[i]=0;
        
        NMCR+=1;
        for(L=1;L<=gas->MSP;L++)
        {
            SUM[0]=SUM[0]+output->CS[0][N][L];
            SUM[1]=SUM[1]+output->CS[1][N][L];
            SUM[2]=SUM[2]+gas->SP[5][L]*output->CS[0][N][L];
            for(K=1;K<=3;K++)
            {
                SUM[K+2]=SUM[K+2]+gas->SP[5][L]*output->CS[K+1][N][L];
                if(output->CS[1][N][L]>1.1e00)
                {
                    output->VARSP[K+1][N][L]=output->CS[K+4][N][L]/output->CS[1][N][L];
                    //--VARSP(2,3,4 are temporarily the mean of the squares of the velocities
                    output->VARSP[K+8][N][L]=output->CS[K+1][N][L]/output->CS[1][N][L];
                }
            }
            SUM[6]=SUM[6]+gas->SP[5][L]*(output->CS[5][N][L]+output->CS[6][N][L]+output->CS[7][N][L]);
            SUM[10]=SUM[10]+gas->SP[5][L]*output->CS[5][N][L];
            SUM[12]=SUM[11]+gas->SP[5][L]*output->CS[6][N][L];
            SUM[12]=SUM[12]+gas->SP[5][L]*output->CS[7][N][L];
            SUM[13]=SUM[13]+output->CS[9][N][L];
            if(output->CS[1][N][L]>0.5e00)
                SUM[7]=SUM[7]+output->CS[5][N][L]+output->CS[6][N][L]+output->CS[7][N][L];
            if(gas->ISPR[1][L]>0)
            {
                SUM[8]=SUM[8]+output->CS[8][N][L];
                SUM[9]=SUM[9]+output->CS[1][N][L]*gas->ISPR[1][L];
            }
        }
        AVW=0;
        for(L=1;L<=gas->MSP;L++)
        {
            output->VARSP[0][N][L]=output->CS[1][N][L];
            output->VARSP[1][N][L]=0.e00;
            output->VARSP[6][N][L]=0.0;
            output->VARSP[7][N][L]=0.0;
            output->VARSP[8][N][L]=0.0;
            if(SUM[1]>0.1)
            {
                output->VARSP[1][N][L]=output->CS[1][N][L]/SUM[1];
                AVW=AVW+gas->SP[3][L]*output->CS[1][N][L]/SUM[1];
                if(gas->ISPR[1][L]>0 && output->CS[1][N][L]>0.5)
                    output->VARSP[6][N][L]=(2.e00/BOLTZ)*output->CS[8][N][L]/((double)(gas->ISPR[1][L])*output->CS[1][N][L]);
            }
            output->VARSP[5][N][L]=0;
            for(K=1;K<=3;K++)
            {
                output->VARSP[K+1][N][L]=(gas->SP[5][L]/BOLTZ)*(output->VARSP[K+1][N][L]-pow(output->VARSP[K+8][N][L],2));
                output->VARSP[5][N][L]=output->VARSP[5][N][L]+output->VARSP[K+1][N][L];
            }
            output->VARSP[5][N][L]=output->VARSP[5][N][L]/3.e00;
            output->VARSP[8][N][L]=(3.e00*output->VARSP[5][N][L]+(double)gas->ISPR[1][L]*output->VARSP[6][N][L])/(3.e00+(double)(gas->ISPR[1][L]));
        }
        if(geom->IVB==0)
            output->VAR[1][N]=geom->CELL[1][N];
        if(geom->IVB==1)
        {
            C=(geom->XB[2]+geom->VELOB*calc->FTIME-geom->XB[1])/(double)(geom->NDIV); //new DDIV
            output->VAR[1][N]=geom->XB[1]+((double)(N-1)+0.5)*C;
        }
        output->VAR[2][N]=SUM[0];
        if(SUM[1]>0.5)
        {
            output->VAR[3][N]=SUM[1]*A;//--number density Eqn. (4.28)
            output->VAR[4][N]=output->VAR[3][N]*SUM[2]/SUM[1]; //--density  Eqn. (4.29)
            output->VAR[5][N]=SUM[3]/SUM[2];//--u velocity component  Eqn. (4.30)
            output->VAR[6][N]=SUM[4]/SUM[2]; //--v velocity component  Eqn. (4.30)
            output->VAR[7][N]=SUM[5]/SUM[2]; //--w velocity component  Eqn. (4.30)
            UU= pow(output->VAR[5][N],2)+pow(output->VAR[6][N],2)+pow(output->VAR[7][N],2);
            if(SUM[1]>1)
            {   
                output->VAR[8][N]=(fabs(SUM[6]-SUM[2]*UU))/(3.e00*BOLTZ*SUM[1]); //Eqn. (4.39)
                //--translational temperature
                output->VAR[19][N]=fabs(SUM[10]-SUM[2]*pow(output->VAR[5][N],2))/(BOLTZ*SUM[1]);
                output->VAR[20][N]=fabs(SUM[11]-SUM[2]*pow(output->VAR[6][N],2))/(BOLTZ*SUM[1]);
                output->VAR[21][N]=fabs(SUM[12]-SUM[2]*pow(output->VAR[7][N],2))/(BOLTZ*SUM[1]);
            }
            else
            {
                output->VAR[8][N]=1.0;
                output->VAR[19][N]=1.0;
                output->VAR[20][N]=1.0;
                output->VAR[21][N]=1.0;
            }
            if(SUM[9]>0.1e00)
            {
                output->VAR[9][N]=(2.e00/BOLTZ)*SUM[8]/SUM[9]; ////--rotational temperature Eqn. (4.36)
            }
            else
                output->VAR[9][N]=0.0;
            
            output->VAR[10][N]=gas->FTMP[1]; ////vibration default
            DOF=(3.e00+SUM[9])/SUM[1];
            output->VAR[11][N]=(3.0*output->VAR[8][N]+(SUM[9]/SUM[1]))*output->VAR[9][N]/DOF;
            //--overall temperature based on translation and rotation
            output->VAR[18][N]=output->VAR[3][N]*BOLTZ*output->VAR[8][N];
            //--scalar pressure (now (from V3) based on the translational temperature)
            if(gas->MMVM>0)
            {
                for(L=1;L<=gas->MSP;L++)
                {
                    VDOF[L]=0.0;
                    //dout
                    if(gas->ISPV[L] > 0)
                    {
                        for(K=1;K<=gas->ISPV[L];K++)
                        {
                            if(output->CS[K+9][N][L]<BOLTZ)
                            {
                                TV[K][L]=0.0;
                                DF[N][K][L]=0.0;
                            }
                            else
                            {
                                TV[K][L]=gas->SPVM[1][K][L]/log(1.0+output->CS[1][N][L]/output->CS[K+9][N][L]) ;//--Eqn.(4.45)
                                DF[N][K][L]=2.0*(output->CS[K+9][N][L]/output->CS[1][N][L])*log(1.0+output->CS[1][N][L]/output->CS[K+9][N][L]); //--Eqn. (4.46)
                            }
                            VDOF[L]=VDOF[L]+DF[N][K][L];
                        }
                        //memset(TVIB,0.0,sizeof(*TVIB));
                        for(int i=0;i<gas->MSP+1;i++)
                            TVIB[i]=0.0;
                        
                        for(K=1;K<=gas->ISPV[L];K++)
                        {
                            if(VDOF[L]>1.e-6)
                            {
                                TVIB[L]=TVIB[L]+TV[K][L]*DF[N][K][L]/VDOF[L];
                            }
                            else
                                TVIB[L]=gas->FVTMP[1];
                        }
                    }
                    else
                    {
                        TVIB[L]=calc->TREF;
                        VDOF[L]=0.0;
                    }
                    output->VARSP[7][N][L]=TVIB[L];
                }
                VDOFM=0.0;
                TVIBM=0.0;
                A=0.e00;
                for(L=1;L<=gas->MSP;L++)
                {
                    //dout
                    if(gas->ISPV[L] > 0)
                    {
                        A=A+output->CS[1][N][L];
                    }
                }
                for(L=1;L<=gas->MSP;L++)
                {
                    //dout
                    if(gas->ISPV[L] > 0)
                    {
                        VDOFM=VDOFM+VDOF[L]-output->CS[1][N][L]/A;
                        TVIBM=TVIBM+TVIB[L]-output->CS[1][N][L]/A;
                    }
                }
                output->VAR[10][N]=TVIBM;
            }
            for(L=1;L<=gas->MSP;L++)
            {
                if(output->VARSP[0][N][L]>0.5)
                {
                    //--convert the species velocity components to diffusion velocities
                    for(K=1;K<=3;K++)
                    {
                        output->VARSP[K+8][N][L]=output->VARSP[K+8][N][L]-output->VAR[K+4][N];
                    }
                    if(gas->MELE>1)
                    {
                        //--calculate the electronic temperatures for the species
                        //memset(ELDOF,0.e00,sizeof(*ELDOF));
                        for(int i=0;i<gas->MSP+1;i++)
                            ELDOF[i] = 0.e00;
                        //dout
                        //memset(TEL,0.e00,sizeof(*TEL));
                        for(int i=0;i<gas->MSP+1;i++)
                            TEL[i] = 0.e00;
                        if(gas->MELE>1)
                        {
                            A=0.e00;
                            B=0.e00;
                            for(M=1;M<=gas->NELL[L];M++)
                            {
                                if(output->VARSP[5][N][L]>1.e00)
                                {
                                    C=gas->QELC[2][M][L]/output->VARSP[5][N][L];
                                    A=A+gas->QELC[1][M][L]*exp(-C);
                                    B=B+gas->QELC[1][M][L]*C*exp(-C);
                                }
                            }
                            if(B>1.e-10)
                            {
                                TEL[L]=output->CS[9][N][L]/output->CS[1][N][L]/(BOLTZ*B/A);
                            }
                            else
                                TEL[L]=output->VAR[11][N];
                            output->VARSP[12][N][L]=TEL[L];
                            ELDOF[L]=0.e00;
                            if(output->VARSP[5][N][L]>1.e00)
                                ELDOF[L]=2.e00*output->CS[9][N][L]/output->CS[1][N][L]/(BOLTZ*output->VARSP[5][N][L]);
                            if(ELDOF[L]<0.01)
                            {
                                output->VARSP[12][N][L]=output->VAR[11][N];
                            }
                        }
                        else
                        {
                            ELDOF[L]=0.0;
                        }
                    }
                }
                else
                {
                    for(K=8;K<=12;K++)
                    {
                        output->VARSP[K][N][L]=0.e00;
                    }
                }
            }
            //--set the overall electronic temperature
            if(gas->MELE>1)
            {
                C=0.e00;
                for(L=1;L<=gas->MSP;L++)
                {
                    if(ELDOF[L]>1.e-5)
                        C=C+output->CS[1][N][L];
                }
                if(C>0.e00)
                {
                    A=0.e00;
                    B=0.e00;
                    for(L=1;L<=gas->MSP;L++)
                    {
                        if(ELDOF[L]>1.e-5)
                        {
                            A=A+output->VARSP[12][N][L]*output->CS[1][N][L];
                            B=B+output->CS[1][N][L];
                        }
                    }
                    output->VAR[22][N]=A/B;
                }
                else{
                    output->VAR[22][N]=output->VAR[11][N];
                }
            }
            else{
                output->VAR[22][N]=gas->FTMP[1];
            }
            if(gas->MMVM>0)
            {
                //--set the overall temperature and degrees of freedom for the individual species
                for(L=1;L<=gas->MSP;L++)
                {
                    if(gas->MELE>1){
                        SDOF[L]=3.e00+gas->ISPR[1][L]+VDOF[L]+ELDOF[L];
                        output->VARSP[8][N][L]=(3.0*output->VARSP[5][N][L]+gas->ISPR[1][L]*output->VARSP[6][N][L]+VDOF[L]*output->VARSP[7][N][L]+ELDOF[L]*output->VARSP[12][N][L])/SDOF[L];
                    }
                    else{
                        SDOF[L]=3.e00+gas->ISPR[1][L]+VDOF[L]+ELDOF[L];
                        output->VARSP[8][N][L]=(3.0*output->VARSP[5][N][L]+gas->ISPR[1][L]*output->VARSP[6][N][L]+VDOF[L]*output->VARSP[7][N][L])/SDOF[L];
                    }
                }
                //--the overall species temperature now includes vibrational and electronic excitation
                //--the overall gas temperature can now be set
                A=0.e00;
                B=0.e00;
                for(L=1;L<=gas->MSP;L++)
                {
                    A=A+SDOF[L]+output->VARSP[8][N][L]*output->CS[1][N][L];
                    B=B+SDOF[L]*output->CS[1][N][L];
                }
                output->VAR[11][N]=A/B;
            }
            VEL=sqrt(pow(output->VAR[5][N],2)+pow(output->VAR[6][N],2)+pow(output->VAR[7][N],2));
            output->VAR[12][N]=VEL/sqrt((DOF+2.e00)*output->VAR[11][N]*(SUM[1]*BOLTZ/SUM[2]))/DOF;
            //--Mach number
            output->VAR[13][N]=SUM[0]/output->NSAMP; ////--average number of molecules in cell
            //dout
            if(output->COLLS[N] > 2.0)
            {
                output->VAR[14][N]=0.5e00*(calc->FTIME-output->TISAMP)*(SUM[1]/output->NSAMP)/output->WCOLLS[N];
                //--mean collision time
                output->VAR[15][N]=0.92132e00*sqrt(fabs(SUM[7]/SUM[1]-UU))*output->VAR[14][N];
                //--mean free path (based on r.m.s speed with correction factor based on equilibrium)
                output->VAR[16][N]=output->CLSEP[N]/(output->COLLS[N]*output->VAR[15][N]);
            }
            else{
                output->VAR[14][N]=1.e10;
                output->VAR[15][N]=1.e10/output->VAR[3][N];
                //--m.f.p set by nominal values
            }
        }
        else
        {
            for(L=3;L<=22;L++)
            {
                output->VAR[L][N]=0.0;
            }
        }
        output->VAR[17][N]=VEL;
    }
    if(calc->FTIME>0.e00*calc->DTM)
    {
        if(calc->ICLASS==1){
            if(geom->IFX==0)
                file_3<<"DSMC program for a one-dimensional plane flow"<<endl;//WRITE (3,*) 'DSMC program for a one-dimensional plane flow';
            if(geom->IFX==1)
                file_3<<"DSMC program for a cylindrical flow"<<endl;//WRITE (3,*) 'DSMC program for a one-dimensional plane flow';
            if(geom->IFX==2)
                file_3<<"DSMC program for a spherical flow"<<endl;//WRITE (3,*) 'DSMC program for a one-dimensional plane flow';
        }
        file_3<<endl;//WRITE (3,*)
        file_3<<"Interval "<<output->NOUT<<" Time "<<calc->FTIME<< " with "<<output->NSAMP<<" samples from "<<output->TISAMP<<endl;
        //WRITE (3,*) 'Interval',output->NOUT,'Time ',calc->FTIME, ' with',output->NSAMP,' samples from',output->TISAMP
        //990 FORMAT(I7,G13.5,I7,G13.5)
        //Dout
        NNN=calc->TOTMOV;
        cout<<"TOTAL MOLECULES = "<< molecs->NM<<endl;
        //dout
        //NMS=0;
        for(int i=0;i<gas->MSP+1;i++)
            NMS[i]=0;

        for(N=1;N<=molecs->NM;N++)
        {
            M=molecs->IPSP[N];
            NMS[M]+=1;
        }
        file_3<<"Total simulated molecules = "<<molecs->NM<<endl;
        for(N=1;N<=gas->MSP;N++)
        {
            cout<< " SPECIES "<<N<<" TOTAL = "<<NMS[N]<<endl;
            file_3<<"Species "<<N<<" total = "<<NMS[N]<<endl;
        }
        if(gas->MEX>0)
        {
            ENERGY(0,A);
            for(N=1;N<=gas->MSP;N++)
            {
                if(gas->ISPV[N]>0){
                    file_9<< "SP "<<N<<" DISSOCS "<<calc->TDISS[N]<<" RECOMBS "<<calc->TRECOMB[N]<<endl;
                    cout<<"SP"<<N<<"DISSOCS"<<calc->TDISS[N]<<" RECOMBS "<<calc->TRECOMB[N]<<endl;
                    file_3<<"SP "<<N<<" DISSOCS "<<calc->TDISS[N]<<" RECOMBS "<<calc->TRECOMB[N]<<endl;
                }
            }
            for(N=1;N<=gas->MSP;N++)
            {
                cout<<"EX,C reaction"<<N<<" number"<<gas->TNEX[N]<<endl;
                file_9<<"EX,C reaction "<<N<<" number "<<gas->TNEX[N]<<endl;
                file_3<<"EX,C reaction "<<N<<" number "<<gas->TNEX[N]<<endl;
                
            }
        }
        
        file_3<<"Total molecule moves   = "<<NNN<<endl;
        //dout
        NNN=calc->TOTCOL;
        file_3<<"Total collision events = "<<NNN<<endl;
        //
        file_3<<"Species dependent collision numbers in current sample"<<endl;
        for(N=1;N<=gas->MSP;N++)
        {
            if(gas->IGAS!=8){
                for(M=1;M<=gas->MSP;M++)
                    file_3<<calc->TCOL[N][M]<<"\t";
                file_3<<endl;
                //WRITE(3,901) (calc->TCOL[N][M],M=1,gas->MSP);
            }
            if(gas->IGAS==8){
                for(M=1;M<=gas->MSP;M++)
                    file_3<<calc->TCOL[N][M]<<"\t";
                file_3<<endl;
                // WRITE(3,902) (calc->TCOL[N][M],M=1,gas->MSP);
            }
        }
        //Dout
        //901 FORMAT(5G13.5)
        //902 FORMAT(8G13.5)
        //dout
        CTIME=clock();
        file_3<<"Computation time "<<(double)CTIME/1000.0<< "seconds"<<endl;
        file_3<<"Collision events per second "<<(calc->TOTCOL-calc->TOTCOLI)*1000.e00/(double)CTIME<<endl;
        file_3<<"Molecule moves per secon "<<(calc->TOTMOV-calc->TOTMOVI)*1000.e00/(double)CTIME<<endl;
        if(calc->ICLASS==0&& gas->MMVM==0&&calc->ISF==0){
            //--a homogeneous gas with no vibratioal modes - assume that it is a collision test run
            //******PRODUCES DATA FOR TABLES 6.1 AND 6.2 IN SECTION 6.2*******
            //
            A=0.e00;
            B=0.e00;
            C=0.e00;
            for(N=1;N<=geom->NCCELLS;N++)
            {
                A+=geom->CCELL[5][N];
                B+=geom->CCELL[4][N];
                C+=geom->CCELL[3][N];
            }
            file_3<<"Overall time step "<<calc->DTM<<endl;
            file_3<<"Molecules per collision cell "<<(double)(molecs->NM)/(double)(geom->NCCELLS)<<endl;
            file_3<<"Mean cell time ratio "<< A/((double)(geom->NCCELLS)*calc->FTIME)<<endl;
            file_3<<"Mean value of cross-section and relative speed "<<B/(double)(geom->NCCELLS)<<endl;
            file_3<<"Mean half collision cell time step "<<C/(double)(geom->NCCELLS)<<endl;
            if(gas->MSP==1){
                A=2.e00*SPI*output->VAR[3][1]  *(pow(gas->SP[1][1],2))*sqrt(4.e00*BOLTZ*gas->SP[2][1]/gas->SP[5][1])*pow((output->VAR[11][1])/gas->SP[2][1],(1.e00-gas->SP[3][1]));
                //--Eqn. (2.33) for equilibhrium collision rate
                file_3<<"Coll. rate ratio to equilib "<<calc->TCOL[1][1]/((double)(molecs->NM)*(calc->FTIME-output->TISAMP))/A<<endl;
            }
            else{
                file_3<<"Species collision rate ratios to equilibrium"<<endl;
                for(N=1;N<=gas->MSP;N++){
                    file_3<<"Collision rate for species "<<N<<endl;
                    for(M=1;M<=gas->MSP;M++)
                    {
                        THCOL[N][M]=2.e00*(1.e00/SPI)*output->VAR[3][1]*output->VARSP[1][1][M]*gas->SPM[2][N][M]*sqrt(2.e00*BOLTZ*gas->SPM[5][N][M]/gas->SPM[1][N][M])*pow(output->VAR[11][1]/gas->SPM[5][N][M],1.e00-gas->SPM[3][N][M]);
                        //--Eqn. (2.36) for equilibhrium collision rate of species N with species M
                        file_3<<"with species "<<M<<" "<<calc->TCOL[N][M]/((double)(molecs->NM)*gas->FSP[N][1]*(calc->FTIME-output->TISAMP))/THCOL[N][M]<<endl;
                    }
                }
                file_3<<endl;
                for(N=1;N<=gas->MSP;N++){
                    file_3<<"Collision numbers for species "<<N<<endl;
                    for(M=1;M<=gas->MSP;M++){
                        file_3<<"with species "<<M<<" "<<calc->TCOL[N][M]<<endl;
                    }
                }
            }
        }
        file_3<<endl;
        if(geom->ITYPE[1]==2|| geom->ITYPE[2]==1)
            file_3<<"Surface quantities"<<endl;
        for(JJ=1;JJ<=2;JJ++)
        {
            if(geom->ITYPE[JJ]==2){
                file_3<<endl;
                file_3<<"Surface at "<<geom->XB[JJ]<<endl;
                file_3<<"Incident sample "<<output->VARS[0][JJ]<<endl;
                file_3<<"Number flux "<<output->VARS[3][JJ]<<" /sq m/s"<<endl;
                file_3<<"Inc pressure "<<output->VARS[5][JJ]<<" Refl pressure "<<output->VARS[6][JJ]<<endl;
                file_3<<"Pressure "<< output->VARS[5][JJ]+output->VARS[6][JJ]<<" N/sq m"<<endl;
                file_3<<"Inc y shear "<<output->VARS[7][JJ]<<" Refl y shear "<<output->VARS[8][JJ]<<endl;
                file_3<<"Net y shear "<<output->VARS[7][JJ]-output->VARS[8][JJ]<<" N/sq m"<<endl;
                file_3<<"Net z shear "<<output->VARS[9][JJ]-output->VARS[10][JJ]<<" N/sq m"<<endl;
                file_3<<"Incident translational heat flux "<<output->VARS[11][JJ]<<" W/sq m"<<endl;
                if(gas->MMRM>0)
                    file_3<<"Incident rotational heat flux "<<output->VARS[13][JJ]<<" W/sq m"<<endl;
                if(gas->MMVM>0)
                    file_3<<"Incident vibrational heat flux "<<output->VARS[15][JJ]<<" W/sq m"<<endl;
                if(gas->MELE>1)
                    file_3<<"Incident electronic heat flux "<<output->VARS[33][JJ]<<" W/sq m"<<endl;
                file_3<<"Total incident heat flux "<<output->VARS[29][JJ]<<" W/sq m"<<endl;
                file_3<<"Reflected translational heat flux "<<output->VARS[12][JJ]<<" W/sq m"<<endl;
                if(gas->MMRM>0)
                    file_3<<"Reflected rotational heat flux "<<output->VARS[14][JJ]<<" W/sq m"<<endl;
                if(gas->MMVM>0)
                    file_3<<"Reflected vibrational heat flux "<<output->VARS[16][JJ]<<" W/sq m"<<endl;
                if(gas->MELE>1)
                    file_3<<"Reflected electronic heat flux "<<output->VARS[34][JJ]<<" W/sq m"<<endl;
                file_3<<"Total reflected heat flux "<<output->VARS[30][JJ]<<" W/sq m"<<endl;
                file_3<<"Net heat flux "<<output->VARS[31][JJ]<<" W/sq m"<<endl;
                file_3<<"Slip velocity (y direction) "<<output->VARS[19][JJ]<<" m/s"<<endl;
                file_3<<"Translational temperature slip"<<output->VARS[20][JJ]<<" K"<<endl;
                if(gas->MMRM>0)
                    file_3<<"Rotational temperature slip "<<output->VARS[21][JJ]<<" K"<<endl;
                if(gas->MSP>1)
                {
                    for(L=1;L<=gas->MSP;L++)
                    {
                        file_3<<"Species "<<L<<" percentage "<<output->VARS[L+35][JJ]<<endl;
                    }
                }
            }
        }

        file_3<<endl;
        //PPA=0;
        for(int i=0;i<gas->MSP+1;i++)
            PPA[i]=0;

        for(N=1;N<=geom->NCELLS;N++)
        {
            for(M=1;M<=gas->MSP;M++){
                PPA[M]=PPA[M]+output->VARSP[0][N][M];
            }
        }
        // WRITE (*,*)
        //cin.get();
        if(gas->MSP>1)
        {
            file_3<<"GAINS FROM REACTIONS"<<endl;
            file_3<<"                          Dissoc.     Recomb. Endo. Exch.  Exo. Exch."<<endl;
            for(M=1;M<=gas->MSP;M++){
                file_3<<"                          SPECIES "<<M<<" "<<gas->TREACG[1][M]<<" "<<gas->TREACG[2][M]<<" "<<gas->TREACG[3][M]<<" "<<gas->TREACG[4][M]<<endl;
            }
            file_3<<endl;
            file_3<<"LOSSES FROM REACTIONS"<<endl;
            file_3<<"                          Dissoc.     Recomb. Endo. Exch.  Exo. Exch."<<endl;
            for(M=1;M<=gas->MSP;M++){
                file_3<<"                          SPECIES "<<M<<" "<<gas->TREACL[1][M]<<" "<<gas->TREACL[2][M]<<" "<<gas->TREACL[3][M]<<" "<<gas->TREACL[4][M]<<endl;
            }
            file_3<<endl;
            file_3<<"TOTALS"<<endl;
            for(M=1;M<=gas->MSP;M++){
                file_3<<"                        SPECIES "<<M<<" GAINS "<<gas->TREACG[1][M]+gas->TREACG[2][M]+gas->TREACG[3][M]+gas->TREACG[4][M]<<" LOSSES "<<gas->TREACL[1][M]+gas->TREACL[2][M]+gas->TREACL[3][M]+gas->TREACL[4][M]<<endl;
            }
        }
        file_3<<endl;
        file_3<<"Flowfield properties "<<endl;
        file_3<< output->NSAMP<<" Samples"<<endl;
        file_3<<"Overall gas"<<endl;
        file_3<<"Cell x coord.      Sample       Number Dens. Density      u velocity   v velocity   w velocity   Trans. Temp. Rot. Temp.   Vib. Temp.    El. Temp.  Temperature  Mach no.     Mols/cell    m.c.t        m.f.p        mcs/mfp        speed      Pressure      TTX         TTY         TTZ   Species Fractions "<<endl;
        for(N=1;N<=geom->NCELLS;N++)
        {
            file_3<< N<<" ";
            for(M=1;M<=10;M++){
                file_3<<output->VAR[M][N]<<" ";
            }
            file_3<<output->VAR[22][N]<<" ";
            for(M=11;M<=21;M++){
                file_3<<output->VAR[M][N]<<" ";
            }
            for(L=1;M<=gas->MSP;M++){
                file_3<<output->VARSP[1][N][L]<<" ";
            }
            file_3<<endl;
        }
        file_3<<"Individual molecular species"<<endl;
        for(L=1;L<=gas->MSP;L++){
            file_3<<"Species "<<L<<endl;
            file_3<<"Cell x coord.      Sample       Percentage   Species TTx   Species TTy  Species TTz  Trans. Temp.  Rot. Temp.  Vib. Temp.   Spec. Temp  u Diff. Vel. v Diff. Vel. w. Diff. Vel. Elec. Temp."<<endl;
            for(N=1;N<=geom->NCELLS;N++){
                file_3<< N<<" "<<output->VAR[1][N]<<" ";
                for(M=0;M<=12;M++)
                    file_3<<output->VARSP[M][N][L]<<" ";
                file_3<<endl;
            }
        }
        //dout
        //999 FORMAT (I5,30G13.5)
        //998 FORMAT (G280.0)
        // 997 FORMAT (G188.0)
        // CLOSE (3)
        file_3.close();
    }
    if(calc->ICLASS==0 && calc->ISF==1){
        //--a homogeneous gas and the "unsteady sampling" option has been chosen-ASSUME THAT IT IS A RELAXATION TEST CASE FOR SECTION 6.2
        INITIALISE_SAMPLES();
        //write a special output file for internal temperatures and temperature versus collision number
        //dout
        file_10.open("RELAX.DAT", ios::app | ios::out);
        if(file_10.is_open()){
            cout<<"RELAX.DAT is opened"<<endl;
        }
        else{
            cout<<"RELAX.DAT not opened"<<endl;
        }
        // OPEN (10,FILE='RELAX.DAT',ACCESS='APPEND')
        A=2.0*calc->TOTCOL/molecs->NM; //--mean collisions
        //--VAR(11,N)   //--overall
        //--VAR(8,N)    //--translational
        //--VAR(9,N)    //--rotational
        //--VAR(10,N)   //--vibrational
        //--VAR(22,N)   //--electronic
        //file_10<<std::right<<setw(15)<<A<<setw(15)<<output->VAR[8][1]<<setw(15)<<output->VAR[9][1]<<setw(15)<<output->VAR[8][1]-output->VAR[9][1]<<endl;
        file_10<<std::right<<setw(15)<<A<<setw(15)<<output->VAR[11][1]<<setw(15)<<output->VAR[8][1]<<setw(15)<<output->VAR[9][1]<<setw(15)<<output->VAR[10][1]<<setw(15)<<output->VAR[22][1]<<endl;
        //file_10<<std::right<<setw(15)<<A<<setw(15)<<output->VAR[8][1]<<setw(15)<<output->VAR[9][1]<<setw(15)<<output->VAR[8][1]-output->VAR[9][1]<<endl;
        //  WRITE (10,950) A,VAR(8,1),VAR(9,1),VAR(8,1)-VAR(9,1)   //--Generates output for Figs. 6.1 and 6.2
        //  WRITE (10,950) A,VAR(11,1),VAR(8,1),VAR(9,1),VAR(10,1),VAR(22,1)   //--Generates output for modal temperatures in Figs. 6.3, 6.5 +
        //  WRITE (10,950) A,0.5D00*(VAR(8,1)+VAR(9,1)),VAR(10,1),0.5D00*(VAR(8,1)+VAR(9,1))-VAR(10,1)  //--Generates output for Figs. 6.4
        //
        //--VARSP(8,N,L) //--overall temperature of species L
        //  WRITE (10,950) A,VARSP(8,1,3),VARSP(8,1,2),VARSP(8,1,5),VARSP(8,1,4),A  //--output for Fig 6.17
        // CLOSE (10)
        file_10.close();
    }
    //dout
    // 950 FORMAT (6G13.5)
    if(gas->IGAS==8||gas->IGAS==6||gas->IGAS==4)
    {
        //--Write a special output file for the composition of a reacting gas as a function of time
        //dout
        //OPEN (10,FILE='COMPOSITION.DAT',ACCESS='APPEND')
        file_10.open("COMPOSITION.DAT", ios::app | ios::out);
        if(file_10.is_open()){
            cout<<"COMPOSITION.DAT is opened"<<endl;
        }
        else{
            cout<<"COMPOSITION.DAT not opened"<<endl;
        }
        AS=molecs->NM;
        //dout
        AT=calc->FTIME*1.e6;
        if (gas->IGAS == 4)
            file_10<< AT <<" "<<(double)(NMS[1])/1000000<<" "<<A<<" "<<output->VAR[11][1]<<endl;    //--Data for fig
        if (gas->IGAS == 8)
            file_10<<AT<<" "<<NMS[1]/AS<<" "<<NMS[2]/AS<<" "<<NMS[3]/AS<<" "<<NMS[4]/AS<<" "<<NMS[5]/AS<<" "<<NMS[6]/AS<<" "<<NMS[7]/AS<<" "<<NMS[8]/AS<<" "<<output->VAR[11][1]<<endl;
        if (gas->IGAS == 6)
            file_10<<AT<<" "<<NMS[1]/AS<<" "<<NMS[2]/AS<<" "<<NMS[3]/AS<<" "<<NMS[4]/AS<<" "<<NMS[5]/AS<<" "<<output->VAR[11][1]<<endl;
        //dout
        // 888 FORMAT(10G13.5)
        file_10.close();
    }
    if(calc->FTIME>0.5e00*calc->DTM){
        //
        //--reset collision and transit times etc.
        //
        cout<<"Output files written "<<endl;
        DTMI=calc->DTM;
        if(calc->IMTS<2){
            if(calc->ICLASS>0)
                calc->DTM*=2;
            //--this makes it possible for DTM to increase, it will be reduced as necessary
            for(NN=1;NN<=geom->NCELLS;NN++)
            {
                CDTM[NN]=calc->DTM;
                B=geom->CELL[3][NN]-geom->CELL[2][NN] ;//--sampling cell width
                if(output->VAR[13][NN]>20.e00){
                    //consider the local collision rate
                    CDTM[NN]=output->VAR[14][NN]*calc->CPDTM;
                    //look also at sampling cell transit time based on the local flow speed
                    A=(B/(fabs(output->VAR[5][NN])))*calc->TPDTM;
                    if(A<CDTM[NN])
                        CDTM[NN]=A;
                }
                else{
                    //-- base the time step on a sampling cell transit time at the refence vmp
                    A=calc->TPDTM*B/gas->VMPM;
                    if(A<CDTM[NN])
                        CDTM[NN]=A;
                }
                if(CDTM[NN]<calc->DTM)
                    calc->DTM=CDTM[NN];
            }
        }
        else
        {
            //dout
            //memset(CDTM, calc->DTM, sizeof(*CDTM));
            for(int i=0;i<geom->NCELLS+1;i++)
                CDTM[i]= calc->DTM;
            //CDTM=calc->DTM;
        }
        for(N=1;N<=geom->NCELLS;N++){
            NN=geom->ICCELL[3][N];
            geom->CCELL[3][N]=0.5*CDTM[NN];
        }
        file_9<<"DTM changes  from "<<DTMI<<" to "<<calc->DTM<<endl;
        calc->DTSAMP=calc->DTSAMP*calc->DTM/DTMI;
        calc->DTOUT=calc->DTOUT*calc->DTM/DTMI;
    }
    else
    {
        INITIALISE_SAMPLES();
    }
    if(calc->ICLASS==1&& calc->ISF==1)
    {
        //*************************************************************************
        //--write TECPLOT data files for x-t diagram (unsteady calculation only)
        //--comment out if not needed
        //dout
        file_18.open("DS1xt.DAT", ios::app | ios::out);
        if(file_18.is_open()){
            cout<<"DS1xt.DAT is opened"<<endl;
        }
        else
            cout<<"DS1xt.DAT not opened"<<endl;
        // OPEN (18,FILE='DS1xt.DAT',ACCESS='APPEND')
        //--make sure that it is empty at the stary of the run
        SETXT();
        // CLOSE (18)
        file_18.close();
        //**************************************************************************
    }
    //WRITE (19,*) calc->FTIME,-output->VARS[5][1],-output->VARS[5][1]-output->VARS[6][1]
    
    file_7.open("PROFILE.DAT" , ios::out);
    if(file_7.is_open()){
        cout<<"PROFILE.DAT is opened"<<endl;
    }
    else
        cout<<"PROFILE.DAT not opened"<<endl;
    // OPEN (7,FILE='PROFILE.DAT',FORM='FORMATTED')
    //
    //OPEN (8,FILE='ENERGYPROF.DAT',FORM='FORMATTED')
    //
    // 995 FORMAT (22G13.5)
    // 996 FORMAT (12G14.6)
    for(N=1;N<=geom->NCELLS;N++)
    {
        //
        //--the following line is the default output
        //  WRITE (7,995) VAR(1,N),VAR(4,N),VAR(3,N),VAR(11,N),VAR(18,N),VAR(5,N),VAR(12,N),VAR(8,N),VAR(9,N),VAR(10,N),VAR(22,N),     &
        //        (VARSP(8,N,M),M=1,MSP),(VARSP(1,N,M),M=1,MSP)
        //
        //--calculate energies per unit mass (employed for re-entry shock wave in Section 7.5)
        C1=0.5e00*pow(output->VAR[5][N],2);    //--Kinetic
        C2=0.e00;                 //--Thermal
        C3=0.e00;                //--Rotational
        C4=0.e00;               //--Vibrational
        C5=0.e00;              //--Electronic
        C6=0.e00;             //--Formation
        for(L=1;L<=gas->MSP;L++)
        {
            //    C2=C2+3.D00*BOLTZ*VARSP(5,N,L)*VARSP(1,N,L)/SP(5,L)
            A=(output->CS[1][N][L]/output->VARSP[1][N][L])*gas->SP[5][L];
            if(output->CS[1][N][L]>0.5e00){
                C2=C2+0.5e00*(output->CS[5][N][L]+output->CS[6][N][L]+output->CS[7][N][L])*gas->SP[5][L]/A;
                if(gas->ISPR[1][L]>0)
                    C3=C3+output->CS[8][N][L];
                if(gas->ISPV[L]>0)
                    C4=C4+output->CS[10][N][L]*BOLTZ*gas->SPVM[1][1][L]/A;
                if(gas->NELL[L]>1)
                    C5=C5+output->CS[9][N][L]/A;
                C6=C6+gas->SP[6][L]*output->CS[1][N][L]/A;
            }
        }
        C2=C2-C1;
        //  A=0.5D00*VFX(1)**2+2.5D00*BOLTZ*FTMP(1)/(0.75*SP(5,2)+0.25*SP(5,1))
        C7=C1+C2+C3+C4+C5+C6;
        //
        //  WRITE (8,995) VAR(1,N),C1/A,C2/A,C3/A,C4/A,C5/A,C6/A,C7/A
        //
        //--the following lines are for normalised shock wave output in a simple gas (Sec 7.3)
        C1=gas->FND[2]-gas->FND[1];
        C2=gas->FTMP[2]-gas->FTMP[1];
        
        file_7<<output->VAR[1][N]<<" "<<output->VAR[2][N]<<" "<<(0.5*(output->VAR[20][N]+output->VAR[21][N])-gas->FTMP[1])/C2<<" "<<(output->VAR[19][N]-gas->FTMP[1])/C2<<" "<<(output->VAR[11][N]-gas->FTMP[1])/C2<<" "<<(output->VAR[3][N]-gas->FND[1])/C1<<endl;
        //--the following replaces sample size with density
        //C3=0.D00
        //DO L=1,MSP
        //  C3=C3+FND(1)*FSP(L,1)*SP(5,L)  //--upstream density
        //END DO
        //C4=0.D00
        //DO L=1,MSP
        //  C4=C4+FND(2)*FSP(L,2)*SP(5,L)  //--upstream density
        //END DO
        //
        //  WRITE (7,996) VAR(1,N),(VAR(4,N)-C3)/(C4-C3),(0.5*(VAR(20,N)+VAR(21,N))-FTMP(1))/C2,(VAR(19,N)-FTMP(1))/C2,(VAR(11,N)-FTMP(1))/C2,    &
        //        (VAR(3,N)-FND(1))/C1
        //--the following lines is for a single species in a gas mixture
        //  C1=C1*FSP(3,1)
        //  WRITE (7,996) VAR(1,N),VARSP(1,N,3),(0.5*(VARSP(3,N,3)+VARSP(4,N,3))-FTMP(1))/C2,(VARSP(2,N,3)-FTMP(1))/C2,(VARSP(5,N,3)-FTMP(1))/C2,(VAR(3,N)*VARSP(1,N,3)-FND(1)*FSP(3,1))/C1
        //
        //--the following line is for Couette flow (Sec 7.4)
        //  WRITE (7,996) VAR(1,N),VAR(2,N),VAR(5,N),VAR(6,N),VAR(7,N),VAR(11,N)
        //--the following line is for the breakdown of equilibrium in expansions (Sec 7.10)
        //  WRITE (7,996) VAR(1,N),VAR(2,N),VAR(12,N),VAR(4,N),VAR(5,N),VAR(8,N),VAR(9,N),VAR(10,N),VAR(11,N),VAR(19,N),VAR(20,N),VAR(21,N)
        //
    }
    if(calc->ISF==1)
        INITIALISE_SAMPLES();
    // CLOSE(7)
    file_7.close();
    //
    //--deallocate local variables
    //
    //dout
    for(int i=0;i<gas->MMVM+1;i++){
        delete [] TV[i];
    }
    delete [] TV;
    delete [] TVIB;
    delete [] VDOF;
    for(int i=0;i<gas->MSP+1;i++){
        delete [] THCOL[i];
    }
    delete [] THCOL;
    // DEALLOCATE (TV,TVIB,VDOF,THCOL,STAT=ERROR)
    // if(calc->ERROR)
    //     cout<<"PROGRAM COULD NOT DEALLOCATE OUTPUT VARIABLES"<<calc->ERROR;
    calc->TOUT=calc->TOUT+calc->DTOUT;
    return;
}



// __global__ void kernel(curandState* globalState, test &testy)
// {
//     // generate random numbers
//     //for(int i=0;i<40;i++)
//   //  {
//         float k = generate(globalState, 1);
//         //N[i] = k;
//         printf("yo %.6f\n", k);
//     //}
//         testy.a = generate(globalState, 0);
// }


__global__ void cuda_collisons(hiprandState* globalState, MOLECS *molecs, OUTPUT *output, GEOM_1D *geom, GAS *gas, CALC *calc)
{
    //CALC calc;
    //MOLECS molecs;
    //GAS gas;
    //OUTPUT output;
    //GEOM_1D geom;
    int NN,M,MM,L,LL,K,KK,KT,J,I,II,III,NSP,MAXLEV,IV,NSEL,KV,LS,MS,KS,JS,IIII,LZ,KL,IS,IREC,NLOOP,IA,IDISS,IEX,NEL,NAS,NPS,
    JJ,LIMLEV,KVV,KW,INIL,INIM,JI,LV,IVM,NMC,NVM,LSI,JX,MOLA,KR,JKV,NSC,KKV,IAX,NSTEP,NTRY,NLEVEL,NSTATE,IK,NK,MSI ;
    double A,AA,AAA,AB,B,BB,BBB,ABA,ASEL,DTC,SEP,VR,VRR,ECT,EVIB,ECC,ZV,ERM,C,OC,SD,D,CVR,PROB,RML,RMM,ECTOT,ETI,EREC,ET2,
    XMIN,XMAX,WFC,CENI,CENF,VRRT,EA,DEN,E1,E2,VRI,VRA ;
    double VRC[4],VCM[4],VRCP[4],VRCT[4];
    //   //N,M,K working integer
    // //LS,MS,KS,JS molecular species
    // //VRC components of the relative velocity
    // //RML,RMM molecule mass parameters
    // //VCM components of the center of mass velocity
    // //VRCP post-collision components of the relative velocity
    // //SEP the collision partner separation
    // //VRR the square of the relative speed
    // //VR the relative speed
    // //ECT relative translational energy
    // //EVIB vibrational energy
    // //ECC collision energy (rel trans +vib)
    // //MAXLEV maximum vibrational level
    // //ZV vibration collision number
    // //SDF the number of degrees of freedom associated with the collision
    // //ERM rotational energy
    // //NSEL integer number of selections
    // //NTRY number of attempts to find a second molecule
    // //CVR product of collision cross-section and relative speed
    // //PROB a probability
    // //KT third body molecule code
    // //ECTOT energy added at recmbination
    // //IREC initially 0, becomes 1 of a recombination occurs
    // //WFC weighting factor in the cell
    // //IEX is the reaction that occurs (1 if only one is possible)
    // //EA activation energy
    // //NPS the number of possible electronic states
    // //NAS the number of available electronic states
    //cout<<"START COLLISIONS"<<endl;
    
       int N = threadIdx.x + blockIdx.x * blockDim.x+1;
        if((calc->FTIME-geom->CCELL[5][N]) > (geom->CCELL[3][N]))
        {
            DTC=2.e00*geom->CCELL[3][N];
            //calculate collisions appropriate to  time DTC
            if(geom->ICCELL[2][N]>1)
            {
                //no collisions calculated if there are less than two molecules in collision cell
                NN=geom->ICCELL[3][N];
                WFC=1.e00;
                if(geom->IWF==1 && geom->IVB==0)
                {
                    //dout
                    WFC=1.e00+geom->WFM*powf(geom->CELL[1][NN],geom->IFX);
                }
                geom->CCELL[5][N]=geom->CCELL[5][N]+DTC;
                if(geom->IVB==0)
                {
                    AAA=geom->CCELL[1][N];
                }
                if(geom->IVB==1)
                {
                    C=(geom->XB[2]+geom->VELOB*calc->FTIME-geom->XB[1])/(double)(geom->NDIV*geom->NCIS);
                    //dout
                    XMIN=geom->XB[1]+(double)(N-1)*C;
                    XMAX=XMIN+C;
                    //dout
                    WFC=1.e00+geom->WFM*powf((0.5e00*(XMIN+XMAX)),geom->IFX);
                    if(geom->IFX==0)
                    {
                        AAA=XMAX-XMIN;
                    }
                    if(geom->IFX==1)
                    {
                        AAA=PI*(powf(XMAX,2)-powf(XMIN,2)); //assumes unit length of full cylinder
                    }
                    if(geom->IFX==2)
                    {
                        AAA=1.33333333333333333333e00*PI*(powf(XMAX,3)-powf(XMIN,3));    //flow is in the full sphere
                    }
                }
                //these statements implement the N(N-1) scheme
                ASEL=0.5e00*geom->ICCELL[2][N]*(geom->ICCELL[2][N]-1)*WFC*calc->FNUM*geom->CCELL[4][N]*DTC/AAA+geom->CCELL[2][N];
                NSEL=ASEL;
                //dout
                geom->CCELL[2][N]=ASEL-(double)(NSEL);
                if(NSEL>0)
                {
                    I=0; //counts the number of selections
                    KL=0; //becomes 1 if it is the last selection
                    IIII=0; //becomes 1 if there is a recombination
                    for(KL=1;KL<=NSEL;KL++)
                    {
                        I=I+1;
                        III=0; //becomes 1 if there is no valid collision partner
                        if(geom->ICCELL[2][N]==2)
                        {
                            K=1+geom->ICCELL[1][N];
                            //dout
                            L=molecs->ICREF[K];
                            K=2+geom->ICCELL[1][N];
                            //dout
                            M=molecs->ICREF[K];
                            if(M==molecs->IPCP[L])
                            {
                                III=1;
                                geom->CCELL[5][N]=geom->CCELL[5][N]-DTC;
                            }
                        }
                        else
                        {
                            //dout
                            //                            RANDOM_NUMBER(RANF);
                            calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                            K=(int)(calc->RANF*(double)(geom->ICCELL[2][N]))+geom->ICCELL[1][N]+1;
                            //dout
                            L=molecs->ICREF[K];
                            //one molecule has been selected at random
                            if(calc->NNC==0)
                            {
                                //select the collision partner at random
                                M=L;
                                NTRY=0;
                                while(M==L)
                                {
                                    //dout
                                    //                                    RANDOM_NUMBER(RANF);
                                    calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                    K=(int)(calc->RANF*(double)(geom->ICCELL[2][N]))+geom->ICCELL[1][N]+1;
                                    M=molecs->ICREF[K];
                                    if(M==molecs->IPCP[L])
                                    {
                                        if(NTRY<5*geom->ICCELL[2][N])
                                        {
                                            M=L;
                                        }
                                        else
                                        {
                                            III = 1;
                                            geom->CCELL[5][N]=geom->CCELL[5][N]-DTC/ASEL;
                                            M=L+1;
                                        }
                                    }
                                }
                            }
                            else
                            {
                                //elect the nearest from the total number (< 30) or a random 30
                                if(geom->ICCELL[2][N]<30)
                                {
                                    LL=geom->ICCELL[2][N];
                                }
                                else
                                {
                                    LL=30;
                                }
                                SEP=1.0e10;
                                M=0;
                                for(J=1;J<=LL;J++)
                                {
                                    if(LL<30)
                                    {
                                        K=J+geom->ICCELL[1][N];
                                    }
                                    else
                                    {
                                        //                                        RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        K=(int)(calc->RANF*(double)(geom->ICCELL[2][N]))+geom->ICCELL[1][N]+1;
                                    }
                                    MM=molecs->ICREF[K];
                                    if(MM != L)
                                    {
                                        //exclude the already selected molecule
                                        if(MM != molecs->IPCP[L])
                                        {
                                            //exclude the previous collision partner
                                            //dout
                                            A=fabsf(molecs->PX[1][L]-molecs->PX[1][MM]);
                                            if(A<SEP&& A>1.e-8*geom->DDIV)
                                            {
                                                M=MM;
                                                SEP=A;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                        if(III==0)
                        {
                            for(KK=1;KK<=3;KK++)
                            {
                                VRC[KK]=molecs->PV[KK][L]-molecs->PV[KK][M];
                            }
                            VRR=VRC[1]*VRC[1]+VRC[2]*VRC[2]+VRC[3]*VRC[3];
                            VR=sqrtf(VRR);
                            VRI=VR;
                            //Simple GAs
                            if(gas->MSP==1)
                            {
                                //dout
                                CVR=VR*gas->CXSS*powf(2.e00*BOLTZ*gas->SP[2][1]/(gas->RMAS*VRR),(gas->SP[3][1]-0.5e00))*gas->RGFS;
                                if(CVR>geom->CCELL[4][N])
                                {
                                    geom->CCELL[4][N]=CVR;
                                }
                                //dout
                                //                                RANDOM_NUMBER(RANF);
                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                if(calc->RANF<CVR/geom->CCELL[4][N])
                                {
                                    // the collision occurs
                                    if(M==molecs->IPCP[L]&& L==molecs->IPCP[M])
                                    {
                                        //file_9<<"Duplicate collision"<<endl;
                                    }
                                    //atomicAdd(&calc->TOTCOL,1.e00);
                                    //calc->TOTCOL=calc->TOTCOL+1.e00;
                                    calc->COLL_TOTCOL[N]=calc->COLL_TOTCOL[N]+1.e00;
                                    calc->TCOL[1][1]=calc->TCOL[1][1]+2.e00;
                                    output->COLLS[NN]=output->COLLS[NN]+1.e000;
                                    output->WCOLLS[NN]=output->WCOLLS[NN]+WFC;
                                    //dout
                                    SEP=fabsf(molecs->PX[1][L]-molecs->PX[1][M]);
                                    output->CLSEP[NN]=output->CLSEP[NN]+SEP;
                                    if(gas->ISPR[1][1]>0)
                                    {
                                        //Larsen-Borgnakke serial redistribution
                                        ECT=0.5e00*gas->RMAS*VRR;
                                        for(NSP=1;NSP<=2;NSP++)
                                        {
                                            //consider the molecules in turn
                                            if(NSP==1)
                                            {
                                                K=L;
                                            }
                                            else
                                            {
                                                K=M;
                                            }
                                            if(gas->MMVM>0)
                                            {
                                                if(gas->ISPV[1]>0)
                                                {
                                                    for(KV=1;KV<=gas->ISPV[1];KV++)
                                                    {
                                                        EVIB=(double)(molecs->IPVIB[KV][K]*BOLTZ*gas->SPVM[1][KV][1]);
                                                        ECC=ECT+EVIB;
                                                        if(gas->SPVM[3][KV][1]>0.0)
                                                        {
                                                            MAXLEV=ECC/(BOLTZ*gas->SPVM[1][KV][1]);
                                                            B=gas->SPVM[4][KV][1]/gas->SPVM[3][KV][1]; //Tdiss/Tref
                                                            A= gas->SPVM[4][KV][1]/output->VAR[8][NN] ;//Tdiss/Ttrans
                                                            //ZV=(A**SPM(3,1,1))*(SPVM(3,KV,1)*(B**(-SPM(3,1,1))))**(((A**0.3333333D00)-1.D00)/((B**0.33333D00)-1.D00))
                                                            ZV=powf(A,gas->SPM[3][1][1])*powf(gas->SPVM[3][KV][1]*powf(B,-gas->SPM[3][1][1]),((powf(A,0.3333333e00)-1e00)/(powf(B,33333e00)-1.e00)));
                                                        }
                                                        else
                                                        {
                                                            ZV=gas->SPVM[2][KV][1];
                                                            MAXLEV=ECC/(BOLTZ*gas->SPVM[1][KV][1])+1;
                                                        }
                                                        //dout
                                                        //                                                        RANDOM_NUMBER(RANF);
                                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                        if(1.e00/ZV>calc->RANF)
                                                        {
                                                            II=0;
                                                            while(II==0)
                                                            {
                                                                //dout
                                                                //                                                                RANDOM_NUMBER(RANF);
                                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                                IV=calc->RANF*(MAXLEV+0.99999999e00);
                                                                molecs->IPVIB[KV][K]=IV;
                                                                EVIB=(double)(IV)*BOLTZ;
                                                                if(EVIB<ECC)
                                                                {
                                                                    PROB=powf((1.e00-EVIB/ECC),(1.5e00-gas->SPM[3][KV][1]));
                                                                    //PROB is the probability ratio of eqn (3.28)
                                                                    //dout
                                                                    //                                                                    RANDOM_NUMBER(RANF);
                                                                    calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                                    if(PROB>calc->RANF)
                                                                        II=1;
                                                                }
                                                            }
                                                            ECT=ECC-EVIB;
                                                        }
                                                    }
                                                }
                                            }
                                            //now rotation of this molecule
                                            //dout
                                            if(gas->ISPR[1][1] > 0)
                                            {
                                                if(gas->ISPR[2][1]==0)
                                                {
                                                    B=1.e00/gas->SPR[1][1];
                                                }
                                                else //use molecule rather than mean value
                                                {
                                                    B=1.e00/(gas->SPR[1][1]+gas->SPR[2][1]*output->VAR[8][NN]+gas->SPR[3][1]*powf(output->VAR[8][NN],2));
                                                }
                                                //dout
                                                //                                                RANDOM_NUMBER(RANF);
                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                if(B>calc->RANF)
                                                {
                                                    ECC=ECT +molecs->PROT[K];
                                                    if(gas->ISPR[1][1]==2)
                                                    {
                                                        //dout
                                                        //                                                        RANDOM_NUMBER(RANF);
                                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                        ERM=1.e00-powf(calc->RANF,1.e00/(2.5e00-gas->SP[3][1])); //eqn(5.46)
                                                    }
                                                    else
                                                    {
                                                        //dout
                                                        LBS(globalState, 0.5e00*gas->ISPR[1][1]-1.e00,1.5e00-gas->SP[3][1],ERM);
                                                    }
                                                    molecs->PROT[K]=ERM*ECC;
                                                    ECT=ECC-molecs->PROT[K];
                                                }
                                            }
                                        }
                                        //adjust VR for the change in energy;
                                        VR=sqrtf(2.e00*ECT/gas->SPM[1][1][1]);
                                    }
                                    //end of L-B redistribution
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        VCM[KK]=0.5e00*(molecs->PV[KK][L]+molecs->PV[KK][M]);
                                    }
                                    //dout
                                    if(fabsf(gas->SP[4][1]-1.0) < 0.001)
                                    {
                                        //use the VHS logic //dout
                                        //                                        RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*calc->RANF-1.e00;
                                        //B is the cosine of a random elevation angle
                                        A=sqrtf(1.e00-B*B);
                                        VRCP[1]=B*VR;
                                        //dout
                                        //                                        RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*calc->RANF;
                                        //C is a random azimuth angle
                                        //dout
                                        VRCP[2]=A*cos(C)*VR;
                                        VRCP[3]=A*sin(C)*VR;
                                    }
                                    else
                                    {
                                        //use the VSS logic //dout
                                        //                                        RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*(powf(calc->RANF,gas->SP[4][1]))-1.e00;
                                        //B is the cosine of the deflection angle for the VSS model (Eqn. 11.8) of Bird(1994))
                                        A=sqrtf(1.e00-B*B);
                                        //dout
                                        //                                                 RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*calc->RANF;
                                        //dout
                                        OC=(double)cos(C);
                                        SD=(double)sin(C);
                                        D=sqrtf(powf(VRC[2],2)+powf(VRC[3],2));
                                        VRA=VR/VRI;
                                        VRCP[1]=(B*VRC[1]+A*SD*D)*VRA;
                                        VRCP[2]=(B*VRC[2]+A*(VRI*VRC[3]*OC-VRC[1]*VRC[2]*SD)/D)*VRA;
                                        VRCP[3]=(B*VRC[2]+A*(VRI*VRC[2]*OC-VRC[1]*VRC[3]*SD)/D)*VRA;
                                        //the post-collision rel. velocity components are based on eqn (3.18)
                                    }
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        molecs->PV[KK][L]=VCM[KK]+0.5e00*VRCP[KK];
                                        molecs->PV[KK][M]=VCM[KK]-0.5e00*VRCP[KK];
                                    }
                                    molecs->IPCP[L]=M;
                                    molecs->IPCP[M]=L;
                                }
                            } //collision occurrence
                            else
                            {
                                //Gas Mixture
                                LS=fabsf(molecs->IPSP[L]);
                                MS=fabsf(molecs->IPSP[M]);
                                CVR=VR*gas->SPM[2][LS][MS]*powf(((2.e00*BOLTZ*gas->SPM[5][LS][MS])/((gas->SPM[1][LS][MS])*VRR)),(gas->SPM[3][LS][MS]-0.5e00))*gas->SPM[6][LS][MS];
                                if(CVR>geom->CCELL[4][N])
                                {
                                    geom->CCELL[4][N]=CVR;
                                }
                                //dout
                                //                                    RANDOM_NUMBER(RANF);
                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                if(calc->RANF<CVR/geom->CCELL[4][N] && molecs->IPCELL[L]>0 && molecs->IPCELL[M]>0)
                                {
                                    //the collision occurs (-ve IPCELL indicates recombined molecule marled for removal)
                                    if(M==molecs->IPCP[L] && L==molecs->IPCP[M])
                                    {
                                        //file_9<<"Duplicate collision";
                                    }
                                    //atomicAdd(&calc->TOTCOL,1.e00);
                                    //calc->TOTCOL=calc->TOTCOL+1.e00;
                                    calc->COLL_TOTCOL[N]=calc->COLL_TOTCOL[N]+1.e00;
                                    calc->TCOL[LS][MS]=calc->TCOL[LS][MS]+1.e00;
                                    calc->TCOL[MS][LS]=calc->TCOL[MS][LS]+1.e00;
                                    output->COLLS[NN]=output->COLLS[NN]+1.e00;
                                    output->WCOLLS[NN]=output->WCOLLS[NN]+WFC;
                                    SEP=fabsf(molecs->PX[1][L]-molecs->PX[1][M]);
                                    output->CLSEP[NN]=output->CLSEP[NN]+SEP;
                                    RML=gas->SPM[1][LS][MS]/gas->SP[5][MS];
                                    RMM=gas->SPM[1][LS][MS]/gas->SP[5][LS];
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        VCM[KK]=RML*molecs->PV[KK][L]+RMM*molecs->PV[KK][M];
                                    }
                                    IDISS=0;
                                    IREC=0;
                                    IEX=0;
                                    //check for dissociation
                                    if(gas->ISPR[1][LS]>0 || gas->ISPR[1][MS]>0)
                                    {
                                        ECT=0.5e00*gas->SPM[1][LS][MS]*VRR;
                                        for(NSP=1;NSP<=2;NSP++)
                                        {
                                            if(NSP==1)
                                            {
                                                K=L; KS=LS; JS=MS;
                                            }
                                            else
                                            {
                                                K=M ; KS=MS ; JS=LS;
                                            }
                                            if(gas->MMVM>0)
                                            {
                                                if(gas->ISPV[KS]>0)
                                                {
                                                    for(KV=1;KV<=gas->ISPV[KS];KV++)
                                                    {
                                                        if(molecs->IPVIB[KV][K]>=0 && IDISS==0)
                                                        {
                                                            //do not redistribute to a dissociating molecule marked for removal
                                                            EVIB=(double)(molecs->IPVIB[KV][K]*BOLTZ*gas->SPVM[1][KV][KS]);
                                                            ECC=ECT+EVIB;
                                                            MAXLEV=ECC/(BOLTZ*gas->SPVM[1][KV][KS]);
                                                            LIMLEV=gas->SPVM[4][KV][KS]/gas->SPVM[1][KV][KS];
                                                            if(MAXLEV > LIMLEV)
                                                            {
                                                                //dissociation occurs subject to reduction factor  -  reflects the infinity of levels past the dissociation limit
                                                                //dout
                                                                //                                                                    RANDOM_NUMBER(RANF)
                                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                                if(calc->RANF<gas->SPVM[5][KV][KS])
                                                                {
                                                                    IDISS=1;
                                                                    LZ=molecs->IPVIB[KV][K];
                                                                    output->NDISSL[LZ]=output->NDISSL[LZ]+1;
                                                                    ECT=ECT-BOLTZ*gas->SPVM[4][KV][KS]+EVIB;
                                                                    //adjust VR for the change in energy
                                                                    VRR=2.e00*ECT/gas->SPM[1][LS][MS];
                                                                    VR=sqrtf(VRR);
                                                                    molecs->IPVIB[KV][K]=-1;
                                                                    //a negative IPVIB marks a molecule for dissociation
                                                                }
                                                            }
                                                        }
                                                    }
                                                }
                                            }
                                        }
                                    }
                                    IEX=0;    //becomes the reaction number if a reaction occurs
                                    IREC=0;   //becomes 1 if a recombination occurs
                                    if(IDISS==0)
                                    {
                                        //dissociation has not occurred
                                        //consider possible recombinations
                                        if(gas->ISPRC[LS][MS]>0 && geom->ICCELL[2][N]>2)
                                        {
                                            //possible recombination using model based on collision volume for equilibrium
                                            KT=L;
                                            //NTRY=0
                                            while(KT==L||KT==M)
                                            {
                                                NTRY+=1;
                                                // if(NTRY>100)
                                                // {
                                                //  cout>>"NTRY 3rd body"<<NTRY;
                                                // }
                                                //RANDOM_NUMBER(RANF);
                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);\
                                                K=(int)(calc->RANF*(double)(geom->ICCELL[2][N]))+geom->ICCELL[1][N]+1;
                                                KT=molecs->ICREF[K];
                                            }
                                            KS=molecs->IPSP[KT];
                                            //the potential third body is KT OF species KS
                                            AA=(PI/6.e00)*powf((gas->SP[1][LS]+gas->SP[1][MS]+gas->SP[1][KS]),3); //reference volume
                                            BB=AA*gas->SPRC[1][LS][MS][KS]*powf(output->VAR[8][NN]/gas->SPVM[1][gas->ISPRK[LS][MS]][gas->ISPRC[LS][MS]],gas->SPRC[2][LS][MS][KS]);//collision volume
                                            B=BB*geom->ICCELL[2][N]*calc->FNUM/AAA;
                                            if(B>1.e00)
                                            {
                                                printf("THREE BODY PROBABILITY %f\n", B);
                                                //cout<<"THREE BODY PROBABILITY"<<B;
                                                //for low density flows in which three-body collisions are very rare, it is advisable to consider recombinations in only a small
                                                //fraction of collisions and to increase the pribability by the inverse of this fraction.  This message provides a warning if this
                                                //factor has been set to an excessively large value
                                            }
                                            //dout
                                            //                                                RANDOM_NUMBER(RANF);
                                            calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                            if(calc->RANF<B)
                                            {
                                                IREC=1;
                                                calc->TRECOMB[gas->ISPRC[LS][MS]]=calc->TRECOMB[gas->ISPRC[LS][MS]]+1.e00;
                                                //the collision now becomes a collision between these with L having the center of mass velocity
                                                A=0.5e00*gas->SPM[1][LS][MS]*VRR ;//the relative energy of the recombining molecules
                                                if(gas->ISPR[1][LS]>0)
                                                    A=A+molecs->PROT[L];
                                                if(gas->MELE>1)
                                                    A=A+molecs->PELE[L];
                                                if(gas->ISPV[LS]>0)
                                                {
                                                    for(KVV=1;KVV<=gas->ISPV[LS];KVV++)
                                                    {
                                                        JI=molecs->IPVIB[KVV][L];
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        A=A+(double)(JI)*BOLTZ*gas->SPVM[1][KVV][LS];
                                                    }
                                                }
                                                if(gas->ISPR[1][MS]>0)
                                                    A+=molecs->PROT[M];
                                                if(gas->MELE>1)
                                                    A=A+molecs->PELE[M];
                                                if(gas->ISPV[MS]>0)
                                                {
                                                    for(KVV=1;KVV<=gas->ISPV[MS];KVV++)
                                                    {
                                                        JI=molecs->IPVIB[KVV][M];
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        A=A+(double)(JI)*BOLTZ*gas->SPVM[1][KVV][MS];
                                                    }
                                                }
                                                gas->TREACL[2][LS]=gas->TREACL[2][LS]-1;
                                                gas->TREACL[2][MS]=gas->TREACL[2][MS]-1;
                                                LSI=LS;
                                                MSI=MS;
                                                LS=gas->ISPRC[LS][MS];
                                                molecs->IPSP[L]=LS;
                                                //any additional vibrational modes must be set to zero
                                                IVM=gas->ISPV[LSI];
                                                NMC=molecs->IPSP[L];
                                                NVM=gas->ISPV[NMC];
                                                if(NVM>IVM)
                                                {
                                                    for(KV=IVM+1;KV<=NVM;KV++)
                                                    {
                                                        molecs->IPVIB[KV][L]=0;
                                                    }
                                                }
                                                if(gas->MELE>1)
                                                    molecs->PELE[KV]=0.e00;

                                                molecs->IPCELL[M]=-molecs->IPCELL[M]; //recombining molecule M marked for removal
                                                M=KT; //third body molecule is set as molecule M
                                                MS=KS;
                                                gas->TREACG[2][LS]=gas->TREACG[2][LS]+1;
                                                if(gas->ISPR[1][LS]>0)
                                                {
                                                    molecs->PROT[L]=0.e00;
                                                }
                                                if(gas->MELE>1)
                                                    molecs->PELE[L]=0.e00;
                                                if(gas->ISPV[LS]>0)
                                                {
                                                    for(KVV=1;KVV<=gas->ISPV[LS];KVV++)
                                                    {
                                                        if(molecs->IPVIB[KVV][L]<0)
                                                        {
                                                            molecs->IPVIB[KVV][L]=-99999;
                                                        }
                                                        else
                                                        {
                                                            molecs->IPVIB[KVV][L]=0;
                                                        }
                                                    }
                                                }
                                                if(gas->ISPR[1][MS]>0)
                                                {
                                                    molecs->PROT[M]=molecs->PROT[KT];
                                                }
                                                if(gas->MELE>1)
                                                    molecs->PELE[M]=molecs->PELE[KT];
                                                if(gas->ISPV[MS]>0)
                                                {
                                                    for(KVV=1;KVV<=gas->ISPV[MS];KVV++)
                                                    {
                                                        molecs->IPVIB[KVV][M]=molecs->IPVIB[KVV][KT];
                                                    }
                                                }
                                                ECTOT=A+gas->SPVM[4][1][LS]*BOLTZ ; //the energy added to this collision
                                                for(KK=1;KK<=3;KK++)
                                                {
                                                    molecs->PV[KK][L]=VCM[KK];
                                                }
                                                for(KK=1;KK<=3;KK++)
                                                {
                                                    VRC[KK]=molecs->PV[KK][L]-molecs->PV[KK][M];
                                                }
                                                VRR=VRC[1]*VRC[1]+VRC[2]*VRC[2]+VRC[3]*VRC[3];
                                                ECT=0.5e00*gas->SPM[1][LS][MS]*VRR*ECTOT;
                                                //set the vibrational energy of the recombined molecule L to enforce detailed balance
                                                IK=-1;
                                                NK=-1;
                                                //dout
                                                //                                                    RANDOM_NUMBER(RANF);
                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                //NTRY=0;
                                                while(IK<0)
                                                {
                                                    // NTRY+=1;
                                                    // if(NTRY>100)
                                                    //   cout<<"NTRY VibEn"<<NTRY;
                                                    NK=NK+1;
                                                    BB=(output->VAR[8][NN]-gas->SPRT[1][LSI][MSI])*(gas->SPRP[2][LSI][MSI][NK]-gas->SPRP[1][LSI][MSI][NK])/(gas->SPRT[2][LSI][MSI]-gas->SPRT[1][LSI][MSI])-gas->SPRP[1][LSI][MSI][NK];
                                                    if(calc->RANF<BB)
                                                        IK=NK;
                                                }
                                                molecs->IPVIB[1][L]=IK;
                                                ECT=ECT-(double)(IK)*BOLTZ*gas->SPVM[1][gas->ISPRK[LSI][MSI]][LS];
                                                VRR=2.e00*ECT/gas->SPM[1][LS][MS];
                                                VR=sqrtf(VRR);
                                                RML=gas->SPM[1][LS][MS]/gas->SP[5][MS];
                                                RMM=gas->SPM[1][LS][MS]/gas->SP[5][LS];
                                                for(KK=1;KK<=3;KK++)
                                                {
                                                    VCM[KK]=RML*molecs->PV[KK][L]+RMM*molecs->PV[KK][M];
                                                }
                                            }
                                        }
                                        //consider exchange and chain reactions
                                        if(gas->NSPEX[LS][MS]>0 && IREC==0 && IDISS==0)
                                        {
                                            //possible exchange reaction
                                            //memset(gas->PSF,0.e00,sizeof(*gas->PSF));//gas->PSF=0.e00; //PSF(MMEX) PSF is the probability that this reaction will occur in this collision
                                            for(int i=0;i<gas->MMEX+1;i++)
                                                gas->PSF[i]=0.e00;
                                            
                                            for(JJ=1;JJ<=gas->NSPEX[LS][MS];JJ++)
                                            {
                                                if(LS==gas->ISPEX[JJ][1][LS][MS])
                                                {
                                                    K=L; KS=LS;JS=MS;
                                                }
                                                else
                                                {
                                                    K=M; KS=MS; JS=LS;
                                                }
                                                //the pre-collision molecule that splits is K of species KS
                                                if(gas->SPEX[3][JJ][LS][MS]<0.e00)
                                                    KV=gas->ISPEX[JJ][5][LS][MS];
                                                if(gas->SPEX[3][JJ][LS][MS]>0.e00)
                                                {
                                                    KV=gas->ISPEX[JJ][7][LS][MS];
                                                }
                                                JI=molecs->IPVIB[KV][K];
                                                if(JI<0)
                                                    JI=-JI;
                                                if(JI==99999)
                                                    JI=0;
                                                ECC=0.5e00*gas->SPM[1][LS][MS]*VRR+(double)(JI)*BOLTZ*gas->SPVM[1][KV][KS];
                                                if(gas->SPEX[3][JJ][KS][JS]>0.e00)
                                                {
                                                    //reverse exothermic reaction
                                                    gas->PSF[JJ]=(gas->SPEX[1][JJ][KS][JS]*powf(output->VAR[8][NN]/273.e00,gas->SPEX[2][JJ][KS][JS]))*expf(-gas->SPEX[6][JJ][KS][JS]/(BOLTZ*output->VAR[8][NN]));
                                                }
                                                else
                                                {
                                                    //forward endothermic reaction
                                                    MAXLEV=ECC/(BOLTZ*gas->SPVM[1][KV][KS]);
                                                    EA=fabsf(gas->SPEX[3][JJ][KS][JS]); //temporarily just the heat of reaction;
                                                    if(ECC>EA)
                                                    {
                                                        //the collision energy must exceed the heat of reaction
                                                        EA=EA+gas->SPEX[6][JJ][KS][JS]; //the activation energy now includes the energy barrier
                                                        DEN=0.e00;
                                                        for(IAX=0;IAX<=MAXLEV;IAX++)
                                                        {
                                                            DEN=DEN+powf((1.e00-(double)(IAX)*BOLTZ*gas->SPVM[1][KV][KS]/ECC),(1.5e00-gas->SPM[3][KS][JS]));
                                                        }
                                                        gas->PSF[JJ]=(double)(gas->ISPEX[JJ][6][LS][MS])*powf((1.e00-EA/ECC),(1.5e00-gas->SPM[3][KS][JS]))/DEN;
                                                    }
                                                }
                                            }
                                            if(gas->NSPEX[LS][MS]>1)
                                            {
                                                BB=0.e00;
                                                for(JJ=1;JJ<=gas->NSPEX[LS][MS];JJ++)
                                                {
                                                    BB=BB+gas->PSF[JJ];
                                                }
                                                //BB is the sum of the probabilities
                                                //dout
                                                //                                                    RANDOM_NUMBER(RANF);
                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                if(BB>calc->RANF)
                                                {
                                                    BB=0.e00;
                                                    IEX=0;
                                                    JJ=0;
                                                    //NTRY=0;
                                                    while(JJ<gas->NSPEX[LS][MS]&& IEX==0)
                                                    {
                                                        // NTRY=NTRY+1;
                                                        // if(NTRY>100)
                                                        // {
                                                        //   cout<<"NTRY find IEX"<<NTRY;
                                                        // }
                                                        JJ+=1;
                                                        BB+=gas->PSF[JJ];
                                                        if(BB>calc->RANF)
                                                            IEX=JJ;
                                                    }
                                                }
                                            }
                                            else
                                            {
                                                //dout
                                                //                                                    RANDOM_NUMBER(RANF);
                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                IEX=0;
                                                if(gas->PSF[1]>calc->RANF)
                                                    IEX=1;
                                            }
                                            if(IEX>0)
                                            {
                                                //exchange or chain reaction occurs
                                                JX=gas->NEX[IEX][LS][MS];
                                                //cout<<"Reaction"<<JX;
                                                gas->TNEX[JX]=gas->TNEX[JX]+1.e00;
                                                //cout<<IEX<<L<<M<<LS<<MS;
                                                molecs->IPSP[L]=gas->ISPEX[IEX][3][LS][MS]; //L is now the new molecule that splits
                                                molecs->IPSP[M]=gas->ISPEX[IEX][4][LS][MS];
                                                LSI=LS;
                                                MSI=MS;
                                                //any additional vibrational modes must be set to zero
                                                IVM=gas->ISPV[LS];
                                                NMC=molecs->IPCP[L];
                                                NVM=gas->ISPV[NMC];
                                                if(NVM>IVM)
                                                {
                                                    for(KV=IVM+1;KV<=NVM;KV++)
                                                    {
                                                        molecs->IPVIB[KV][L]=0;
                                                    }
                                                }
                                                IVM=gas->ISPV[MS];
                                                NMC=molecs->IPCP[M];
                                                NVM=gas->ISPV[NMC];
                                                if(NVM>IVM)
                                                {
                                                    for(KV=IVM+1;KV<=NVM;KV++)
                                                    {
                                                        molecs->IPVIB[KV][M]=0;
                                                    }
                                                }
                                                //put all pre-collision energies into the relative translational energy and adjust for the reaction energy
                                                ECT=0.5e00*gas->SPM[1][LS][MS]*VRR;
                                                if(gas->ISPR[1][LS]>0)
                                                    ECT=ECT+molecs->PROT[L];
                                                if(gas->MELE>1)
                                                    ECT=ECT+molecs->PELE[L];
                                                if(gas->ISPV[LS]>0)
                                                {
                                                    for(KV=1;KV<=gas->ISPV[LS];KV++)
                                                    {
                                                        JI=molecs->IPVIB[KV][L];
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        ECT=ECT+(double)(JI)*BOLTZ*gas->SPVM[1][KV][LS];
                                                    }
                                                }
                                                if(gas->ISPR[1][MS]>0)
                                                    ECT=ECT+molecs->PROT[M];
                                                if(gas->MELE>1)
                                                    ECT=ECT+molecs->PELE[M];
                                                if(gas->ISPV[MS]>0)
                                                {
                                                    for(KV=1;KV<=gas->ISPV[MS];KV++)
                                                    {
                                                        JI=molecs->IPVIB[KV][M];
                                                        if(JI<0)
                                                            JI=-JI;
                                                        if(JI==99999)
                                                            JI=0;
                                                        ECT=ECT+(double)(JI)*BOLTZ*gas->SPVM[1][KV][MS];
                                                    }
                                                }
                                                ECT=ECT+gas->SPEX[3][IEX][LS][MS];
                                                if(ECT<0.0)
                                                {
                                                    printf ("-VE ECT %f\n",ECT);
                                                    printf ("REACTION %d",JJ," BETWEEN %d",LS," & %d\n",MS);
                                                    // cout<<"-VE ECT "<<ECT<<endl;
                                                    // cout<<"REACTION "<<JJ<<" BETWEEN "<<LS<<" "<<MS<<endl;
                                                    //dout
                                                    //cin.get();
                                                    return ;
                                                }
                                                if(gas->SPEX[3][IEX][LS][MS]<0.e00)
                                                {
                                                    gas->TREACL[3][LS]=gas->TREACL[3][LS]-1;
                                                    gas->TREACL[3][MS]=gas->TREACL[3][MS]-1;
                                                    LS=molecs->IPSP[L] ;
                                                    MS=molecs->IPSP[M] ;
                                                    gas->TREACG[3][LS]=gas->TREACG[3][LS]+1;
                                                    gas->TREACG[3][MS]=gas->TREACG[3][MS]+1;
                                                }
                                                else
                                                {
                                                    gas->TREACL[4][LS]=gas->TREACL[4][LS]-1;
                                                    gas->TREACL[4][MS]=gas->TREACL[4][MS]-1;
                                                    LS=molecs->IPSP[L] ;
                                                    MS=molecs->IPSP[M] ;
                                                    gas->TREACG[4][LS]=gas->TREACG[4][LS]+1;
                                                    gas->TREACG[4][MS]=gas->TREACG[4][MS]+1;
                                                }
                                                RML=gas->SPM[1][LS][MS]/gas->SP[5][MS];
                                                RMM=gas->SPM[1][LS][MS]/gas->SP[5][LS];
                                                //calculate the new VRR to match ECT using the new molecular masses
                                                VRR=2.e00*ECT/gas->SPM[1][LS][MS];
                                                if(gas->ISPV[LS]>0)
                                                {
                                                    for(KV=1;gas->ISPV[LS];KV++)
                                                    {
                                                        if(molecs->IPVIB[KV][L]<0)
                                                        {
                                                            molecs->IPVIB[KV][L]=-99999;
                                                        }
                                                        else
                                                        {
                                                            molecs->IPVIB[KV][L]=0;
                                                        }
                                                    }
                                                }
                                                if(gas->ISPR[1][LS]>0)
                                                    molecs->PROT[L]=0;
                                                if(gas->MELE>1)
                                                    molecs->PELE[L]=0.e00;
                                                if(gas->ISPV[MS]>0)
                                                {
                                                    for(KV=1;gas->ISPV[MS];KV++)
                                                    {
                                                        if(molecs->IPVIB[KV][M]<0)
                                                        {
                                                            molecs->IPVIB[KV][M]=-99999;
                                                        }
                                                        else
                                                        {
                                                            molecs->IPVIB[KV][M]=0;
                                                        }
                                                    }
                                                }
                                                if(gas->ISPR[1][MS]>0)
                                                    molecs->PROT[M]=0;
                                                if(gas->MELE>1)
                                                    molecs->PELE[M]=0.e00;
                                                //set vibrational level of product molecule in exothermic reaction to enforce detailed balance
                                                if(gas->SPEX[3][IEX][LSI][MSI]>0.e00)
                                                {
                                                    //exothermic exchange or chain reaction
                                                    IK=-1; //becomes 0 when the level is chosen
                                                    NK=-1;
                                                    //dout
                                                    //                                                        RANDOM_NUMBER(RANF);
                                                    calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                    //NTRY=0;
                                                    while(IK<0)
                                                    {
                                                        // NTRY=NTRY+1;
                                                        // if(NTRY>100)
                                                        // {
                                                        //   cout>>"NTRY VibProd"<<NTRY<<endl;
                                                        // }
                                                        NK=NK+1;
                                                        BB=(output->VAR[8][NN]-gas->SPEX[4][IEX][LSI][MSI])*  (gas->SPREX[2][IEX][LSI][MSI][NK]-gas->SPREX[1][IEX][LSI][MSI][NK])/(gas->SPEX[5][IEX][LSI][MSI]-gas->SPEX[4][IEX][LSI][MSI])+gas->SPREX[1][IEX][LSI][MSI][NK];
                                                        if(calc->RANF<BB)
                                                            IK=NK;
                                                    }
                                                    if(gas->NSLEV[1][LS]>0)
                                                    {
                                                        IK+=gas->NSLEV[1][LS];
                                                        gas->NSLEV[1][LS]=0;
                                                    }
                                                    KV=gas->ISPEX[IEX][7][LSI][MSI];
                                                    molecs->IPVIB[KV][L]=IK;
                                                    EVIB=(double)(IK)*BOLTZ*gas->SPVM[1][KV][LS];
                                                    ECT=ECT-EVIB;
                                                    if(ECT<0.e00)
                                                    {
                                                        //NTRY=0;
                                                        while(ECT<0.e00)
                                                        {
                                                            //NTRY+=1;
                                                            // if(NTRY>100)
                                                            //     cout<<"NTRY ECT<0"<<NTRY<<endl;
                                                            molecs->IPVIB[KV][L]=molecs->IPVIB[KV][L]-1;
                                                            gas->NSLEV[1][LS]+=1;
                                                            ECT=ECT+BOLTZ*gas->SPVM[1][KV][LS];
                                                        }
                                                    }
                                                }
                                                else
                                                {
                                                    //for endothermic reaction, select vibration from vib. dist. at macroscopic temperature
                                                    //normal L-B selection would be from the excessively low energy after the endo. reaction
                                                    KV=gas->ISPEX[IEX][5][LS][MS];
                                                    //dout
                                                    SVIB(globalState, LS,output->VAR[8][NN],IK,KV,gas,calc);
                                                    if(gas->NSLEV[2][LS]>0)
                                                    {
                                                        IK=IK+gas->NSLEV[2][LS];
                                                        gas->NSLEV[2][LS]=0;
                                                    }
                                                    molecs->IPVIB[KV][L]=IK;
                                                    EVIB=(double)(IK)*BOLTZ*gas->SPVM[1][KV][LS];
                                                    ECT=ECT-EVIB;
                                                    if(ECT<0.e00)
                                                    {
                                                        //NTRY=0;
                                                        while(ECT<0.e00)
                                                        {
                                                            //NTRY+=1;
                                                            molecs->IPVIB[KV][L]-=1;
                                                            gas->NSLEV[2][LS]+=1;
                                                            ECT=ECT+BOLTZ*gas->SPVM[1][KV][LS];
                                                            // if(NTRY>100)
                                                            // {
                                                            //cout<<"NTRY ECT<0#2"<<NTRY<<endl;
                                                            // molecs->IPVIB[KV][L]=0;
                                                            //   ECT+=EVIB;
                                                            //   gas->NSLEV[2][LS]=0;
                                                            // }
                                                        }
                                                    }
                                                }
                                                //set rotational energy of molecule L to equilibrium at the macroscopic temperature
                                                SROT(globalState, LS,output->VAR[8][NN],molecs->PROT[L],gas,calc);
                                                if(gas->SLER[LS]>1.e-21)
                                                {
                                                    molecs->PROT[L]+=gas->SLER[LS];
                                                    gas->SLER[LS]=1.e-21;
                                                }
                                                ECT-=molecs->PROT[L];
                                                ABA=molecs->PROT[L];
                                                if(ECT<0.e00)
                                                {
                                                    //NTRY=0;
                                                    while(ECT<0.e00)
                                                    {
                                                        //NTRY+=1;
                                                        BB=0.5e00*molecs->PROT[L];
                                                        gas->SLER[LS]+=BB;
                                                        molecs->PROT[L]=BB;
                                                        ECT+=BB;
                                                        // if(NTRY>100)
                                                        // {
                                                        //   cout<<"NTRY ECT<0#3"<<NTRY<<L<<endl;
                                                        //   ECT+=ABA;
                                                        //   molecs->PROT[L]=0;
                                                        //   gas->SLER[LS]=1.e-21;
                                                        // }
                                                    }
                                                }
                                                //calculate the new VRR to match ECT using the new molecular masses
                                                VRR=2.e00*ECT/gas->SPM[1][LS][MS];
                                            }
                                        }
                                    }
                            
                                        //end of reactions other than the deferred dissociation action in the DISSOCIATION subroutine
                                    if(IREC==0 && IDISS==0)
                                    {
                                        //recombined redistribution already made and there is a separate subroutine for dissociation
                                        //Larsen-Borgnakke serial redistribution
                                        ECT=0.5e00*gas->SPM[1][LS][MS]*VRR;
                                        for(NSP=1;NSP<=2;NSP++)
                                        {
                                            if(NSP==1)
                                            {
                                                K=L;KS=LS;JS=MS;
                                            }
                                            else
                                            {
                                                K=M; KS=MS; JS=LS;
                                            }
                                            //now electronic energy for this molecule
                                            if(gas->MELE>1)
                                            {
                                                B=1.e00/gas->QELC[3][1][KS];
                                                //dout
                                                //                                                        RANDOM_NUMBER(RANF);
                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                if(B>calc->RANF)
                                                {
                                                    NPS=0;
                                                    ECC=ECT+molecs->PELE[K];
                                                    if(gas->NELL[KS]==1){
                                                        NPS=gas->QELC[1][1][KS]; //number of possible states is at least the degeneracy of the ground state
                                                    }
                                                    if(gas->NELL[KS]>1)
                                                    {
                                                        for(NEL=1;NEL<=gas->NELL[KS];NEL++)
                                                        {
                                                            if(ECC>BOLTZ*gas->QELC[2][NEL][KS])
                                                                NPS=NPS+gas->QELC[1][NEL][KS];
                                                        }
                                                        II=0;
                                                        //NTRY=0;
                                                        while(II==0)
                                                        {
                                                            //NTRY+=1;
                                                            // if(NTRY>100)
                                                            //           cout<<"NTRY ElecEn"<<NTRY<<endl;
                                                            //dout
                                                            //                                                                    RANDOM_NUMBER(RANF);
                                                            calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                            NSTATE=ceil(calc->RANF*NPS);//random state, now determine the energy level
                                                            NAS=0;
                                                            NLEVEL=-1;
                                                            for(NEL=1;NEL<=gas->NELL[KS];NEL++)
                                                            {
                                                                NAS= NAS+gas->QELC[1][NEL][KS];
                                                                if(NSTATE<=NAS && NLEVEL<0)
                                                                    NLEVEL=NEL;
                                                            }
                                                            //dout
                                                            //                                                                    RANDOM_NUMBER(RANF);
                                                            calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                            if((1.e00/(B*gas->QELC[3][NLEVEL][KS]))<calc->RANF)
                                                            {
                                                                II=1;
                                                            }
                                                            else
                                                            {
                                                                if(ECC>BOLTZ*gas->QELC[2][NLEVEL][KS])
                                                                {
                                                                    PROB=powf(1.e00-BOLTZ*gas->QELC[2][NLEVEL][KS]/ECC,(1.5e00-gas->SPM[3][KS][JS]));
                                                                    //dout
                                                                    //                                                                            RANDOM_NUMBER(RANF);
                                                                    calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                                    if(PROB>calc->RANF)
                                                                    {
                                                                        II=1;
                                                                        molecs->PELE[K]=BOLTZ*gas->QELC[2][NLEVEL][KS];
                                                                    }
                                                                }
                                                            }
                                                        }
                                                        ECT=ECC-molecs->PELE[K];
                                                    }
                                                }
                                            }
                                            //now the vibrational energy for this molecule
                                            if(gas->MMVM>0 && IEX==0)
                                            {
                                                if(gas->ISPV[KS]>0)
                                                {
                                                    for(KV=1;KV<=gas->ISPV[KS];KV++)
                                                    {
                                                        if(molecs->IPVIB[KV][K]>=0 && IDISS==0) //do not redistribute to a dissociating molecule marked for removal
                                                        {
                                                            EVIB=(double)(molecs->IPVIB[KV][K])*BOLTZ*gas->SPVM[1][KV][KS];
                                                            ECC=ECT+EVIB;
                                                            MAXLEV=ECC/(BOLTZ*gas->SPVM[1][KV][KS]);
                                                            if(gas->SPVM[3][KV][KS]>0.0)
                                                            {   
                                                                B=gas->SPVM[4][KV][KS]/gas->SPVM[3][KV][KS];
                                                                A=gas->SPVM[4][KV][KS]/output->VAR[8][NN];
                                                               ZV = powf(A,gas->SPM[3][KS][JS])*powf((gas->SPVM[2][KV][KS]*powf(B,-gas->SPM[3][KS][JS])),((powf(A,0.3333333e00)-1.e00)/(powf(B,0.33333e00)-1.e00)));
                                                               
                                                            }
                                                            else
                                                                ZV=gas->SPVM[2][KV][KS];
                                                            //                                                                    RANDOM_NUMBER(RANF) //dout
                                                            calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                            if(1.e00/ZV>calc->RANF ||IREC==1)
                                                            {
                                                                II=0;
                                                                NSTEP=0;
                                                                while(II==0 && NSTEP<100000)
                                                                {
                                                                    NSTEP+=1;
                                                                    if(NSTEP>99000)
                                                                    {
                                                                        printf("%d %f %d\n",NSTEP,ECC,MAXLEV);
                                                                        //cout<<NSTEP<<" "<<ECC<<" "<<MAXLEV<<endl;
                                                                        //dout
                                                                        return ;
                                                                    }
                                                                    //                                                                            RANDOM_NUMBER(RANF);
                                                                    calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                                    IV=calc->RANF*(MAXLEV+0.99999999e00);
                                                                    molecs->IPVIB[KV][K]=IV;
                                                                    EVIB=(double)(IV)*BOLTZ*gas->SPVM[1][KV][KS];
                                                                    if(EVIB<ECC)
                                                                    {
                                                                        PROB=powf(1.e00-EVIB/ECC,1.5e00-gas->SPVM[3][KS][JS]);
                                                                        //PROB is the probability ratio of eqn (3.28)
                                                                        //                                                                                RANDOM_NUMBER(RANF);
                                                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                                        if(PROB>calc->RANF)
                                                                            II=1;
                                                                    }
                                                                }
                                                                ECT=ECC-EVIB;
                                                            }
                                                        }
                                                    }
                                                }
                                            }
                                            //now rotation of this molecule
                                            //dout
                                            if(gas->ISPR[1][KS] > 0)
                                            {
                                                if(gas->ISPR[2][KS]==0 && gas->ISPR[2][JS]==0)
                                                {
                                                    B=1.e00/gas->SPM[7][KS][JS];
                                                }
                                                else
                                                    B=1.e00/(gas->SPR[1][KS])+gas->SPR[2][KS]*output->VAR[8][NN]+gas->SPR[3][KS]*powf(output->VAR[8][NN],2);
                                                //                                                        RANDOM_NUMBER(RANF);
                                                calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                if(B>calc->RANF|| IREC==1)
                                                {
                                                    ECC=ECT+molecs->PROT[K];
                                                    if(gas->ISPR[1][KS]==2)
                                                    {
                                                        //                                                                RANDOM_NUMBER(RANF);
                                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                                        ERM=1.e00-powf(calc->RANF,(1.e00/(2.5e00-gas->SPM[3][KS][JS])));//eqn(5.46)
                                                    }
                                                    else
                                                        LBS(globalState, 0.5e00*gas->ISPR[1][KS]-1.e00,1.5e00-gas->SPM[3][KS][JS],ERM);
                                                    molecs->PROT[K]=ERM*ECC;
                                                    ECT=ECC-molecs->PROT[K];
                                                }
                                            }
                                        }
                                        //adjust VR for the change in energy
                                        VR=sqrtf(2.e00*ECT/gas->SPM[1][LS][MS]);
                                    }//end of L-B redistribution
                                    if(fabsf(gas->SPM[8][LS][MS]-1.0)<0.001)
                                    {
                                        //use the VHS logic
                                        //                                                RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*calc->RANF-1.e00;
                                        //B is the cosine of a random elevation angle
                                        A=sqrtf(1.e00-B*B);
                                        VRCP[1]=B*VR;
                                        //                                                RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*calc->RANF;
                                        //C is a random azimuth angle;
                                        VRCP[2]=A*(double)cos(C)*VR;
                                        VRCP[3]=A*(double)sin(C)*VR;
                                    }
                                    else
                                    {
                                        //use the VSS logic
                                        //the VRCP terms do not allow properly for the change in VR - see new book  !STILL TO BE FIXED
                                        VRA=VR/VRI;
                                        //                                                RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        B=2.e00*powf(calc->RANF,gas->SP[4][1])-1.e00;
                                        // B is the cosine of the deflection angle for the VSS model
                                        A=sqrtf(1.e00-B*B);
                                        //                                                RANDOM_NUMBER(RANF);
                                        calc->RANF=generate(globalState, 0);//((double)rand()/(double)RAND_MAX);
                                        C=2.e00*PI*calc->RANF;
                                        OC=(double)cos(C);
                                        SD=(double)sin(C);
                                        D=sqrtf(powf(VRC[2],2)+powf(VRC[3],2));
                                        VRCP[1]=(B*VRC[1]+A*SD*D)*VRA;
                                        VRCP[2]=(B*VRC[2]+A*(VRI*VRC[3]*OC-VRC[1]*VRC[2]*SD)/D)*VRA;
                                        VRCP[3]=(B*VRC[3]+A*(VRI*VRC[2]*OC+VRC[1]*VRC[3]*SD)/D)*VRA;
                                        //the post-collision rel. velocity components are based on eqn (3.18)
                                    }
                                    for(KK=1;KK<=3;KK++)
                                    {
                                        molecs->PV[KK][L]=VCM[KK]+RMM*VRCP[KK];
                                        molecs->PV[KK][M]=VCM[KK]-RMM*VRCP[KK];
                                    }
                                    molecs->IPCP[L]=M;
                                    molecs->IPCP[M]=L;
                                    //call energy(0,E2)
                                    // !              IF (Dfabs(E2-E1) > 1.D-14) read(*,*)
                                }////collision occurrence
                            }
                        }//separate simplegas / mixture coding
                    }
                }
            }
        }
    //remove any recombined atoms
    
}

void COLLISIONS()
{   
    start =clock();
    double duration;
    int N=geom->NCCELLS;
    int gridSize;
    int blockSize=128;
    hiprandState* devStates;
    memset(calc->COLL_TOTCOL,0.e00,(N+1)*sizeof(double));
    
    gridSize = (N + blockSize - 1) / blockSize;
    hipMalloc ( &devStates, sizeof( hiprandState ) );
    // setup seeds
    setup_kernel <<< 1, 1 >>> ( devStates,unsigned(time(NULL)) );

    cuda_collisons<<<gridSize,blockSize >>>(devStates, molecs, output, geom, gas, calc);
    hipDeviceSynchronize();

    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

    //std::cout<<"printf: "<< duration <<'\n';
    colltime=duration;
    for(N=1;N<=geom->NCCELLS;N++){
        calc->TOTCOL=calc->TOTCOL+calc->COLL_TOTCOL[N];
    }
    for(int N=1;N<=molecs->NM;N++)
    {
        if(molecs->IPCELL[N]<0)
            REMOVE_MOL(N); 
    }
    return;
} 

